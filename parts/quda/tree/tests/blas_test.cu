#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <quda_internal.h>
#include <color_spinor_field.h>
#include <blas_quda.h>

#include <test_util.h>
#include <face_quda.h>

// include because of nasty globals used in the tests
#include <dslash_util.h>

// google test
#include <gtest.h>

// Wilson, clover-improved Wilson, and twisted mass are supported.
extern QudaDslashType dslash_type;
extern bool tune;
extern int device;
extern int xdim;
extern int ydim;
extern int zdim;
extern int tdim;
extern int gridsize_from_cmdline[];
extern int niter;

extern bool tune;
extern bool verify_results;

extern void usage(char** );

const int Nkernels = 32;

using namespace quda;

cpuColorSpinorField *xH, *yH, *zH, *wH, *vH, *hH, *lH;
cudaColorSpinorField *xD, *yD, *zD, *wD, *vD, *hD, *lD;
int Nspin;

void setPrec(ColorSpinorParam &param, const QudaPrecision precision)
{
  param.precision = precision;
  if (Nspin == 1 || precision == QUDA_DOUBLE_PRECISION) {
    param.fieldOrder = QUDA_FLOAT2_FIELD_ORDER;
  } else {
    param.fieldOrder = QUDA_FLOAT4_FIELD_ORDER;
  }
}

void
display_test_info()
{
  printfQuda("running the following test:\n");
    
  printfQuda("S_dimension T_dimension Nspin\n");
  printfQuda("%d/%d/%d        %d      %d\n", xdim, ydim, zdim, tdim, Nspin);     

  printfQuda("Grid partition info:     X  Y  Z  T\n"); 
  printfQuda("                         %d  %d  %d  %d\n", 
	     dimPartitioned(0),
	     dimPartitioned(1),
	     dimPartitioned(2),
	     dimPartitioned(3)); 
  
  return;  
}

void initFields(int prec)
{
  // precisions used for the source field in the copyCuda() benchmark
  QudaPrecision high_aux_prec;
  QudaPrecision low_aux_prec;

  ColorSpinorParam param;
  param.nColor = 3;
  // set spin according to the type of dslash
  Nspin = (dslash_type == QUDA_ASQTAD_DSLASH || 
	   dslash_type == QUDA_STAGGERED_DSLASH) ? 1 : 4;
  param.nSpin = Nspin;
  param.nDim = 4; // number of spacetime dimensions

  param.pad = 0; // padding must be zero for cpu fields
  param.siteSubset = QUDA_PARITY_SITE_SUBSET;
  if (param.siteSubset == QUDA_PARITY_SITE_SUBSET) param.x[0] = xdim/2;
  else param.x[0] = xdim;
  param.x[1] = ydim;
  param.x[2] = zdim;
  param.x[3] = tdim;

  param.siteOrder = QUDA_EVEN_ODD_SITE_ORDER;
  param.gammaBasis = QUDA_DEGRAND_ROSSI_GAMMA_BASIS;
  param.precision = QUDA_DOUBLE_PRECISION;
  param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;

  param.create = QUDA_ZERO_FIELD_CREATE;

  vH = new cpuColorSpinorField(param);
  wH = new cpuColorSpinorField(param);
  xH = new cpuColorSpinorField(param);
  yH = new cpuColorSpinorField(param);
  zH = new cpuColorSpinorField(param);
  hH = new cpuColorSpinorField(param);
  lH = new cpuColorSpinorField(param);

  vH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  wH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  xH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  yH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  zH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  hH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  lH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);

  // Now set the parameters for the cuda fields
  //param.pad = xdim*ydim*zdim/2;
  
  if (param.nSpin == 4) param.gammaBasis = QUDA_UKQCD_GAMMA_BASIS;
  param.create = QUDA_ZERO_FIELD_CREATE;

  switch(prec) {
  case 0:
    setPrec(param, QUDA_HALF_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_SINGLE_PRECISION;
    break;
  case 1:
    setPrec(param, QUDA_SINGLE_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  case 2:
    setPrec(param, QUDA_DOUBLE_PRECISION);
    high_aux_prec = QUDA_SINGLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  }

  checkCudaError();

  vD = new cudaColorSpinorField(param);
  wD = new cudaColorSpinorField(param);
  xD = new cudaColorSpinorField(param);
  yD = new cudaColorSpinorField(param);
  zD = new cudaColorSpinorField(param);

  setPrec(param, high_aux_prec);
  hD = new cudaColorSpinorField(param);

  setPrec(param, low_aux_prec);
  lD = new cudaColorSpinorField(param);

  // check for successful allocation
  checkCudaError();

  *vD = *vH;
  *wD = *wH;
  *xD = *xH;
  *yD = *yH;
  *zD = *zH;
  *hD = *hH;
  *lD = *lH;
}


void freeFields()
{

  // release memory
  delete vD;
  delete wD;
  delete xD;
  delete yD;
  delete zD;
  delete hD;
  delete lD;

  // release memory
  delete vH;
  delete wH;
  delete xH;
  delete yH;
  delete zH;
  delete hH;
  delete lH;
}


double benchmark(int kernel, const int niter) {

  double a, b, c;
  quda::Complex a2, b2, c2;

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

  {
    switch (kernel) {

    case 0:
      for (int i=0; i < niter; ++i) copyCuda(*yD, *hD);
      break;

    case 1:
      for (int i=0; i < niter; ++i) copyCuda(*yD, *lD);
      break;
      
    case 2:
      for (int i=0; i < niter; ++i) axpbyCuda(a, *xD, b, *yD);
      break;

    case 3:
      for (int i=0; i < niter; ++i) xpyCuda(*xD, *yD);
      break;

    case 4:
      for (int i=0; i < niter; ++i) axpyCuda(a, *xD, *yD);
      break;

    case 5:
      for (int i=0; i < niter; ++i) xpayCuda(*xD, a, *yD);
      break;

    case 6:
      for (int i=0; i < niter; ++i) mxpyCuda(*xD, *yD);
      break;

    case 7:
      for (int i=0; i < niter; ++i) axCuda(a, *xD);
      break;

    case 8:
      for (int i=0; i < niter; ++i) caxpyCuda(a2, *xD, *yD);
      break;

    case 9:
      for (int i=0; i < niter; ++i) caxpbyCuda(a2, *xD, b2, *yD);
      break;

    case 10:
      for (int i=0; i < niter; ++i) cxpaypbzCuda(*xD, a2, *yD, b2, *zD);
      break;

    case 11:
      for (int i=0; i < niter; ++i) axpyBzpcxCuda(a, *xD, *yD, b, *zD, c);
      break;

    case 12:
      for (int i=0; i < niter; ++i) axpyZpbxCuda(a, *xD, *yD, *zD, b);
      break;

    case 13:
      for (int i=0; i < niter; ++i) caxpbypzYmbwCuda(a2, *xD, b2, *yD, *zD, *wD);
      break;
      
    case 14:
      for (int i=0; i < niter; ++i) cabxpyAxCuda(a, b2, *xD, *yD);
      break;

    case 15:
      for (int i=0; i < niter; ++i) caxpbypzCuda(a2, *xD, b2, *yD, *zD);
      break;

    case 16:
      for (int i=0; i < niter; ++i) caxpbypczpwCuda(a2, *xD, b2, *yD, c2, *zD, *wD);
      break;

    case 17:
      for (int i=0; i < niter; ++i) caxpyXmazCuda(a2, *xD, *yD, *zD);
      break;

      // double
    case 18:
      for (int i=0; i < niter; ++i) normCuda(*xD);
      break;

    case 19:
      for (int i=0; i < niter; ++i) reDotProductCuda(*xD, *yD);
      break;

    case 20:
      for (int i=0; i < niter; ++i) axpyNormCuda(a, *xD, *yD);
      break;

    case 21:
      for (int i=0; i < niter; ++i) xmyNormCuda(*xD, *yD);
      break;
      
    case 22:
      for (int i=0; i < niter; ++i) caxpyNormCuda(a2, *xD, *yD);
      break;

    case 23:
      for (int i=0; i < niter; ++i) caxpyXmazNormXCuda(a2, *xD, *yD, *zD);
      break;

    case 24:
      for (int i=0; i < niter; ++i) cabxpyAxNormCuda(a, b2, *xD, *yD);
      break;

    // double2
    case 25:
      for (int i=0; i < niter; ++i) cDotProductCuda(*xD, *yD);
      break;

    case 26:
      for (int i=0; i < niter; ++i) xpaycDotzyCuda(*xD, a, *yD, *zD);
      break;
      
    case 27:
      for (int i=0; i < niter; ++i) caxpyDotzyCuda(a2, *xD, *yD, *zD);
      break;

    // double3
    case 28:
      for (int i=0; i < niter; ++i) cDotProductNormACuda(*xD, *yD);
      break;

    case 29:
      for (int i=0; i < niter; ++i) cDotProductNormBCuda(*xD, *yD);
      break;

    case 30:
      for (int i=0; i < niter; ++i) caxpbypzYmbwcDotProductUYNormYCuda(a2, *xD, b2, *yD, *zD, *wD, *vD);
      break;

    case 31:
      for (int i=0; i < niter; ++i) HeavyQuarkResidualNormCuda(*xD, *yD);
      break;

    default:
      errorQuda("Undefined blas kernel %d\n", kernel);
    }
  }
  
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float runTime;
  hipEventElapsedTime(&runTime, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);

  double secs = runTime / 1000;
  return secs;
}

#define ERROR(a) fabs(norm2(*a##D) - norm2(*a##H)) / norm2(*a##H)

double test(int kernel) {

  double a = M_PI, b = M_PI*exp(1.0), c = sqrt(M_PI);
  quda::Complex a2(a, b), b2(b, -c), c2(a+b, c*a);
  double error = 0;

  switch (kernel) {

  case 0:
    *hD = *hH;
    copyCuda(*yD, *hD);
    yH->copy(*hH);
    error = ERROR(y);
    break;

  case 1:
    *lD = *lH;
    copyCuda(*yD, *lD);
    yH->copy(*lH);
    error = ERROR(y);
    break;
      
  case 2:
    *xD = *xH;
    *yD = *yH;
    axpbyCuda(a, *xD, b, *yD);
    axpbyCpu(a, *xH, b, *yH);
    error = ERROR(y);
    break;

  case 3:
    *xD = *xH;
    *yD = *yH;
    xpyCuda(*xD, *yD);
    xpyCpu(*xH, *yH);
    error = ERROR(y);
    break;

  case 4:
    *xD = *xH;
    *yD = *yH;
    axpyCuda(a, *xD, *yD);
    axpyCpu(a, *xH, *yH);
    error = ERROR(y);
    break;

  case 5:
    *xD = *xH;
    *yD = *yH;
    xpayCuda(*xD, a, *yD);
    xpayCpu(*xH, a, *yH);
    error = ERROR(y);
    break;

  case 6:
    *xD = *xH;
    *yD = *yH;
    mxpyCuda(*xD, *yD);
    mxpyCpu(*xH, *yH);
    error = ERROR(y);
    break;

  case 7:
    *xD = *xH;
    axCuda(a, *xD);
    axCpu(a, *xH);
    error = ERROR(x);
    break;

  case 8:
    *xD = *xH;
    *yD = *yH;
    caxpyCuda(a2, *xD, *yD);
    caxpyCpu(a2, *xH, *yH);
    error = ERROR(y);
    break;

  case 9:
    *xD = *xH;
    *yD = *yH;
    caxpbyCuda(a2, *xD, b2, *yD);
    caxpbyCpu(a2, *xH, b2, *yH);
    error = ERROR(y);
    break;

  case 10:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    cxpaypbzCuda(*xD, a2, *yD, b2, *zD);
    cxpaypbzCpu(*xH, a2, *yH, b2, *zH);
    error = ERROR(z);
    break;

  case 11:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    axpyBzpcxCuda(a, *xD, *yD, b, *zD, c);
    axpyBzpcxCpu(a, *xH, *yH, b, *zH, c);
    error = ERROR(x) + ERROR(y);
    break;

  case 12:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    axpyZpbxCuda(a, *xD, *yD, *zD, b);
    axpyZpbxCpu(a, *xH, *yH, *zH, b);
    error = ERROR(x) + ERROR(y);
    break;

  case 13:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    caxpbypzYmbwCuda(a2, *xD, b2, *yD, *zD, *wD);
    caxpbypzYmbwCpu(a2, *xH, b2, *yH, *zH, *wH);
    error = ERROR(z) + ERROR(y);
    break;
      
  case 14:
    *xD = *xH;
    *yD = *yH;
    cabxpyAxCuda(a, b2, *xD, *yD);
    cabxpyAxCpu(a, b2, *xH, *yH);
    error = ERROR(y) + ERROR(x);
    break;

  case 15:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {caxpbypzCuda(a2, *xD, b2, *yD, *zD);
      caxpbypzCpu(a2, *xH, b2, *yH, *zH);
      error = ERROR(z); }
    break;
    
  case 16:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    {caxpbypczpwCuda(a2, *xD, b2, *yD, c2, *zD, *wD);
      caxpbypczpwCpu(a2, *xH, b2, *yH, c2, *zH, *wH);
      error = ERROR(w); }
    break;

  case 17:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {caxpyXmazCuda(a, *xD, *yD, *zD);
     caxpyXmazCpu(a, *xH, *yH, *zH);
     error = ERROR(y) + ERROR(x);}
    break;

    // double
  case 18:
    *xD = *xH;
    error = fabs(normCuda(*xD) - normCpu(*xH)) / normCpu(*xH);
    break;
    
  case 19:
    *xD = *xH;
    *yD = *yH;
    error = fabs(reDotProductCuda(*xD, *yD) - reDotProductCpu(*xH, *yH)) / fabs(reDotProductCpu(*xH, *yH));
    break;

  case 20:
    *xD = *xH;
    *yD = *yH;
    {double d = axpyNormCuda(a, *xD, *yD);
    double h = axpyNormCpu(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 21:
    *xD = *xH;
    *yD = *yH;
    {double d = xmyNormCuda(*xD, *yD);
    double h = xmyNormCpu(*xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;
    
  case 22:
    *xD = *xH;
    *yD = *yH;
    {double d = caxpyNormCuda(a, *xD, *yD);
    double h = caxpyNormCpu(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 23:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {double d = caxpyXmazNormXCuda(a, *xD, *yD, *zD);
      double h = caxpyXmazNormXCpu(a, *xH, *yH, *zH);
      error = ERROR(y) + ERROR(x) + fabs(d-h)/fabs(h);}
    break;

  case 24:
    *xD = *xH;
    *yD = *yH;
    {double d = cabxpyAxNormCuda(a, b2, *xD, *yD);
      double h = cabxpyAxNormCpu(a, b2, *xH, *yH);
      error = ERROR(x) + ERROR(y) + fabs(d-h)/fabs(h);}
    break;

    // double2
  case 25:
    *xD = *xH;
    *yD = *yH;
    error = abs(cDotProductCuda(*xD, *yD) - cDotProductCpu(*xH, *yH)) / abs(cDotProductCpu(*xH, *yH));
    break;
    
  case 26:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { quda::Complex d = xpaycDotzyCuda(*xD, a, *yD, *zD);
      quda::Complex h = xpaycDotzyCpu(*xH, a, *yH, *zH);
      error =  fabs(norm2(*yD) - norm2(*yH)) / norm2(*yH) + abs(d-h)/abs(h);
    }
    break;
    
  case 27:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {quda::Complex d = caxpyDotzyCuda(a, *xD, *yD, *zD);
      quda::Complex h = caxpyDotzyCpu(a, *xH, *yH, *zH);
    error = ERROR(y) + abs(d-h)/abs(h);}
    break;

    // double3
  case 28:
    *xD = *xH;
    *yD = *yH;
    { double3 d = cDotProductNormACuda(*xD, *yD);
      double3 h = cDotProductNormACpu(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;
    
  case 29:
    *xD = *xH;
    *yD = *yH;
    { double3 d = cDotProductNormBCuda(*xD, *yD);
      double3 h = cDotProductNormBCpu(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;
    
  case 30:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    *vD = *vH;
    { double3 d = caxpbypzYmbwcDotProductUYNormYCuda(a2, *xD, b2, *yD, *zD, *wD, *vD);
      double3 h = caxpbypzYmbwcDotProductUYNormYCpu(a2, *xH, b2, *yH, *zH, *wH, *vH);
      error = ERROR(z) + ERROR(y) + fabs(d.x - h.x) / fabs(h.x) + 
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 31:
    *xD = *xH;
    *yD = *yH;
    { double3 d = HeavyQuarkResidualNormCuda(*xD, *yD);
      double3 h = HeavyQuarkResidualNormCpu(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + 
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  default:
    errorQuda("Undefined blas kernel %d\n", kernel);
  }

  return error;
}

int Nprec = 3;

const char *prec_str[] = {"half", "single", "double"};

const char *names[] = {
  "copyHS",
  "copyLS",
  "axpby",
  "xpy",
  "axpy",
  "xpay",
  "mxpy",
  "ax",
  "caxpy",
  "caxpby",
  "cxpaypbz",
  "axpyBzpcx",
  "axpyZpbx",
  "caxpbypzYmbw",
  "cabxpyAx",
  "caxpbypz",
  "caxpbypczpw",
  "caxpyXmaz",
  "norm",
  "reDotProduct",
  "axpyNorm",
  "xmyNorm",
  "caxpyNorm",
  "caxpyXmazNormX",
  "cabxpyAxNorm",
  "cDotProduct",
  "xpaycDotzy",
  "caxpyDotzy",
  "cDotProductNormA",
  "cDotProductNormB",
  "caxpbypzYmbwcDotProductWYNormY",
  "HeavyQuarkResidualNorm"
};

int main(int argc, char** argv)
{
  for (int i = 1; i < argc; i++){
    if(process_command_line_option(argc, argv, &i) == 0){
      continue;
    } 
    printfQuda("ERROR: Invalid option:%s\n", argv[i]);
    usage(argv);
  }

  setSpinorSiteSize(24);
  initComms(argc, argv, gridsize_from_cmdline);
  display_test_info();
  initQuda(device);

  // enable the tuning
  setTuning(tune ? QUDA_TUNE_YES : QUDA_TUNE_NO);
  setVerbosity(QUDA_SILENT);

  for (int prec = 0; prec < Nprec; prec++) {

    printfQuda("\nBenchmarking %s precision with %d iterations...\n\n", prec_str[prec], niter);
    initFields(prec);

    for (int kernel = 0; kernel < Nkernels; kernel++) {
      // only benchmark "high precision" copyCuda() if double is supported
      if ((Nprec < 3) && (kernel == 0)) continue;

      // do the initial tune
      benchmark(kernel, 1);
    
      // now rerun with more iterations to get accurate speed measurements
      quda::blas_flops = 0;
      quda::blas_bytes = 0;
      
      double secs = benchmark(kernel, niter);
      
      double gflops = (quda::blas_flops*1e-9)/(secs);
      double gbytes = quda::blas_bytes/(secs*1e9);
    
      printfQuda("%-31s: Gflop/s = %6.1f, GB/s = %6.1f\n", names[kernel], gflops, gbytes);
    }
    freeFields();
  }

  // clear the error state
  hipGetLastError();

  // lastly check for correctness
  if (verify_results) {
    ::testing::InitGoogleTest(&argc, argv);
    if (RUN_ALL_TESTS() != 0) warningQuda("Tests failed");
  }

  endQuda();

  finalizeComms();
}

// The following tests each kernel at each precision using the google testing framework

class BlasTest : public ::testing::TestWithParam<int2> {
protected:
  int2 param;

public:
  virtual ~BlasTest() { }
  virtual void SetUp() { 
    param = GetParam();
    initFields(param.x); 
  }
  virtual void TearDown() { freeFields(); }

  virtual void NormalExit() { printf("monkey\n"); }

};

TEST_P(BlasTest, verify) {
  int prec = param.x;
  int kernel = param.y;
  double deviation = test(kernel);
  printfQuda("%-35s error = %e\n", names[kernel], deviation);
  double tol = (prec == 2 ? 1e-12 : (prec == 1 ? 1e-5 : 1e-3));
  tol = (kernel < 2) ? 1e-4 : tol; // use different tolerance for copy
  EXPECT_LE(deviation, tol) << "CPU and CUDA implementations do not agree";
}

// half precision
INSTANTIATE_TEST_CASE_P(copyHS_half, BlasTest, ::testing::Values( make_int2(0,0) ));
INSTANTIATE_TEST_CASE_P(copyLS_half, BlasTest, ::testing::Values( make_int2(0,1) ));
INSTANTIATE_TEST_CASE_P(axpby_half, BlasTest, ::testing::Values( make_int2(0,2) ));
INSTANTIATE_TEST_CASE_P(xpy_half, BlasTest, ::testing::Values( make_int2(0,3) ));
INSTANTIATE_TEST_CASE_P(axpy_half, BlasTest, ::testing::Values( make_int2(0,4) ));
INSTANTIATE_TEST_CASE_P(xpay_half, BlasTest, ::testing::Values( make_int2(0,5) ));
INSTANTIATE_TEST_CASE_P(mxpy_half, BlasTest, ::testing::Values( make_int2(0,6) ));
INSTANTIATE_TEST_CASE_P(ax_half, BlasTest, ::testing::Values( make_int2(0,7) ));
INSTANTIATE_TEST_CASE_P(caxpy_half, BlasTest, ::testing::Values( make_int2(0,8) ));
INSTANTIATE_TEST_CASE_P(caxpby_half, BlasTest, ::testing::Values( make_int2(0,9) ));
INSTANTIATE_TEST_CASE_P(cxpaypbz_half, BlasTest, ::testing::Values( make_int2(0,10) ));
INSTANTIATE_TEST_CASE_P(axpyBzpcx_half, BlasTest, ::testing::Values( make_int2(0,11) ));
INSTANTIATE_TEST_CASE_P(axpyZpbx_half, BlasTest, ::testing::Values( make_int2(0,12) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbw_half, BlasTest, ::testing::Values( make_int2(0,13) ));
INSTANTIATE_TEST_CASE_P(cabxpyAx_half, BlasTest, ::testing::Values( make_int2(0,14) ));
INSTANTIATE_TEST_CASE_P(caxpbypz_half, BlasTest, ::testing::Values( make_int2(0,15) ));
INSTANTIATE_TEST_CASE_P(caxpbypczpw_half, BlasTest, ::testing::Values( make_int2(0,16) ));
INSTANTIATE_TEST_CASE_P(caxpyXmaz_half, BlasTest, ::testing::Values( make_int2(0,17) ));
INSTANTIATE_TEST_CASE_P(norm2_half, BlasTest, ::testing::Values( make_int2(0,18) ));
INSTANTIATE_TEST_CASE_P(reDotProduct_half, BlasTest, ::testing::Values( make_int2(0,19) ));
INSTANTIATE_TEST_CASE_P(axpyNorm_half, BlasTest, ::testing::Values( make_int2(0,20) ));
INSTANTIATE_TEST_CASE_P(xmyNorm_half, BlasTest, ::testing::Values( make_int2(0,21) ));
INSTANTIATE_TEST_CASE_P(caxpyNorm_half, BlasTest, ::testing::Values( make_int2(0,22) ));
INSTANTIATE_TEST_CASE_P(caxpyXmazNormX_half, BlasTest, ::testing::Values( make_int2(0,23) ));
INSTANTIATE_TEST_CASE_P(cabxpyAxNorm_half, BlasTest, ::testing::Values( make_int2(0,24) ));
INSTANTIATE_TEST_CASE_P(cDotProduct_half, BlasTest, ::testing::Values( make_int2(0,25) ));
INSTANTIATE_TEST_CASE_P(xpaycDotzy_half, BlasTest, ::testing::Values( make_int2(0,26) ));
INSTANTIATE_TEST_CASE_P(caxpyDotzy_half, BlasTest, ::testing::Values( make_int2(0,27) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormA_half, BlasTest, ::testing::Values( make_int2(0,28) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormB_half, BlasTest, ::testing::Values( make_int2(0,29) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbwcDotProductWYNormY_half, BlasTest, ::testing::Values( make_int2(0,30) ));
INSTANTIATE_TEST_CASE_P(HeavyQuarkResidualNorm_half, BlasTest, ::testing::Values( make_int2(0,31) ));

// single precision
INSTANTIATE_TEST_CASE_P(copyHS_single, BlasTest, ::testing::Values( make_int2(1,0) ));
INSTANTIATE_TEST_CASE_P(copyLS_single, BlasTest, ::testing::Values( make_int2(1,1) ));
INSTANTIATE_TEST_CASE_P(axpby_single, BlasTest, ::testing::Values( make_int2(1,2) ));
INSTANTIATE_TEST_CASE_P(xpy_single, BlasTest, ::testing::Values( make_int2(1,3) ));
INSTANTIATE_TEST_CASE_P(axpy_single, BlasTest, ::testing::Values( make_int2(1,4) ));
INSTANTIATE_TEST_CASE_P(xpay_single, BlasTest, ::testing::Values( make_int2(1,5) ));
INSTANTIATE_TEST_CASE_P(mxpy_single, BlasTest, ::testing::Values( make_int2(1,6) ));
INSTANTIATE_TEST_CASE_P(ax_single, BlasTest, ::testing::Values( make_int2(1,7) ));
INSTANTIATE_TEST_CASE_P(caxpy_single, BlasTest, ::testing::Values( make_int2(1,8) ));
INSTANTIATE_TEST_CASE_P(caxpby_single, BlasTest, ::testing::Values( make_int2(1,9) ));
INSTANTIATE_TEST_CASE_P(cxpaypbz_single, BlasTest, ::testing::Values( make_int2(1,10) ));
INSTANTIATE_TEST_CASE_P(axpyBzpcx_single, BlasTest, ::testing::Values( make_int2(1,11) ));
INSTANTIATE_TEST_CASE_P(axpyZpbx_single, BlasTest, ::testing::Values( make_int2(1,12) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbw_single, BlasTest, ::testing::Values( make_int2(1,13) ));
INSTANTIATE_TEST_CASE_P(cabxpyAx_single, BlasTest, ::testing::Values( make_int2(1,14) ));
INSTANTIATE_TEST_CASE_P(caxpbypz_single, BlasTest, ::testing::Values( make_int2(1,15) ));
INSTANTIATE_TEST_CASE_P(caxpbypczpw_single, BlasTest, ::testing::Values( make_int2(1,16) ));
INSTANTIATE_TEST_CASE_P(caxpyXmaz_single, BlasTest, ::testing::Values( make_int2(1,17) ));
INSTANTIATE_TEST_CASE_P(norm2_single, BlasTest, ::testing::Values( make_int2(1,18) ));
INSTANTIATE_TEST_CASE_P(reDotProduct_single, BlasTest, ::testing::Values( make_int2(1,19) ));
INSTANTIATE_TEST_CASE_P(axpyNorm_single, BlasTest, ::testing::Values( make_int2(1,20) ));
INSTANTIATE_TEST_CASE_P(xmyNorm_single, BlasTest, ::testing::Values( make_int2(1,21) ));
INSTANTIATE_TEST_CASE_P(caxpyNorm_single, BlasTest, ::testing::Values( make_int2(1,22) ));
INSTANTIATE_TEST_CASE_P(caxpyXmazNormX_single, BlasTest, ::testing::Values( make_int2(1,23) ));
INSTANTIATE_TEST_CASE_P(cabxpyAxNorm_single, BlasTest, ::testing::Values( make_int2(1,24) ));
INSTANTIATE_TEST_CASE_P(cDotProduct_single, BlasTest, ::testing::Values( make_int2(1,25) ));
INSTANTIATE_TEST_CASE_P(xpaycDotzy_single, BlasTest, ::testing::Values( make_int2(1,26) ));
INSTANTIATE_TEST_CASE_P(caxpyDotzy_single, BlasTest, ::testing::Values( make_int2(1,27) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormA_single, BlasTest, ::testing::Values( make_int2(1,28) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormB_single, BlasTest, ::testing::Values( make_int2(1,29) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbwcDotProductWYNormY_single, BlasTest, ::testing::Values( make_int2(1,30) ));
INSTANTIATE_TEST_CASE_P(HeavyQuarkResidualNorm_single, BlasTest, ::testing::Values( make_int2(1,31) ));

// double precision
INSTANTIATE_TEST_CASE_P(copyHS_double, BlasTest, ::testing::Values( make_int2(2,0) ));
INSTANTIATE_TEST_CASE_P(copyLS_double, BlasTest, ::testing::Values( make_int2(2,1) ));
INSTANTIATE_TEST_CASE_P(axpby_double, BlasTest, ::testing::Values( make_int2(2,2) ));
INSTANTIATE_TEST_CASE_P(xpy_double, BlasTest, ::testing::Values( make_int2(2,3) ));
INSTANTIATE_TEST_CASE_P(axpy_double, BlasTest, ::testing::Values( make_int2(2,4) ));
INSTANTIATE_TEST_CASE_P(xpay_double, BlasTest, ::testing::Values( make_int2(2,5) ));
INSTANTIATE_TEST_CASE_P(mxpy_double, BlasTest, ::testing::Values( make_int2(2,6) ));
INSTANTIATE_TEST_CASE_P(ax_double, BlasTest, ::testing::Values( make_int2(2,7) ));
INSTANTIATE_TEST_CASE_P(caxpy_double, BlasTest, ::testing::Values( make_int2(2,8) ));
INSTANTIATE_TEST_CASE_P(caxpby_double, BlasTest, ::testing::Values( make_int2(2,9) ));
INSTANTIATE_TEST_CASE_P(cxpaypbz_double, BlasTest, ::testing::Values( make_int2(2,10) ));
INSTANTIATE_TEST_CASE_P(axpyBzpcx_double, BlasTest, ::testing::Values( make_int2(2,11) ));
INSTANTIATE_TEST_CASE_P(axpyZpbx_double, BlasTest, ::testing::Values( make_int2(2,12) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbw_double, BlasTest, ::testing::Values( make_int2(2,13) ));
INSTANTIATE_TEST_CASE_P(cabxpyAx_double, BlasTest, ::testing::Values( make_int2(2,14) ));
INSTANTIATE_TEST_CASE_P(caxpbypz_double, BlasTest, ::testing::Values( make_int2(2,15) ));
INSTANTIATE_TEST_CASE_P(caxpbypczpw_double, BlasTest, ::testing::Values( make_int2(2,16) ));
INSTANTIATE_TEST_CASE_P(caxpyXmaz_double, BlasTest, ::testing::Values( make_int2(2,17) ));
INSTANTIATE_TEST_CASE_P(norm2_double, BlasTest, ::testing::Values( make_int2(2,18) ));
INSTANTIATE_TEST_CASE_P(reDotProduct_double, BlasTest, ::testing::Values( make_int2(2,19) ));
INSTANTIATE_TEST_CASE_P(axpyNorm_double, BlasTest, ::testing::Values( make_int2(2,20) ));
INSTANTIATE_TEST_CASE_P(xmyNorm_double, BlasTest, ::testing::Values( make_int2(2,21) ));
INSTANTIATE_TEST_CASE_P(caxpyNorm_double, BlasTest, ::testing::Values( make_int2(2,22) ));
INSTANTIATE_TEST_CASE_P(caxpyXmazNormX_double, BlasTest, ::testing::Values( make_int2(2,23) ));
INSTANTIATE_TEST_CASE_P(cabxpyAxNorm_double, BlasTest, ::testing::Values( make_int2(2,24) ));
INSTANTIATE_TEST_CASE_P(cDotProduct_double, BlasTest, ::testing::Values( make_int2(2,25) ));
INSTANTIATE_TEST_CASE_P(xpaycDotzy_double, BlasTest, ::testing::Values( make_int2(2,26) ));
INSTANTIATE_TEST_CASE_P(caxpyDotzy_double, BlasTest, ::testing::Values( make_int2(2,27) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormA_double, BlasTest, ::testing::Values( make_int2(2,28) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormB_double, BlasTest, ::testing::Values( make_int2(2,29) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbwcDotProductWYNormY_double, BlasTest, ::testing::Values( make_int2(2,30) ));
INSTANTIATE_TEST_CASE_P(HeavyQuarkResidualNorm_double, BlasTest, ::testing::Values( make_int2(2,31) ));

