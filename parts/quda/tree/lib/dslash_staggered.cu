#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

//these are access control for staggered action
#ifdef GPU_STAGGERED_DIRAC
#if (__COMPUTE_CAPABILITY__ >= 300) // Kepler works best with texture loads only
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#else // Fermi
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#endif

#endif // GPU_STAGGERED_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

namespace quda {

  namespace staggered {
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

#undef GPU_CLOVER_DIRAC
#undef GPU_DOMAIN_WALL_DIRAC
#define DD_IMPROVED 0
#include <staggered_dslash_def.h> // staggered Dslash kernels
#undef DD_IMPROVED

#include <dslash_quda.cuh>
  } // end namespace staggered

  // declare the dslash events
#include <dslash_events.cuh>

  using namespace staggered;

  template<typename T> struct RealType {};
  template<> struct RealType<double2> { typedef double type; };
  template<> struct RealType<float2> { typedef float type; };
  template<> struct RealType<float4> { typedef float type; };
  template<> struct RealType<short2> { typedef short type; };
  template<> struct RealType<short4> { typedef short type; };

#ifdef GPU_STAGGERED_DIRAC
  template <typename sFloat, typename gFloat>
  class StaggeredDslashCuda : public DslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const double a;

  protected:
    unsigned int sharedBytesPerThread() const
    {
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return 6 * reg_size;
    }

  public:
    StaggeredDslashCuda(cudaColorSpinorField *out, const gFloat *gauge0, const gFloat *gauge1,
			const QudaReconstructType reconstruct, const cudaColorSpinorField *in,
			const cudaColorSpinorField *x, const double a, const int dagger)
      : DslashCuda(out, in, x, reconstruct, dagger), gauge0(gauge0), gauge1(gauge1), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x);
    }

    virtual ~StaggeredDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
      STAGGERED_DSLASH(gridDim, tp.block, tp.shared_bytes, stream, dslashParam,
		       (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
		       (sFloat*)in->V(), (float*)in->Norm(), 
		       (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a); 
    }

    int Nface() { return 2; } 
  };
#endif // GPU_STAGGERED_DIRAC

#include <dslash_policy.cuh>

  void staggeredDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
			   const cudaColorSpinorField *in, const int parity, 
			   const int dagger, const cudaColorSpinorField *x,
			   const double &k, const int *commOverride, TimeProfile &profile, const QudaDslashPolicy &dslashPolicy)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_STAGGERED_DIRAC

    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code

    dslashParam.parity = parity;
    dslashParam.gauge_stride = gauge.Stride();
    dslashParam.fat_link_max = gauge.LinkMax(); // May need to use this in the preconditioning step 
    // in the solver for the improved staggered action

    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }
    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision()) {
      errorQuda("Mixing precisions gauge=%d and spinor=%d not supported",
		gauge.Precision(), in->Precision());
    }

    if (gauge.Reconstruct() == QUDA_RECONSTRUCT_9 || gauge.Reconstruct() == QUDA_RECONSTRUCT_13) {
      errorQuda("Reconstruct %d not supported", gauge.Reconstruct());
    }

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new StaggeredDslashCuda<double2, double2>
	(out, (double2*)gauge0, (double2*)gauge1, gauge.Reconstruct(), in, x, k, dagger);
      regSize = sizeof(double);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new StaggeredDslashCuda<float2, float2>
	(out, (float2*)gauge0, (float2*)gauge1, gauge.Reconstruct(), in, x, k, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {	
      dslash = new StaggeredDslashCuda<short2, short2>
	(out, (short2*)gauge0, (short2*)gauge1, gauge.Reconstruct(), in, x, k, dagger);
    }

#ifndef GPU_COMMS
    DslashPolicyImp* dslashImp = DslashFactory::create(dslashPolicy);
#else
    DslashPolicyImp* dslashImp = DslashFactory::create(QUDA_GPU_COMMS_DSLASH);
#endif

    (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), regSize, parity, dagger, in->Volume(), in->GhostFace(), profile);
    delete dslashImp;

    delete dslash;
    unbindGaugeTex(gauge);

    checkCudaError();

#else
    errorQuda("Staggered dslash has not been built");
#endif  // GPU_STAGGERED_DIRAC
  }

}
