#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC


#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

namespace quda {

  namespace ndegtwisted {

#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

    // Enable shared memory dslash for Fermi architecture
    //#define SHARED_WILSON_DSLASH
    //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#ifdef GPU_NDEG_TWISTED_MASS_DIRAC
#include <tm_ndeg_dslash_def.h>   // Non-degenerate twisted Mass
#endif

#ifndef NDEGTM_SHARED_FLOATS_PER_THREAD
#define NDEGTM_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>

  } // end namespace twisted
  
  // declare the dslash events
#include <dslash_events.cuh>

  using namespace ndegtwisted;

#ifdef GPU_NDEG_TWISTED_MASS_DIRAC
  template <typename sFloat, typename gFloat>
  class NdegTwistedDslashCuda : public SharedDslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const QudaTwistDslashType dslashType;
    double a, b, c, d;

  protected:
    unsigned int sharedBytesPerThread() const
    {
      if (dslashParam.kernel_type == INTERIOR_KERNEL) {
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return NDEGTM_SHARED_FLOATS_PER_THREAD * reg_size;
      } else {
	return 0;
      }
    }

  public:
    NdegTwistedDslashCuda(cudaColorSpinorField *out, const gFloat *gauge0, const gFloat *gauge1, 
		      const QudaReconstructType reconstruct, const cudaColorSpinorField *in,  const cudaColorSpinorField *x, 
		      const QudaTwistDslashType dslashType, const double kappa, const double mu, 
		      const double epsilon, const double k, const int dagger)
      : SharedDslashCuda(out, in, x, reconstruct, dagger), gauge0(gauge0), gauge1(gauge1), dslashType(dslashType)
    { 
      bindSpinorTex<sFloat>(in, out, x); 
      a = kappa;
      b = mu;
      c = epsilon;
      d = k;
      if (dslashType != QUDA_NONDEG_DSLASH) errorQuda("Invalid dslashType for non-degenerate twisted-mass Dslash");
      dslashParam.fl_stride = in->VolumeCB()/2;
    }
    virtual ~NdegTwistedDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      strcat(key.aux,",NdegDslash");
      return key;
    }

    void apply(const hipStream_t &stream)
    {

#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
	errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      NDEG_TM_DSLASH(twistedNdegMassDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
		     (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
		     (sFloat*)in->V(), (float*)in->Norm(), a, b, c, d, (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0));
    }

    long long flops() const {
      int twisted_flops = 48;
      long long flops = DslashCuda::flops();
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
      case EXTERIOR_KERNEL_ALL:
	break;
      case INTERIOR_KERNEL:
	// twisted-mass flops are done in the interior kernel
	flops += twisted_flops * in->VolumeCB();	  
	break;
      }
      return flops;
    }
  };
#endif // GPU_NDEG_TWISTED_MASS_DIRAC


#include <dslash_policy.cuh> 

  void ndegTwistedMassDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
				 const cudaColorSpinorField *in, const int parity, const int dagger, 
				 const cudaColorSpinorField *x, const QudaTwistDslashType type, 
				 const double &kappa, const double &mu, const double &epsilon, 
				 const double &k,  const int *commOverride, TimeProfile &profile, 
				 const QudaDslashPolicy &dslashPolicy)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL
#ifdef GPU_NDEG_TWISTED_MASS_DIRAC
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code

    int ghost_threads[4] = {0};
    int bulk_threads = in->Volume() / 2;

    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
      ghost_threads[i] = in->GhostFace()[i] / 2;
    }

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge and spinor precision not supported");

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new NdegTwistedDslashCuda<double2,double2>(out, (double2*)gauge0,(double2*)gauge1, gauge.Reconstruct(), in, x, type, kappa, mu, epsilon, k, dagger);
      regSize = sizeof(double);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new NdegTwistedDslashCuda<float4,float4>(out, (float4*)gauge0,(float4*)gauge1, gauge.Reconstruct(), in, x, type, kappa, mu, epsilon, k, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new NdegTwistedDslashCuda<short4,short4>(out, (short4*)gauge0,(short4*)gauge1, gauge.Reconstruct(), in, x, type, kappa, mu, epsilon, k, dagger);
    }

#ifndef GPU_COMMS
    DslashPolicyImp* dslashImp = DslashFactory::create(dslashPolicy);
#else
    DslashPolicyImp* dslashImp = DslashFactory::create(QUDA_GPU_COMMS_DSLASH);
#endif
    (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), regSize, parity, dagger, bulk_threads, ghost_threads, profile);
    delete dslashImp;

    delete dslash;

    unbindGaugeTex(gauge);

    checkCudaError();
#else
    errorQuda("Non-degenerate twisted mass dslash has not been built");
#endif
  }

}
