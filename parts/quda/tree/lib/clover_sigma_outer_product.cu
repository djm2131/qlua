#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <tune_quda.h>
#include <quda_internal.h>
#include <gauge_field_order.h>
#include <quda_matrix.h>
#include <color_spinor.h>
#include <dslash_quda.h>

namespace quda {

#ifdef GPU_CLOVER_DIRAC

  namespace { // anonymous
#include <texture.h>
  }
  
  template<typename Complex, typename Output, typename InputA, typename InputB>
  struct CloverSigmaOprodArg {
    unsigned int length;
    unsigned int parity;
    InputA inA;
    InputB inB;
    Output oprod;
    typename RealTypeId<Complex>::Type coeff;
    int mu;
    int nu;
    int count;
      
    CloverSigmaOprodArg(const unsigned int parity,
			const double coeff,
			int mu,
			int nu,
			int count,
			InputA& inA,
			InputB& inB,
			Output& oprod,
			GaugeField &meta) : length(meta.VolumeCB()), parity(parity), 
					    inA(inA), inB(inB), oprod(oprod), 
					    coeff(coeff), mu(mu), nu(nu), count(count)
    {

    }
  };

  template<typename Complex, typename Output, typename InputA, typename InputB>
  __global__ void sigmaOprodKernel(CloverSigmaOprodArg<Complex, Output, InputA, InputB> arg) {
    typedef typename RealTypeId<Complex>::Type real;
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    ColorSpinor<real,3,4> A, B;
    Matrix<Complex,3> result, temp;

    // workaround for code that hangs generated with CUDA 5.x
#if (CUDA_VERSION < 6000)
    if (idx >= arg.length) idx = arg.length - 1;
#else
    while(idx<arg.length){
#endif // CUDA_VERSION
      arg.inA.load(static_cast<Complex*>(A.data), idx);
      arg.inB.load(static_cast<Complex*>(B.data), idx);

      // multiply by sigma_mu_nu
      ColorSpinor<real,3,4> C = A.sigma(arg.mu,arg.nu);
      result = outerProdSpinTrace(C,B);

      if (arg.count > 0) {
	arg.oprod.load(reinterpret_cast<real*>(temp.data), idx, 0, arg.parity); 
	temp = arg.coeff*result + temp;
      } else {
	temp = arg.coeff*result;
      }
      arg.oprod.save(reinterpret_cast<real*>(temp.data), idx, 0, arg.parity); 
#if (CUDA_VERSION >= 6000)
      idx += gridDim.x*blockDim.x;
    }
#endif // CUDA_VERSION
    return;
  } // sigmaOprodKernel

  
  template<typename Complex, typename Output, typename InputA, typename InputB> 
  class CloverSigmaOprod : public Tunable {
    
  private:
    CloverSigmaOprodArg<Complex,Output,InputA,InputB> &arg;
    const GaugeField &meta;
    QudaFieldLocation location; // location of the lattice fields
    
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }
    
    unsigned int minThreads() const { return arg.length; }
    bool tuneGridDim() const { return false; }
    
  public:
    CloverSigmaOprod(CloverSigmaOprodArg<Complex,Output,InputA,InputB> &arg,
		     const GaugeField &meta, QudaFieldLocation location)
      : arg(arg), meta(meta), location(location) {
      writeAuxString("prec=%lu,stride=%d,mu=%d,nu=%d", 
		     sizeof(Complex)/2, arg.inA.Stride(), arg.mu, arg.nu);
      // this sets the communications pattern for the packing kernel
    } 
    
    virtual ~CloverSigmaOprod() {}
    
    void apply(const hipStream_t &stream){
      if(location == QUDA_CUDA_FIELD_LOCATION){
	// Disable tuning for the time being
	TuneParam tp = tuneLaunch(*this,getTuning(),getVerbosity());
	sigmaOprodKernel<<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }else{ // run the CPU code
	errorQuda("No CPU support for staggered outer-product calculation\n");
      }
    } // apply
    
    void preTune(){
      this->arg.oprod.save();
    }
    void postTune(){
      this->arg.oprod.load();
    }
  
    long long flops() const { 
      ((long long)arg.length)*(0 + 144 + 36); // spin_mu_nu + spin trace + multiply-add
    }
    long long bytes() const { 
      ((long long)arg.length)*(arg.inA.Bytes() + arg.inB.Bytes() + 2*arg.oprod.Bytes());
    }
  
    TuneKey tuneKey() const { 
      return TuneKey(meta.VolString(), typeid(*this).name(), aux);
    }
  }; // CloverSigmaOprod
  
  template<typename Complex, typename Output, typename InputA, typename InputB>
  void computeCloverSigmaOprodCuda(Output oprod, cudaGaugeField& out, InputA& inA, InputB& inB,
				   const unsigned int parity, const double coeff, int mu, int nu, int shift) {
    // Create the arguments 
    CloverSigmaOprodArg<Complex,Output,InputA,InputB> arg(parity, coeff, mu, nu, shift, inA, inB, oprod, out);
    CloverSigmaOprod<Complex,Output,InputA,InputB> sigma_oprod(arg, out, QUDA_CUDA_FIELD_LOCATION);
    sigma_oprod.apply(0);
  } // computeCloverSigmaOprodCuda
  
#endif // GPU_CLOVER_FORCE

  void computeCloverSigmaOprod(cudaGaugeField& oprod,
			       cudaColorSpinorField& x,  
			       cudaColorSpinorField& p,
			       const double coeff, int mu, int nu, int shift)
  {

#ifdef GPU_CLOVER_DIRAC
    if(oprod.Order() != QUDA_FLOAT2_GAUGE_ORDER)
      errorQuda("Unsupported output ordering: %d\n", oprod.Order());    

    if(x.Precision() != oprod.Precision()) errorQuda("Mixed precision not supported: %d %d\n", x.Precision(), oprod.Precision());

    for (int parity=0; parity<2; parity++) {
      cudaColorSpinorField& inA = (parity&1) ? x.Odd() : x.Even();
      cudaColorSpinorField& inB = (parity&1) ? p.Odd() : p.Even();

      if(x.Precision() == QUDA_DOUBLE_PRECISION){
	Spinor<double2, double2, double2, 12, 0, 0> spinorA(inA);
	Spinor<double2, double2, double2, 12, 0, 1> spinorB(inB);
	computeCloverSigmaOprodCuda<double2>(FloatNOrder<double, 18, 2, 18>(oprod), 
					     oprod, spinorA, spinorB, parity, coeff, mu, nu, shift);
      } else {
	errorQuda("Unsupported precision: %d\n", x.Precision());
      }
    } // parity

#else // GPU_CLOVER_DIRAC not defined
    errorQuda("Clover Dirac operator has not been built!"); 
#endif

    checkCudaError();
    return;
  } // computeCloverForce  

} // namespace quda
