#include "hip/hip_runtime.h"
#include <gauge_field_order.h>

namespace quda {
  template <typename Order, int nDim>
  struct ExtractGhostArg {
    Order order;
    const unsigned char nFace;
    unsigned short X[nDim];
    unsigned short A[nDim];
    unsigned short B[nDim];
    unsigned short C[nDim];
    int f[nDim][nDim];
    bool localParity[nDim];
    ExtractGhostArg(const Order &order, int nFace, const int *X_, const int *A_,
		    const int *B_, const int *C_, const int f_[nDim][nDim], const int *localParity_) 
  : order(order), nFace(nFace) { 
      for (int d=0; d<nDim; d++) {
	X[d] = X_[d];
	A[d] = A_[d];
	B[d] = B_[d];
	C[d] = C_[d];
	for (int e=0; e<nDim; e++) f[d][e] = f_[d][e];
	localParity[d] = localParity_[d]; 
      }
    }
  };

  /**
     Generic CPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
  */
  template <typename Float, int length, int nDim, typename Order>
  void extractGhost(ExtractGhostArg<Order,nDim> arg) {  
    typedef typename mapper<Float>::type RegType;

    for (int parity=0; parity<2; parity++) {

      for (int dim=0; dim<nDim; dim++) {

	// linear index used for writing into ghost buffer
	int indexDst = 0;
	// the following 4-way loop means this is specialized for 4 dimensions 

	// FIXME redefine a, b, c, d such that we always optimize for locality
	for (int d=arg.X[dim]-arg.nFace; d<arg.X[dim]; d++) { // loop over last nFace faces in this dimension
	  for (int a=0; a<arg.A[dim]; a++) { // loop over the surface elements of this face
	    for (int b=0; b<arg.B[dim]; b++) { // loop over the surface elements of this face
	      for (int c=0; c<arg.C[dim]; c++) { // loop over the surface elements of this face
		// index is a checkboarded spacetime coordinate
		int indexCB = (a*arg.f[dim][0] + b*arg.f[dim][1] + c*arg.f[dim][2] + d*arg.f[dim][3]) >> 1;
		// we only do the extraction for parity we are currently working on
		int oddness = (a+b+c+d) & 1;
		if (oddness == parity) {
		  RegType u[length];
		  arg.order.load(u, indexCB, dim, parity); // load the ghost element from the bulk
		  arg.order.saveGhost(u, indexDst, dim, (parity+arg.localParity[dim])&1);
		  indexDst++;
		} // oddness == parity
	      } // c
	    } // b
	  } // a
	} // d

	//assert(indexDst == arg.nFace*arg.surfaceCB[dim]);
	assert(indexDst == arg.order.faceVolumeCB[dim]);
      } // dim

    } // parity

  }

  /**
     Generic GPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
     FIXME this implementation will have two-way warp divergence
  */
  template <typename Float, int length, int nDim, typename Order>
  __global__ void extractGhostKernel(ExtractGhostArg<Order,nDim> arg) {  
    typedef typename mapper<Float>::type RegType;

    for (int parity=0; parity<2; parity++) {
      for (int dim=0; dim<nDim; dim++) {

	// linear index used for writing into ghost buffer
	int X = blockIdx.x * blockDim.x + threadIdx.x; 	
	//if (X >= 2*arg.nFace*arg.surfaceCB[dim]) continue;
	if (X >= 2*arg.order.faceVolumeCB[dim]) continue;
	// X = ((d * A + a)*B + b)*C + c
	int dab = X/arg.C[dim];
	int c = X - dab*arg.C[dim];
	int da = dab/arg.B[dim];
	int b = dab - da*arg.B[dim];
	int d = da / arg.A[dim];
	int a = da - d * arg.A[dim];
	d += arg.X[dim]-arg.nFace;

	// index is a checkboarded spacetime coordinate
	int indexCB = (a*arg.f[dim][0] + b*arg.f[dim][1] + c*arg.f[dim][2] + d*arg.f[dim][3]) >> 1;
	// we only do the extraction for parity we are currently working on
	int oddness = (a+b+c+d)&1;
	if (oddness == parity) {
	  RegType u[length];
	  arg.order.load(u, indexCB, dim, parity); // load the ghost element from the bulk
	  arg.order.saveGhost(u, X>>1, dim, (parity+arg.localParity[dim])&1);
	} // oddness == parity

      } // dim

    } // parity

  }

  template <typename Float, int length, int nDim, typename Order>
  class ExtractGhost : Tunable {
    ExtractGhostArg<Order,nDim> arg;
    int size;
    const GaugeField &meta;

  private:
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0 ;}

    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return size; }

  public:
    ExtractGhost(ExtractGhostArg<Order,nDim> &arg, const GaugeField &meta) : arg(arg), meta(meta) { 
      int faceMax = 0;
      for (int d=0; d<nDim; d++) 
	faceMax = (arg.order.faceVolumeCB[d] > faceMax ) 
	  ? arg.order.faceVolumeCB[d] : faceMax;
      size = 2 * faceMax; // factor of comes from parity

      writeAuxString("stride=%d", arg.order.stride);
    }

    virtual ~ExtractGhost() { ; }
  
    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      extractGhostKernel<Float, length, nDim, Order> 
	<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    std::string paramString(const TuneParam &param) const { // Don't bother printing the grid dim.
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }

    long long flops() const { return 0; } 
    long long bytes() const { 
      int sites = 0;
      for (int d=0; d<nDim; d++) sites += arg.order.faceVolumeCB[d];
      return 2 * sites * 2 * arg.order.Bytes(); // parity * sites * i/o * vec size
    } 
  };


  /**
     Generic CPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
  */
  template <typename Float, int length, typename Order>
  void extractGhost(Order order, const GaugeField &u, QudaFieldLocation location) {  
    const int *X = u.X();
    const int nFace = u.Nface();
    const int nDim = 4;
    //loop variables: a, b, c with a the most signifcant and c the least significant
    //A, B, C the maximum value
    //we need to loop in d as well, d's vlaue dims[dir]-3, dims[dir]-2, dims[dir]-1
    int A[nDim], B[nDim], C[nDim];
    A[0] = X[3]; B[0] = X[2]; C[0] = X[1]; // X dimension face
    A[1] = X[3]; B[1] = X[2]; C[1] = X[0]; // Y dimension face
    A[2] = X[3]; B[2] = X[1]; C[2] = X[0]; // Z dimension face
    A[3] = X[2]; B[3] = X[1]; C[3] = X[0]; // T dimension face    

    //multiplication factor to compute index in original cpu memory
    int f[nDim][nDim]={
      {X[0]*X[1]*X[2],  X[0]*X[1], X[0],               1},
      {X[0]*X[1]*X[2],  X[0]*X[1],    1,            X[0]},
      {X[0]*X[1]*X[2],       X[0],    1,       X[0]*X[1]},
      {     X[0]*X[1],       X[0],    1,  X[0]*X[1]*X[2]}
    };

    //set the local processor parity 
    //switching odd and even ghost gauge when that dimension size is odd
    //only switch if X[dir] is odd and the gridsize in that dimension is greater than 1
    // FIXME - I don't understand this, shouldn't it be commDim(dim) == 0 ?
    int localParity[nDim];
    for (int dim=0; dim<nDim; dim++) 
      //localParity[dim] = (X[dim]%2==0 || commDim(dim)) ? 0 : 1;
      localParity[dim] = ((X[dim] % 2 ==1) && (commDim(dim) > 1)) ? 1 : 0;

    ExtractGhostArg<Order, nDim> arg(order, nFace, X, A, B, C, f, localParity);
    if (location==QUDA_CPU_FIELD_LOCATION) {
      extractGhost<Float,length,nDim,Order>(arg);
    } else {
      ExtractGhost<Float,length,nDim,Order> extract(arg, u);
      extract.apply(0);
    }

  }

  /** This is the template driver for extractGhost */
  template <typename Float>
    void extractGhost(const GaugeField &u, Float **Ghost) {

    const int length = 18;

    QudaFieldLocation location = 
      (typeid(u)==typeid(cudaGaugeField)) ? QUDA_CUDA_FIELD_LOCATION : QUDA_CPU_FIELD_LOCATION;

    if (u.isNative()) {
      if (u.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	if (typeid(Float)==typeid(short) && u.LinkType() == QUDA_ASQTAD_FAT_LINKS) {
	  extractGhost<short,length>(FloatNOrder<short,length,2,19>
				     (u, 0, (short**)Ghost), u, location);
	} else {
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type G;
	  extractGhost<Float,length>(G(u, 0, Ghost), u, location);
	}
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_12) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type G;
	extractGhost<Float,length>(G(u, 0, Ghost), u, location);
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_8) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type G;
	extractGhost<Float,length>(G(u, 0, Ghost), u, location);
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_13) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_13>::type G;
	extractGhost<Float,length>(G(u, 0, Ghost), u, location);
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_9) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_9>::type G;
	extractGhost<Float,length>(G(u, 0, Ghost), u, location);
      }
    } else if (u.Order() == QUDA_QDP_GAUGE_ORDER) {
      
#ifdef BUILD_QDP_INTERFACE
      extractGhost<Float,length>(QDPOrder<Float,length>(u, 0, Ghost), u, location);
#else
      errorQuda("QDP interface has not been built\n");
#endif
      
    } else if (u.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      extractGhost<Float,length>(QDPJITOrder<Float,length>(u, 0, Ghost), u, location);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      extractGhost<Float,length>(CPSOrder<Float,length>(u, 0, Ghost), u, location);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      extractGhost<Float,length>(MILCOrder<Float,length>(u, 0, Ghost), u, location);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      extractGhost<Float,length>(BQCDOrder<Float,length>(u, 0, Ghost), u, location);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      extractGhost<Float,length>(TIFROrder<Float,length>(u, 0, Ghost), u, location);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", u.Order());
    }

  }

  void extractGaugeGhost(const GaugeField &u, void **ghost) {

    if (u.Precision() == QUDA_DOUBLE_PRECISION) {
      extractGhost(u, (double**)ghost);
    } else if (u.Precision() == QUDA_SINGLE_PRECISION) {
      extractGhost(u, (float**)ghost);
    } else if (u.Precision() == QUDA_HALF_PRECISION) {
      extractGhost(u, (short**)ghost);      
    } else {
      errorQuda("Unknown precision type %d", u.Precision());
    }
  }

} // namespace quda
