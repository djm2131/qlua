#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>

// For kernels with precision conversion built in
#define checkSpinorLength(a, b)						\
  {									\
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %d %d", a.Length(), b.Length());	\
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

namespace quda {

  hipStream_t* getBlasStream();
    
  namespace copy {

#include <texture.h>

    static struct {
      const char *vol_str;
      const char *aux_str;      
    } blasStrings;

    template <typename FloatN, int N, typename Output, typename Input>
    __global__ void copyKernel(Output Y, Input X, int length) {
      unsigned int i = blockIdx.x*(blockDim.x) + threadIdx.x;
      unsigned int gridSize = gridDim.x*blockDim.x;

      while (i < length) {
	FloatN x[N];
	X.load(x, i);
	Y.save(x, i);
	i += gridSize;
      }
    }

    template <typename FloatN, int N, typename Output, typename Input>
    class CopyCuda : public Tunable {

    private:
      Input &X;
      Output &Y;
      const int length;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      virtual bool advanceSharedBytes(TuneParam &param) const
      {
	TuneParam next(param);
	advanceBlockDim(next); // to get next blockDim
	int nthreads = next.block.x * next.block.y * next.block.z;
	param.shared_bytes = sharedBytesPerThread()*nthreads > sharedBytesPerBlock(param) ?
	  sharedBytesPerThread()*nthreads : sharedBytesPerBlock(param);
	return false;
      }

    public:
      CopyCuda(Output &Y, Input &X, int length) : X(X), Y(Y), length(length) { }
      virtual ~CopyCuda() { ; }

      inline TuneKey tuneKey() const {
	return TuneKey(blasStrings.vol_str, "copyKernel", blasStrings.aux_str); 
      }

      inline void apply(const hipStream_t &stream) {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	copyKernel<FloatN, N><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(Y, X, length);
      }

      void preTune() { ; } // no need to save state for copy kernels
      void postTune() { ; } // no need to restore state for copy kernels

      long long flops() const { return 0; }
      long long bytes() const { 
	const int Ninternal = (sizeof(FloatN)/sizeof(((FloatN*)0)->x))*N;
	size_t bytes = (X.Precision() + Y.Precision())*Ninternal;
	if (X.Precision() == QUDA_HALF_PRECISION) bytes += sizeof(float);
	if (Y.Precision() == QUDA_HALF_PRECISION) bytes += sizeof(float);
	return bytes*length; 
      }
      int tuningIter() const { return 3; }
    };

    void copyCuda(cudaColorSpinorField &dst, const cudaColorSpinorField &src) {
      if (&src == &dst) return; // aliasing fields
      if (src.Nspin() != 1 && src.Nspin() != 4) errorQuda("nSpin(%d) not supported\n", src.Nspin());

      if (dst.SiteSubset() == QUDA_FULL_SITE_SUBSET || src.SiteSubset() == QUDA_FULL_SITE_SUBSET) {
	if (src.SiteSubset() != dst.SiteSubset()) 
	  errorQuda("Spinor fields do not have matching subsets dst=%d src=%d\n", 
		    dst.SiteSubset(), src.SiteSubset());
	copy::copyCuda(dst.Even(), src.Even());
	copy::copyCuda(dst.Odd(), src.Odd());
	return;
      }

      checkSpinorLength(dst, src);

      blasStrings.vol_str = src.VolString();
      char tmp[256];
      strcpy(tmp, "dst=");
      strcat(tmp, dst.AuxString());
      strcat(tmp, ",src=");
      strcat(tmp, src.AuxString());
      blasStrings.aux_str = tmp;

      // For a given dst precision, there are two non-trivial possibilities for the
      // src precision.

      // FIXME: use traits to encapsulate register type for shorts -
      // will reduce template type parameters from 3 to 2

      blas_bytes += (unsigned long long)src.RealLength()*(src.Precision() + dst.Precision());
      
      if (dst.Precision() == src.Precision()) {
	if (src.Bytes() != dst.Bytes()) errorQuda("Precisions match, but bytes do not");
	hipMemcpy(dst.V(), src.V(), dst.Bytes(), hipMemcpyDeviceToDevice);
	if (dst.Precision() == QUDA_HALF_PRECISION) {
	  hipMemcpy(dst.Norm(), src.Norm(), dst.NormBytes(), hipMemcpyDeviceToDevice);
	  blas_bytes += 2*(unsigned long long)dst.RealLength()*sizeof(float);
	}
      } else if (dst.Precision() == QUDA_DOUBLE_PRECISION && src.Precision() == QUDA_SINGLE_PRECISION) {
	if (src.Nspin() == 4){
	  Spinor<float4, float4, float4, 6, 0, 0> src_tex(src);
	  Spinor<float4, float2, double2, 6, 1> dst_spinor(dst);
	  CopyCuda<float4, 6, Spinor<float4, float2, double2, 6, 1>, 
		   Spinor<float4, float4, float4, 6, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //src.Nspin() == 1
	  Spinor<float2, float2, float2, 3, 0, 0> src_tex(src);
	  Spinor<float2, float2, double2, 3, 1> dst_spinor(dst);
	  CopyCuda<float2, 3, Spinor<float2, float2, double2, 3, 1>,
		   Spinor<float2, float2, float2, 3, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
    } 
  } else if (dst.Precision() == QUDA_SINGLE_PRECISION && src.Precision() == QUDA_DOUBLE_PRECISION) {
	if (src.Nspin() == 4){
	  Spinor<float4, float2, double2, 6, 0, 0> src_tex(src);
	  Spinor<float4, float4, float4, 6, 1> dst_spinor(dst);
	  CopyCuda<float4, 6, Spinor<float4, float4, float4, 6, 1>,
		   Spinor<float4, float2, double2, 6, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //src.Nspin() ==1
	  Spinor<float2, float2, double2, 3, 0, 0> src_tex(src);
	  Spinor<float2, float2, float2, 3, 1> dst_spinor(dst);
	  CopyCuda<float2, 3, Spinor<float2, float2, float2, 3, 1>,
		   Spinor<float2, float2, double2, 3, 0, 0> >
	  copy(dst_spinor, src_tex, src.Volume());
  copy.apply(*getBlasStream());	
}
  } else if (dst.Precision() == QUDA_SINGLE_PRECISION && src.Precision() == QUDA_HALF_PRECISION) {
	blas_bytes += (unsigned long long)src.Volume()*sizeof(float);
	if (src.Nspin() == 4){      
	  Spinor<float4, float4, short4, 6, 0, 0> src_tex(src);
	  Spinor<float4, float4, float4, 6, 1> dst_spinor(dst);
	  CopyCuda<float4, 6, Spinor<float4, float4, float4, 6, 1>,
		   Spinor<float4, float4, short4, 6, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //nSpin== 1;
	  Spinor<float2, float2, short2, 3, 0, 0> src_tex(src);
	  Spinor<float2, float2, float2, 3, 1> dst_spinor(dst);
	  CopyCuda<float2, 3, Spinor<float2, float2, float2, 3, 1>,
		   Spinor<float2, float2, short2, 3, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
    }
  } else if (dst.Precision() == QUDA_HALF_PRECISION && src.Precision() == QUDA_SINGLE_PRECISION) {
	blas_bytes += (unsigned long long)dst.Volume()*sizeof(float);
	if (src.Nspin() == 4){
	  Spinor<float4, float4, float4, 6, 0, 0> src_tex(src);
	  Spinor<float4, float4, short4, 6, 1> dst_spinor(dst);
	  CopyCuda<float4, 6, Spinor<float4, float4, short4, 6, 1>,
		   Spinor<float4, float4, float4, 6, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //nSpin == 1
	  Spinor<float2, float2, float2, 3, 0, 0> src_tex(src);
	  Spinor<float2, float2, short2, 3, 1> dst_spinor(dst);
	  CopyCuda<float2, 3, Spinor<float2, float2, short2, 3, 1>,
		   Spinor<float2, float2, float2, 3, 0, 0> >
	  copy(dst_spinor, src_tex, src.Volume());
  copy.apply(*getBlasStream());	
}
  } else if (dst.Precision() == QUDA_DOUBLE_PRECISION && src.Precision() == QUDA_HALF_PRECISION) {
	blas_bytes += (unsigned long long)src.Volume()*sizeof(float);
	if (src.Nspin() == 4){
	  Spinor<double2, float4, short4, 12, 0, 0> src_tex(src);
	  Spinor<double2, double2, double2, 12, 1> dst_spinor(dst);
	  CopyCuda<double2, 12, Spinor<double2, double2, double2, 12, 1>,
		   Spinor<double2, float4, short4, 12, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //nSpin == 1
	  Spinor<double2, float2, short2, 3, 0, 0> src_tex(src);
	  Spinor<double2, double2, double2, 3, 1> dst_spinor(dst);
	  CopyCuda<double2, 3, Spinor<double2, double2, double2, 3, 1>,
		   Spinor<double2, float2, short2, 3, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
    }
  } else if (dst.Precision() == QUDA_HALF_PRECISION && src.Precision() == QUDA_DOUBLE_PRECISION) {
	blas_bytes += (unsigned long long)dst.Volume()*sizeof(float);
	if (src.Nspin() == 4){
	  Spinor<double2, double2, double2, 12, 0, 0> src_tex(src);
	  Spinor<double2, double4, short4, 12, 1> dst_spinor(dst);
	  CopyCuda<double2, 12, Spinor<double2, double4, short4, 12, 1>,
		   Spinor<double2, double2, double2, 12, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //nSpin == 1
	  Spinor<double2, double2, double2, 3, 0, 0> src_tex(src);
	  Spinor<double2, double2, short2, 3, 1> dst_spinor(dst);
	  CopyCuda<double2, 3, Spinor<double2, double2, short2, 3, 1>,
		   Spinor<double2, double2, double2, 3, 0, 0> >
	  copy(dst_spinor, src_tex, src.Volume());
  copy.apply(*getBlasStream());	
}
  } else {
	errorQuda("Invalid precision combination dst=%d and src=%d", dst.Precision(), src.Precision());
      }
      
      checkCudaError();
    }

  } // namespace copy

  void copyCuda(cudaColorSpinorField &dst, const cudaColorSpinorField &src) {
    copy::copyCuda(dst, src);
  }
  
} // namespace quda
