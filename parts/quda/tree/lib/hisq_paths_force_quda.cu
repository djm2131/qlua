#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <lattice_field.h>
#include <read_gauge.h>
#include <gauge_field.h>
#include <ks_improved_force.h>
#include <hw_quda.h>
#include <hisq_force_macros.h>
#include <utility>
#include <quda_matrix.h>
#include <force_common.h>
#include <tune_quda.h>
#include <color_spinor_field.h>
#include <face_quda.h>
#include <index_helper.cuh>

#ifdef GPU_HISQ_FORCE

//DEBUG : control compile 
#define COMPILE_HISQ_DP_18 
#define COMPILE_HISQ_DP_12 
#define COMPILE_HISQ_SP_18 
#define COMPILE_HISQ_SP_12

// Disable texture read for now. Need to revisit this.
#define HISQ_SITE_MATRIX_LOAD_TEX 1
#define HISQ_NEW_OPROD_LOAD_TEX 1

#ifdef USE_TEXTURE_OBJECTS
#define TEX1DFETCH(type, tex, idx) tex1Dfetch<type>((tex), idx)
#else
#define TEX1DFETCH(type, tex, idx) tex1Dfetch((tex), idx)
#endif


template<typename Tex>
static __inline__ __device__ double fetch_double(Tex t, int i)
{
  int2 v = TEX1DFETCH(int2, t, i);
  return __hiloint2double(v.y, v.x);
}

template <typename Tex>
static __inline__ __device__ double2 fetch_double2(Tex t, int i)
{
  int4 v = TEX1DFETCH(int4, t, i);
  return make_double2(__hiloint2double(v.y, v.x), __hiloint2double(v.w, v.z));
}

static __inline__ __device__ double2 fetch_double2_old(texture<int4, 1> t, int i)
{
  int4 v = tex1Dfetch(t,i);
  return make_double2(__hiloint2double(v.y, v.x), __hiloint2double(v.w, v.z));
}





namespace quda {
  namespace fermion_force {

    struct hisq_kernel_param_t{
      unsigned long threads;
      int X[4];
      int D[4];
      int D1h;
      int base_idx[4];
      int ghostDim[4];
      int color_matrix_stride;
      int thin_link_stride;
      int momentum_stride;

      mutable int oddness_change;

      void setStride(const QudaGaugeParam& param){
        int half_volume = param.X[0]*param.X[1]*param.X[2]*param.X[3]/2;
#ifdef MULTI_GPU
        int extended_half_volume = (param.X[0]+4)*(param.X[1]+4)*(param.X[2]+4)*(param.X[3]+4)/2;
        thin_link_stride = extended_half_volume + param.site_ga_pad;
        color_matrix_stride = extended_half_volume;
#else
        thin_link_stride  = half_volume + param.site_ga_pad;
        color_matrix_stride = half_volume;
#endif
        momentum_stride = half_volume + param.mom_ga_pad;
      }
    };


    //Double precision for site link
    texture<int4, 1> thinLink0TexDouble;
    texture<int4, 1> thinLink1TexDouble;

    //Single precision for site link
    texture<float2, 1, hipReadModeElementType> thinLink0TexSingle;
    texture<float2, 1, hipReadModeElementType> thinLink1TexSingle;

    texture<float4, 1, hipReadModeElementType> thinLink0TexSingle_recon;
    texture<float4, 1, hipReadModeElementType> thinLink1TexSingle_recon;


    texture<int4, 1> newOprod0TexDouble;
    texture<int4, 1> newOprod1TexDouble;
    texture<float2, 1, hipReadModeElementType>  newOprod0TexSingle;
    texture<float2, 1, hipReadModeElementType> newOprod1TexSingle;

    template <int dir>
    inline __device__ __host__ void updateCoords(int x[], int shift, const int X[4], const int partitioned[]){
#ifdef MULTI_GPU
      if (shift == 1) {
        x[dir] = (partitioned[dir] || (x[dir] != X[dir]+1)) ? x[dir]+1 : 2;
      } else if (shift == -1) {
        x[dir] = (partitioned[dir] || (x[dir] != 2)) ? x[dir]-1 : X[dir]+1;
      }
#else 
      x[dir] = (x[dir]+shift + X[dir])%X[dir];
#endif
    }

    inline __device__ __host__ void updateCoords(int x[], int dir, int shift, const int X[4], const int partitioned[]) {
      switch (dir) {
        case 0:
	  updateCoords<0>(x, shift, X, partitioned);
	  break;
        case 1:
	  updateCoords<1>(x, shift, X, partitioned);
	  break;
        case 2:
	  updateCoords<2>(x, shift, X, partitioned);
	  break;
        case 3:
	  updateCoords<3>(x, shift, X, partitioned);
	  break;
      }

      return;
    }


    __device__ __host__ inline int posDir(int dir){
      return (dir >= 4) ? 7-dir : dir;
    }


    //struct for holding the fattening path coefficients
    template<class Real>
      struct PathCoefficients
      {
        Real one; 
        Real three;
        Real five;
        Real seven;
        Real naik;
        Real lepage;
      };


    inline __device__ float2 operator*(float a, const float2 & b)
    {
      return make_float2(a*b.x,a*b.y);
    }

    inline __device__ double2 operator*(double a, const double2 & b)
    {
      return make_double2(a*b.x,a*b.y);
    }

    inline __device__ const float2 & operator+=(float2 & a, const float2 & b)
    {
      a.x += b.x;
      a.y += b.y;
      return a;
    }

    inline __device__ const double2 & operator+=(double2 & a, const double2 & b)
    {
      a.x += b.x;
      a.y += b.y;
      return a;
    }

    inline __device__ const float4 & operator+=(float4 & a, const float4 & b)
    {
      a.x += b.x;
      a.y += b.y;
      a.z += b.z;
      a.w += b.w;
      return a;
    }

    // Replication of code 
    // This structure is already defined in 
    // unitarize_utilities.h

    template<class T>
      struct RealTypeId; 

    template<>
      struct RealTypeId<float2>
      {
        typedef float Type;
      };

    template<>
      struct RealTypeId<double2>
      {
        typedef double Type;
      };


    template<class T>
      inline __device__
      void adjointMatrix(T* mat)
      {
#define CONJ_INDEX(i,j) j*3 + i

        T tmp;
        mat[CONJ_INDEX(0,0)] = Conj(mat[0]);
        mat[CONJ_INDEX(1,1)] = Conj(mat[4]);
        mat[CONJ_INDEX(2,2)] = Conj(mat[8]);
        tmp  = Conj(mat[1]);
        mat[CONJ_INDEX(1,0)] = Conj(mat[3]);
        mat[CONJ_INDEX(0,1)] = tmp;	
        tmp = Conj(mat[2]);
        mat[CONJ_INDEX(2,0)] = Conj(mat[6]);
        mat[CONJ_INDEX(0,2)] = tmp;
        tmp = Conj(mat[5]);
        mat[CONJ_INDEX(2,1)] = Conj(mat[7]);
        mat[CONJ_INDEX(1,2)] = tmp;

#undef CONJ_INDEX
        return;
      }


    template<int N, class T>
      inline __device__
      void loadMatrixFromField(const T* const field_even, const T* const field_odd,
          int dir, int idx, T* const mat, int oddness, int stride)
      {
        const T* const field = (oddness)?field_odd:field_even;
        for(int i = 0;i < N ;i++){
          mat[i] = field[idx + dir*N*stride + i*stride];          
        }
        return;
      }

    template<class T>
      inline __device__
      void loadMatrixFromField(const T* const field_even, const T* const field_odd,
          int dir, int idx, T* const mat, int oddness, int stride)
      {
        loadMatrixFromField<9> (field_even, field_odd, dir, idx, mat, oddness, stride);
        return;
      }



    inline __device__
      void loadMatrixFromField(const float4* const field_even, const float4* const field_odd, 
          int dir, int idx, float2* const mat, int oddness, int stride)
      {
        const float4* const field = oddness?field_odd: field_even;
        float4 tmp;
        tmp = field[idx + dir*stride*3];
        mat[0] = make_float2(tmp.x, tmp.y);
        mat[1] = make_float2(tmp.z, tmp.w);
        tmp = field[idx + dir*stride*3 + stride];
        mat[2] = make_float2(tmp.x, tmp.y);
        mat[3] = make_float2(tmp.z, tmp.w);
        tmp = field[idx + dir*stride*3 + 2*stride];
        mat[4] = make_float2(tmp.x, tmp.y);
        mat[5] = make_float2(tmp.z, tmp.w);
        return;
      }

    template<class T>
      inline __device__
      void loadMatrixFromField(const T* const field_even, const T* const field_odd, int idx, T* const mat, int oddness, int stride)
      {
        const T* const field = (oddness)?field_odd:field_even;
        mat[0] = field[idx];
        mat[1] = field[idx + stride];
        mat[2] = field[idx + stride*2];
        mat[3] = field[idx + stride*3];
        mat[4] = field[idx + stride*4];
        mat[5] = field[idx + stride*5];
        mat[6] = field[idx + stride*6];
        mat[7] = field[idx + stride*7];
        mat[8] = field[idx + stride*8];

        return;
      }

    template<class U>
      inline __device__
      void  addMatrixToNewOprod(const double2* const mat,  int dir, int idx, U coeff, 
          double2* const field_even, double2* const field_odd, int oddness, int stride){
        double2* const field = (oddness)?field_odd: field_even;		
        double2 value[9];			

#if (HISQ_NEW_OPROD_LOAD_TEX == 1)
        value[0] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9); 
        value[1] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9 + stride); 
        value[2] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9 + 2*stride); 
        value[3] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9 + 3*stride); 
        value[4] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9 + 4*stride); 
        value[5] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9 + 5*stride); 
        value[6] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9 + 6*stride); 
        value[7] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9 + 7*stride); 
        value[8] = READ_DOUBLE2_TEXTURE( ((oddness)?newOprod1TexDouble:newOprod0TexDouble), field, idx+dir*stride*9 + 8*stride); 
#else
        for(int i=0; i<9; ++i) value[i] = field[i];
#endif

        field[idx + dir*stride*9]              = value[0] + coeff*mat[0]; 
        field[idx + dir*stride*9 + stride]     = value[1] + coeff*mat[1];	
        field[idx + dir*stride*9 + stride*2]   = value[2] + coeff*mat[2];	
        field[idx + dir*stride*9 + stride*3]   = value[3] + coeff*mat[3];	
        field[idx + dir*stride*9 + stride*4]   = value[4] + coeff*mat[4];	
        field[idx + dir*stride*9 + stride*5]   = value[5] + coeff*mat[5];	
        field[idx + dir*stride*9 + stride*6]   = value[6] + coeff*mat[6];	
        field[idx + dir*stride*9 + stride*7]   = value[7] + coeff*mat[7];	
        field[idx + dir*stride*9 + stride*8]   = value[8] + coeff*mat[8];	

        return;
      }					


    template<class U>
      inline __device__
      void  addMatrixToNewOprod(const float2* const mat,  int dir, int idx, U coeff, 
          float2* const field_even, float2* const field_odd, int oddness, int stride){
        float2* const field = (oddness)?field_odd: field_even;		
        float2 value[9];			

#if (HISQ_NEW_OPROD_LOAD_TEX == 1)
        value[0] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9); 
        value[1] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9 + stride); 
        value[2] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9 + 2*stride); 
        value[3] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9 + 3*stride); 
        value[4] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9 + 4*stride); 
        value[5] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9 + 5*stride); 
        value[6] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9 + 6*stride); 
        value[7] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9 + 7*stride); 
        value[8] = tex1Dfetch( ((oddness)?newOprod1TexSingle:newOprod0TexSingle),  idx+dir*stride*9 + 8*stride); 
#else 
        for(int i=0; i<9; ++i) value[i] = field[i];
#endif
        field[idx + dir*stride*9]              = value[0] + coeff*mat[0]; 
        field[idx + dir*stride*9 + stride]     = value[1] + coeff*mat[1];	
        field[idx + dir*stride*9 + stride*2]   = value[2] + coeff*mat[2];	
        field[idx + dir*stride*9 + stride*3]   = value[3] + coeff*mat[3];	
        field[idx + dir*stride*9 + stride*4]   = value[4] + coeff*mat[4];	
        field[idx + dir*stride*9 + stride*5]   = value[5] + coeff*mat[5];	
        field[idx + dir*stride*9 + stride*6]   = value[6] + coeff*mat[6];	
        field[idx + dir*stride*9 + stride*7]   = value[7] + coeff*mat[7];	
        field[idx + dir*stride*9 + stride*8]   = value[8] + coeff*mat[8];	

        return;
      }					


    // only works if Promote<T,U>::Type = T

    template<class T, class U>   
      inline __device__
      void addMatrixToField(const T* const mat, int dir, int idx, U coeff, 
          T* const field_even, T* const field_odd, int oddness, int stride)
      {
        T* const field = (oddness)?field_odd: field_even;
        field[idx + dir*stride*9]          += coeff*mat[0];
        field[idx + dir*stride*9 + stride]     += coeff*mat[1];
        field[idx + dir*stride*9 + stride*2]   += coeff*mat[2];
        field[idx + dir*stride*9 + stride*3]   += coeff*mat[3];
        field[idx + dir*stride*9 + stride*4]   += coeff*mat[4];
        field[idx + dir*stride*9 + stride*5]   += coeff*mat[5];
        field[idx + dir*stride*9 + stride*6]   += coeff*mat[6];
        field[idx + dir*stride*9 + stride*7]   += coeff*mat[7];
        field[idx + dir*stride*9 + stride*8]   += coeff*mat[8];

        return;
      }


    template<class T, class U>
      inline __device__
      void addMatrixToField(const T* const mat, int idx, U coeff, T* const field_even,
          T* const field_odd, int oddness, int stride)
      {
        T* const field = (oddness)?field_odd: field_even;
        field[idx ]         += coeff*mat[0];
        field[idx + stride]     += coeff*mat[1];
        field[idx + stride*2]   += coeff*mat[2];
        field[idx + stride*3]   += coeff*mat[3];
        field[idx + stride*4]   += coeff*mat[4];
        field[idx + stride*5]   += coeff*mat[5];
        field[idx + stride*6]   += coeff*mat[6];
        field[idx + stride*7]   += coeff*mat[7];
        field[idx + stride*8]   += coeff*mat[8];

        return;
      }

    template<class T, class U>
      inline __device__
      void addMatrixToField_test(const T* const mat, int idx, U coeff, T* const field_even,
          T* const field_odd, int oddness, int stride)
      {
        T* const field = (oddness)?field_odd: field_even;
        //T oldvalue=field[idx];
        field[idx ]         += coeff*mat[0];
        field[idx + stride]     += coeff*mat[1];
        field[idx + stride*2]   += coeff*mat[2];
        field[idx + stride*3]   += coeff*mat[3];
        field[idx + stride*4]   += coeff*mat[4];
        field[idx + stride*5]   += coeff*mat[5];
        field[idx + stride*6]   += coeff*mat[6];
        field[idx + stride*7]   += coeff*mat[7];
        field[idx + stride*8]   += coeff*mat[8];

        printf("value is  coeff(%f) * mat[0].x(%f)=%f\n", coeff, mat[0].x, field[idx].x);
        return;
      }

    template<class T>
      inline __device__
      void storeMatrixToField(const T* const mat, int dir, int idx, T* const field_even, T* const field_odd, int oddness, int stride)
      {
        T* const field = (oddness)?field_odd: field_even;
        field[idx + dir*stride*9]          = mat[0];
        field[idx + dir*stride*9 + stride]     = mat[1];
        field[idx + dir*stride*9 + stride*2]   = mat[2];
        field[idx + dir*stride*9 + stride*3]   = mat[3];
        field[idx + dir*stride*9 + stride*4]   = mat[4];
        field[idx + dir*stride*9 + stride*5]   = mat[5];
        field[idx + dir*stride*9 + stride*6]   = mat[6];
        field[idx + dir*stride*9 + stride*7]   = mat[7];
        field[idx + dir*stride*9 + stride*8]   = mat[8];

        return;
      }


    template<class T>
      inline __device__
      void storeMatrixToField(const T* const mat, int idx, T* const field_even, T* const field_odd, int oddness, int stride)
      {
        T* const field = (oddness)?field_odd: field_even;
        field[idx]          = mat[0];
        field[idx + stride]     = mat[1];
        field[idx + stride*2]   = mat[2];
        field[idx + stride*3]   = mat[3];
        field[idx + stride*4]   = mat[4];
        field[idx + stride*5]   = mat[5];
        field[idx + stride*6]   = mat[6];
        field[idx + stride*7]   = mat[7];
        field[idx + stride*8]   = mat[8];

        return;
      }


    template<class T, class U> 
      inline __device__
      void storeMatrixToMomentumField(const T* const mat, int dir, int idx, U coeff, 
          T* const mom_even, T* const mom_odd, int oddness, int stride)
      {
        T* const mom_field = (oddness)?mom_odd:mom_even;
        T temp2;
        temp2.x = (mat[1].x - mat[3].x)*0.5*coeff;
        temp2.y = (mat[1].y + mat[3].y)*0.5*coeff;
        mom_field[idx + dir*stride*5] = temp2;	

        temp2.x = (mat[2].x - mat[6].x)*0.5*coeff;
        temp2.y = (mat[2].y + mat[6].y)*0.5*coeff;
        mom_field[idx + dir*stride*5 + stride] = temp2;

        temp2.x = (mat[5].x - mat[7].x)*0.5*coeff;
        temp2.y = (mat[5].y + mat[7].y)*0.5*coeff;
        mom_field[idx + dir*stride*5 + stride*2] = temp2;

        const typename RealTypeId<T>::Type temp = (mat[0].y + mat[4].y + mat[8].y)*0.3333333333333333333333333;
        temp2.x =  (mat[0].y-temp)*coeff; 
        temp2.y =  (mat[4].y-temp)*coeff;
        mom_field[idx + dir*stride*5 + stride*3] = temp2;

        temp2.x = (mat[8].y - temp)*coeff;
        temp2.y = 0.0;
        mom_field[idx + dir*stride*5 + stride*4] = temp2;

        return;
      }

    __device__ __host__ inline int CoeffSign(int pos_dir, int odd_lattice) {
      return 2*((pos_dir + odd_lattice + 1) & 1) - 1;
    }

    __device__ __host__ inline int Sign(int parity) {
      return parity ? -1 : 1;
    }

    template<class RealX>
      struct ArrayLength
      {
        static const int result=9;
      };

    template<>
      struct ArrayLength<float4>
      {
        static const int result=5;
      };




    // Flops: four matrix additions per lattice site = 72 Flops per lattice site
    template<class RealA>
      __global__ void 
      do_one_link_term_kernel(const RealA* const oprodEven, const RealA* const oprodOdd,
          typename RealTypeId<RealA>::Type coeff,
          RealA* const outputEven, RealA* const outputOdd, hisq_kernel_param_t kparam)
      {
        int sid = blockIdx.x * blockDim.x + threadIdx.x;
        if (sid >= kparam.threads) return;
	int oddBit = threadIdx.y;
#ifdef MULTI_GPU
        int dx[4] = {0,0,0,0};
        int x[4];
        getCoords(x, sid, kparam.X, oddBit);
        int E[4] = {kparam.X[0]+4, kparam.X[1]+4, kparam.X[2]+4, kparam.X[3]+4};
        for(int dir=0; dir<4; ++dir) x[dir] += 2; 
        int new_sid = linkIndexShift(x,dx,E);
#else
        int new_sid = sid;
#endif
	for(int sig=0; sig<4; ++sig){
          RealA COLOR_MAT_W[ArrayLength<RealA>::result];
          loadMatrixFromField(oprodEven, oprodOdd, sig, new_sid, COLOR_MAT_W, oddBit, kparam.color_matrix_stride);
          addMatrixToField(COLOR_MAT_W, sig, new_sid, coeff, outputEven, outputOdd, oddBit, kparam.color_matrix_stride);
	}
        return;
      }


    template<int N>
      __device__ void loadLink(const double2* const linkEven, const double2* const linkOdd, int dir, int idx, double2* const var, int oddness, int stride){
#if (HISQ_SITE_MATRIX_LOAD_TEX == 1)
        HISQ_LOAD_MATRIX_18_DOUBLE_TEX((oddness)?thinLink1TexDouble:thinLink0TexDouble,  (oddness)?linkOdd:linkEven, dir, idx, var, stride);        
#else
        loadMatrixFromField(linkEven, linkOdd, dir, idx, var, oddness, stride);
#endif
      }

    template<>
      void loadLink<12>(const double2* const linkEven, const double2* const linkOdd, int dir, int idx, double2* const var, int oddness, int stride){
#if (HISQ_SITE_MATRIX_LOAD_TEX == 1)
        HISQ_LOAD_MATRIX_12_DOUBLE_TEX((oddness)?thinLink1TexDouble:thinLink0TexDouble,  (oddness)?linkOdd:linkEven,dir, idx, var, stride);        
#else
        loadMatrixFromField<6>(linkEven, linkOdd, dir, idx, var, oddness, stride);
#endif
      }

    template<int N>
      __device__ void loadLink(const float4* const linkEven, const float4* const linkOdd, int dir, int idx, float2* const var, int oddness, int stride){
#if (HISQ_SITE_MATRIX_LOAD_TEX == 1)
        HISQ_LOAD_MATRIX_12_SINGLE_TEX((oddness)?thinLink1TexSingle_recon:thinLink0TexSingle_recon, dir, idx, var, stride);  
#else
        loadMatrixFromField(linkEven, linkOdd, dir, idx, var, oddness, stride);      
#endif  
      }

    template<int N>
      __device__ void loadLink(const float2* const linkEven, const float2* const linkOdd, int dir, int idx, float2* const var , int oddness, int stride){
#if (HISQ_SITE_MATRIX_LOAD_TEX == 1)
        HISQ_LOAD_MATRIX_18_SINGLE_TEX((oddness)?thinLink1TexSingle:thinLink0TexSingle, dir, idx, var, stride);        
#else
        loadMatrixFromField(linkEven, linkOdd, dir, idx, var, oddness, stride);        
#endif
      }



#define DD_CONCAT(n,r) n ## r ## kernel

#define HISQ_KERNEL_NAME(a,b) DD_CONCAT(a,b)
    //precision: 0 is for double, 1 is for single

    //double precision, recon=18
#define PRECISION 0
#define RECON 18
#include "hisq_paths_force_core.h"
#undef PRECISION
#undef RECON

    //double precision, recon=12
#define PRECISION 0
#define RECON 12
#include "hisq_paths_force_core.h"
#undef PRECISION
#undef RECON

    //single precision, recon=18  
#define PRECISION 1
#define RECON 18
#include "hisq_paths_force_core.h"
#undef PRECISION
#undef RECON

    //single precision, recon=12
#define PRECISION 1
#define RECON 12
#include "hisq_paths_force_core.h"
#undef PRECISION
#undef RECON





    template<class RealA, class RealB>
      class MiddleLink : public TunableLocalParity {

        private:
          const cudaGaugeField &link;
          const cudaGaugeField &oprod;
          const cudaGaugeField &Qprev;
          const int sig;
          const int mu;
          const typename RealTypeId<RealA>::Type &coeff; 
          cudaGaugeField &Pmu;
          cudaGaugeField &P3;
          cudaGaugeField &Qmu;
          cudaGaugeField &newOprod;
          const hisq_kernel_param_t &kparam;
          unsigned int minThreads() const { return kparam.threads; }

        public:
          MiddleLink(const cudaGaugeField &link, 
              const cudaGaugeField &oprod,
              const cudaGaugeField &Qprev,
              int sig, int mu,
              const typename RealTypeId<RealA>::Type &coeff, 
              cudaGaugeField &Pmu, // write only
              cudaGaugeField &P3,  // write only
              cudaGaugeField &Qmu,
              cudaGaugeField &newOprod,
              const hisq_kernel_param_t &kparam) :
            link(link), oprod(oprod), Qprev(Qprev), sig(sig), mu(mu), 
            coeff(coeff), Pmu(Pmu), P3(P3), Qmu(Qmu), newOprod(newOprod), kparam(kparam)
        {	; }
          // need alternative constructor to hack around null pointer passing
          MiddleLink(const cudaGaugeField &link, 
              const cudaGaugeField &oprod,
              int sig, int mu,
              const typename RealTypeId<RealA>::Type &coeff, 
              cudaGaugeField &Pmu, // write only
              cudaGaugeField &P3,  // write only
              cudaGaugeField &Qmu,
              cudaGaugeField &newOprod,
              const hisq_kernel_param_t &kparam) :
            link(link), oprod(oprod), Qprev(link), sig(sig), mu(mu), 
            coeff(coeff), Pmu(Pmu), P3(P3), Qmu(Qmu), newOprod(newOprod), kparam(kparam)
        {	; }
          virtual ~MiddleLink() { ; }

          TuneKey tuneKey() const {
            std::stringstream vol, aux;
            vol << kparam.D[0] << "x";
            vol << kparam.D[1] << "x";
            vol << kparam.D[2] << "x";
            vol << kparam.D[3];    
            aux << "threads=" << kparam.threads << ",prec=" << link.Precision();
            aux << ",recon=" << link.Reconstruct() << ",sig=" << sig << ",mu=" << mu;
            return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
          }  


#define CALL_ARGUMENTS(typeA, typeB) <<<tp.grid, tp.block>>>		\
          ((typeA*)oprod.Even_p(), (typeA*)oprod.Odd_p(),			\
           (typeA*)Qprev_even, (typeA*)Qprev_odd,				\
           (typeB*)link.Even_p(), (typeB*)link.Odd_p(),			\
           sig, mu, coeff,							\
           (typeA*)Pmu.Even_p(), (typeA*)Pmu.Odd_p(),			\
           (typeA*)P3.Even_p(), (typeA*)P3.Odd_p(),				\
           (typeA*)Qmu.Even_p(), (typeA*)Qmu.Odd_p(),			\
           (typeA*)newOprod.Even_p(), (typeA*)newOprod.Odd_p(), kparam)


#define CALL_MIDDLE_LINK_KERNEL(sig_sign, mu_sign)			\
      if (sizeof(RealA) == sizeof(float2)) {				\
	if (recon  == QUDA_RECONSTRUCT_NO) {				\
	  do_middle_link_sp_18_kernel<float2, float2, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float2); \
	} else {							\
	  do_middle_link_sp_12_kernel<float2, float4, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float4); \
	}								\
      } else {								\
	if (recon  == QUDA_RECONSTRUCT_NO) {				\
	  do_middle_link_dp_18_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
	} else {							\
	  do_middle_link_dp_12_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
	}								\
      }

          void apply(const hipStream_t &stream) {
            TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
            QudaReconstructType recon = link.Reconstruct();
            kparam.oddness_change = (kparam.base_idx[0] + kparam.base_idx[1]
				  + kparam.base_idx[2] + kparam.base_idx[3])&1;

            const void *Qprev_even = (&Qprev == &link) ? NULL : Qprev.Even_p();
            const void *Qprev_odd = (&Qprev == &link) ? NULL : Qprev.Odd_p();

            if (GOES_FORWARDS(sig) && GOES_FORWARDS(mu)){	
              CALL_MIDDLE_LINK_KERNEL(1,1);
            }else if (GOES_FORWARDS(sig) && GOES_BACKWARDS(mu)){
              CALL_MIDDLE_LINK_KERNEL(1,0);
            }else if (GOES_BACKWARDS(sig) && GOES_FORWARDS(mu)){
              CALL_MIDDLE_LINK_KERNEL(0,1);
            }else{
              CALL_MIDDLE_LINK_KERNEL(0,0);
            }
          }

#undef CALL_ARGUMENTS	
#undef CALL_MIDDLE_LINK_KERNEL

          void preTune() {
            Pmu.backup();
            P3.backup();
            Qmu.backup();
            newOprod.backup();
          }

          void postTune() {
            Pmu.restore();
            P3.restore();
            Qmu.restore();
            newOprod.restore();
          }

          long long flops() const { return 0; }
      };


    template<class RealA, class RealB>
      class LepageMiddleLink : public TunableLocalParity {

        private:
          const cudaGaugeField &link;
          const cudaGaugeField &oprod;
          const cudaGaugeField &Qprev;
          const int sig;
          const int mu;
          const typename RealTypeId<RealA>::Type &coeff; 
          cudaGaugeField &P3; // write only
          cudaGaugeField &newOprod;
          const hisq_kernel_param_t &kparam;
          unsigned int minThreads() const { return kparam.threads; }

        public:
          LepageMiddleLink(const cudaGaugeField &link, 
              const cudaGaugeField &oprod, 
              const cudaGaugeField &Qprev,
              int sig, int mu,
              const typename RealTypeId<RealA>::Type &coeff, 
              cudaGaugeField &P3, cudaGaugeField &newOprod,
              const hisq_kernel_param_t &kparam) :
            link(link), oprod(oprod), Qprev(Qprev), sig(sig), mu(mu), 
            coeff(coeff), P3(P3), newOprod(newOprod), kparam(kparam)
        {	; }
          virtual ~LepageMiddleLink() { ; }

          TuneKey tuneKey() const {
            std::stringstream vol, aux;
            vol << kparam.D[0] << "x";
            vol << kparam.D[1] << "x";
            vol << kparam.D[2] << "x";
            vol << kparam.D[3];    
            aux << "threads=" << kparam.threads << ",prec=" << link.Precision();
            aux << ",recon=" << link.Reconstruct() << ",sig=" << sig << ",mu=" << mu;
            return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
          }  

#define CALL_ARGUMENTS(typeA, typeB) <<<tp.grid, tp.block>>>		\
      ((typeA*)oprod.Even_p(), (typeA*)oprod.Odd_p(),			\
       (typeA*)Qprev.Even_p(), (typeA*)Qprev.Odd_p(),			\
       (typeB*)link.Even_p(), (typeB*)link.Odd_p(),			\
       sig, mu, coeff,							\
       (typeA*)P3.Even_p(), (typeA*)P3.Odd_p(),				\
       (typeA*)newOprod.Even_p(), (typeA*)newOprod.Odd_p(),		\
       kparam)
      
#define CALL_MIDDLE_LINK_KERNEL(sig_sign, mu_sign)			\
      if (sizeof(RealA) == sizeof(float2)) {				\
	if (recon  == QUDA_RECONSTRUCT_NO) {				\
	  do_lepage_middle_link_sp_18_kernel<float2, float2, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float2); \
	} else {							\
	  do_lepage_middle_link_sp_12_kernel<float2, float4, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float4); \
	}								\
      } else {								\
	if (recon  == QUDA_RECONSTRUCT_NO) {				\
	  do_lepage_middle_link_dp_18_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
	} else {							\
	  do_lepage_middle_link_dp_12_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
	}								\
      }									\
  
      void apply(const hipStream_t &stream) {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	QudaReconstructType recon = link.Reconstruct();
	kparam.oddness_change = (kparam.base_idx[0] + kparam.base_idx[1]
				 + kparam.base_idx[2] + kparam.base_idx[3])&1;
	
	if (GOES_FORWARDS(sig) && GOES_FORWARDS(mu)){	
	  CALL_MIDDLE_LINK_KERNEL(1,1);
	}else if (GOES_FORWARDS(sig) && GOES_BACKWARDS(mu)){
	  CALL_MIDDLE_LINK_KERNEL(1,0);
	}else if (GOES_BACKWARDS(sig) && GOES_FORWARDS(mu)){
	  CALL_MIDDLE_LINK_KERNEL(0,1);
	}else{
	  CALL_MIDDLE_LINK_KERNEL(0,0);
	}
	
      }
      
#undef CALL_ARGUMENTS	
#undef CALL_MIDDLE_LINK_KERNEL
      
      void preTune() {
	P3.backup();
	newOprod.backup();
      }
      
      void postTune() {
	P3.restore();
	newOprod.restore();
      }
      
      long long flops() const { 
	if(GOES_FORWARDS(sig)) return 810*kparam.X[0]*kparam.X[1]*kparam.X[2]*kparam.X[3];
	return kparam.X[0]*kparam.X[1]*kparam.X[2]*kparam.X[3]*396; 
      }
    };

    template<class RealA, class RealB>
      class SideLink : public TunableLocalParity {

        private:
          const cudaGaugeField &link;
          const cudaGaugeField &P3;
          const cudaGaugeField &oprod;
          const int sig;
          const int mu;
          const typename RealTypeId<RealA>::Type &coeff; 
          const typename RealTypeId<RealA>::Type &accumu_coeff;
          cudaGaugeField &shortP;
          cudaGaugeField &newOprod;
          const hisq_kernel_param_t &kparam;

          unsigned int minThreads() const { return kparam.threads; }

        public:
          SideLink(const cudaGaugeField &link, 
              const cudaGaugeField &P3,
              const cudaGaugeField &oprod,
              int sig, int mu, 
              const typename RealTypeId<RealA>::Type &coeff, 
              const typename RealTypeId<RealA>::Type &accumu_coeff,
              cudaGaugeField &shortP,
              cudaGaugeField &newOprod,
              const hisq_kernel_param_t &kparam) :
            link(link), P3(P3), oprod(oprod), 
            sig(sig), mu(mu), coeff(coeff), accumu_coeff(accumu_coeff), 
            shortP(shortP), newOprod(newOprod), kparam(kparam)
        {	; }
          virtual ~SideLink() { ; }

          TuneKey tuneKey() const {
            std::stringstream vol, aux;
            vol << kparam.D[0] << "x";
            vol << kparam.D[1] << "x";
            vol << kparam.D[2] << "x";
            vol << kparam.D[3];    
            aux << "threads=" << kparam.threads << ",prec=" << link.Precision();
            aux << ",recon=" << link.Reconstruct() << ",sig=" << sig << ",mu=" << mu;
            return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
          }  

#define CALL_ARGUMENTS(typeA, typeB) <<<tp.grid, tp.block>>>		\
          ((typeA*)P3.Even_p(), (typeA*)P3.Odd_p(),				\
           (typeA*)oprod.Even_p(),  (typeA*)oprod.Odd_p(),			\
           (typeB*)link.Even_p(), (typeB*)link.Odd_p(),			\
           sig, mu,								\
           coeff,			\
           (typename RealTypeId<typeA>::Type) accumu_coeff,			\
           (typeA*)shortP.Even_p(), (typeA*)shortP.Odd_p(),			\
           (typeA*)newOprod.Even_p(), (typeA*)newOprod.Odd_p(),		\
           kparam)

#define CALL_SIDE_LINK_KERNEL(sig_sign, mu_sign)			\
      if (sizeof(RealA) == sizeof(float2)) {				\
	if (recon  == QUDA_RECONSTRUCT_NO) {				\
	  do_side_link_sp_18_kernel<float2, float2, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float2); \
	} else {							\
	  do_side_link_sp_12_kernel<float2, float4, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float4); \
	}								\
      } else {								\
	if(recon  == QUDA_RECONSTRUCT_NO){				\
	  do_side_link_dp_18_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
	} else {							\
	  do_side_link_dp_12_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
	}								\
      }

          void apply(const hipStream_t &stream) {
            TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
            QudaReconstructType recon = link.Reconstruct();
            kparam.oddness_change = (kparam.base_idx[0] + kparam.base_idx[1]
                + kparam.base_idx[2] + kparam.base_idx[3])&1;

            if (GOES_FORWARDS(sig) && GOES_FORWARDS(mu)){
              CALL_SIDE_LINK_KERNEL(1,1);
            }else if (GOES_FORWARDS(sig) && GOES_BACKWARDS(mu)){
              CALL_SIDE_LINK_KERNEL(1,0); 
            }else if (GOES_BACKWARDS(sig) && GOES_FORWARDS(mu)){
              CALL_SIDE_LINK_KERNEL(0,1);
            }else{
              CALL_SIDE_LINK_KERNEL(0,0);
            }
          }

#undef CALL_SIDE_LINK_KERNEL
#undef CALL_ARGUMENTS      

          void preTune() {
            shortP.backup();
            newOprod.backup();
          }

          void postTune() {
            shortP.restore();
            newOprod.restore();
          }

          long long flops() const { return 0; }
      };


    template<class RealA, class RealB>
      class SideLinkShort : public TunableLocalParity {

        private:
          const cudaGaugeField &link;
          const cudaGaugeField &P3; 
          const int sig;
          const int mu;
          const typename RealTypeId<RealA>::Type &coeff; 
          cudaGaugeField &newOprod;
          const hisq_kernel_param_t &kparam;

          unsigned int minThreads() const { return kparam.threads; }

        public:
          SideLinkShort(const cudaGaugeField &link, const cudaGaugeField &P3, int sig, int mu, 
              const typename RealTypeId<RealA>::Type &coeff, cudaGaugeField &newOprod,
              const hisq_kernel_param_t &kparam) :
            link(link), P3(P3), sig(sig), mu(mu), coeff(coeff), newOprod(newOprod), kparam(kparam)
        {	; }
          virtual ~SideLinkShort() { ; }

          TuneKey tuneKey() const {
            std::stringstream vol, aux;
            vol << kparam.D[0] << "x";
            vol << kparam.D[1] << "x";
            vol << kparam.D[2] << "x";
            vol << kparam.D[3];    
            aux << "threads=" << kparam.threads << ",prec=" << link.Precision();
            aux << ",recon=" << link.Reconstruct() << ",sig=" << sig << ",mu=" << mu;
            return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
          }  

#define CALL_ARGUMENTS(typeA, typeB) <<<tp.grid, tp.block>>>		\
          ((typeA*)P3.Even_p(), (typeA*)P3.Odd_p(),				\
           (typeB*)link.Even_p(), (typeB*)link.Odd_p(),			\
           sig, mu,	(typename RealTypeId<typeA>::Type) coeff,		\
           (typeA*)newOprod.Even_p(), (typeA*)newOprod.Odd_p(), kparam)


#define CALL_SIDE_LINK_KERNEL(sig_sign, mu_sign)			\
    if (sizeof(RealA) == sizeof(float2)) {				\
      if (recon  == QUDA_RECONSTRUCT_NO) {				\
	do_side_link_short_sp_18_kernel<float2, float2, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float2); \
      }else{								\
	do_side_link_short_sp_12_kernel<float2, float4, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float4); \
      }									\
    } else {								\
      if(recon  == QUDA_RECONSTRUCT_NO){				\
	do_side_link_short_dp_18_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
      }else{								\
	do_side_link_short_dp_12_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
      }									\
    }

          void apply(const hipStream_t &stream) {
            TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
            QudaReconstructType recon = link.Reconstruct();
            kparam.oddness_change = (kparam.base_idx[0] + kparam.base_idx[1]
                + kparam.base_idx[2] + kparam.base_idx[3])&1;

            if (GOES_FORWARDS(sig) && GOES_FORWARDS(mu)){
              CALL_SIDE_LINK_KERNEL(1,1);
            }else if (GOES_FORWARDS(sig) && GOES_BACKWARDS(mu)){
              CALL_SIDE_LINK_KERNEL(1,0);

            }else if (GOES_BACKWARDS(sig) && GOES_FORWARDS(mu)){
              CALL_SIDE_LINK_KERNEL(0,1);
            }else{
              CALL_SIDE_LINK_KERNEL(0,0);
            }	
          }

#undef CALL_SIDE_LINK_KERNEL
#undef CALL_ARGUMENTS      


          void preTune() {
            newOprod.backup();
          }

          void postTune() {
            newOprod.restore();
          }

          long long flops() const { return 0; }
      };

    template<class RealA, class RealB>
      class AllLink : public TunableLocalParity {

        private:
          const cudaGaugeField &link;
          const cudaGaugeField &oprod;
          const cudaGaugeField &Qprev;
          const int sig;
          const int mu;
          const typename RealTypeId<RealA>::Type &coeff; 
          const typename RealTypeId<RealA>::Type &accumu_coeff;
          cudaGaugeField &shortP;
          cudaGaugeField &newOprod;
          const hisq_kernel_param_t &kparam;

          unsigned int minThreads() const { return kparam.threads; }

        public:
          AllLink(const cudaGaugeField &link, 
              const cudaGaugeField &oprod, 
              const cudaGaugeField &Qprev, 
              int sig, int mu,
              const typename RealTypeId<RealA>::Type &coeff, 
              const typename RealTypeId<RealA>::Type &accumu_coeff,
              cudaGaugeField &shortP, cudaGaugeField &newOprod, 
              const hisq_kernel_param_t &kparam) : 
            link(link), oprod(oprod), Qprev(Qprev), sig(sig), mu(mu), 
            coeff(coeff), accumu_coeff(accumu_coeff), shortP(shortP), 
            newOprod(newOprod), kparam(kparam)
        { ; }
          virtual ~AllLink() { ; }

          TuneKey tuneKey() const {
            std::stringstream vol, aux;
            vol << kparam.D[0] << "x";
            vol << kparam.D[1] << "x";
            vol << kparam.D[2] << "x";
            vol << kparam.D[3];    
            aux << "threads=" << kparam.threads << ",prec=" << link.Precision();
            aux << ",recon=" << link.Reconstruct() << ",sig=" << sig << ",mu=" << mu;
            return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
          }  

#define CALL_ARGUMENTS(typeA, typeB) <<<tp.grid, tp.block>>>		\
          ((typeA*)oprod.Even_p(), (typeA*)oprod.Odd_p(),			\
           (typeA*)Qprev.Even_p(), (typeA*)Qprev.Odd_p(),			\
           (typeB*)link.Even_p(), (typeB*)link.Odd_p(), sig,  mu,		\
           (typename RealTypeId<typeA>::Type)coeff,				\
           (typename RealTypeId<typeA>::Type)accumu_coeff,			\
           (typeA*)shortP.Even_p(),(typeA*)shortP.Odd_p(),			\
           (typeA*)newOprod.Even_p(), (typeA*)newOprod.Odd_p(), kparam)

#define CALL_ALL_LINK_KERNEL(sig_sign, mu_sign)				\
      if (sizeof(RealA) == sizeof(float2)) {				\
	if (recon  == QUDA_RECONSTRUCT_NO) {				\
	  do_all_link_sp_18_kernel<float2, float2, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float2); \
	} else {							\
	  do_all_link_sp_12_kernel<float2, float4, sig_sign, mu_sign> CALL_ARGUMENTS(float2, float4); \
	}								\
      } else {								\
	if (recon  == QUDA_RECONSTRUCT_NO) {				\
	  do_all_link_dp_18_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
	} else {							\
	  do_all_link_dp_12_kernel<double2, double2, sig_sign, mu_sign> CALL_ARGUMENTS(double2, double2); \
	}								\
      }

          void apply(const hipStream_t &stream) {
            TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
            QudaReconstructType recon = link.Reconstruct();
            kparam.oddness_change = (kparam.base_idx[0] + kparam.base_idx[1]
                + kparam.base_idx[2] + kparam.base_idx[3])&1;

            if (GOES_FORWARDS(sig) && GOES_FORWARDS(mu)){
              CALL_ALL_LINK_KERNEL(1, 1);
            }else if (GOES_FORWARDS(sig) && GOES_BACKWARDS(mu)){
              CALL_ALL_LINK_KERNEL(1, 0);
            }else if (GOES_BACKWARDS(sig) && GOES_FORWARDS(mu)){
              CALL_ALL_LINK_KERNEL(0, 1);
            }else{
              CALL_ALL_LINK_KERNEL(0, 0);
            }

            return;
          }

#undef CALL_ARGUMENTS
#undef CALL_ALL_LINK_KERNEL	    

          void preTune() {
            shortP.backup();
            newOprod.backup();
          }

          void postTune() {
            shortP.restore();
            newOprod.restore();
          }

          long long flops() const { 
	    if(GOES_FORWARDS(sig)) return kparam.X[0]*kparam.X[1]*kparam.X[2]*kparam.X[3]*1242;
	
	    return kparam.X[0]*kparam.X[1]*kparam.X[2]*kparam.X[3]*828;
	  }
      };


    template<class RealA, class RealB>
      class OneLinkTerm : public TunableLocalParity {

        private:
          const cudaGaugeField &oprod;
          const typename RealTypeId<RealA>::Type &coeff; 
          cudaGaugeField &ForceMatrix;
          int X[4];
          hisq_kernel_param_t kparam;

          unsigned int minThreads() const { return X[0]*X[1]*X[2]*X[3]/2; }

        public:
          OneLinkTerm(const cudaGaugeField &oprod,  
              const typename RealTypeId<RealA>::Type &coeff, 
              cudaGaugeField &ForceMatrix, const QudaGaugeParam& param) :
            oprod(oprod), coeff(coeff), ForceMatrix(ForceMatrix)
        { 
          for(int dir=0; dir<4; ++dir) X[dir] = param.X[dir];

          kparam.threads = X[0]*X[1]*X[2]*X[3]/2;
          for(int dir=0; dir<4; ++dir){
            kparam.X[dir] = X[dir];
          }
          kparam.setStride(param);
        }

          virtual ~OneLinkTerm() { ; }

          TuneKey tuneKey() const {
            std::stringstream vol, aux;
            vol << X[0] << "x";
            vol << X[1] << "x";
            vol << X[2] << "x";
            vol << X[3];    
            int threads = X[0]*X[1]*X[2]*X[3]/2;
            aux << "threads=" << threads << ",prec=" << oprod.Precision();
            aux << ",coeff=" << coeff;
            return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
          }  

          void apply(const hipStream_t &stream) {
            TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

            do_one_link_term_kernel<RealA><<<tp.grid,tp.block>>>(static_cast<const RealA*>(oprod.Even_p()), 
								 static_cast<const RealA*>(oprod.Odd_p()), 
								 coeff,
								 static_cast<RealA*>(ForceMatrix.Even_p()), 
								 static_cast<RealA*>(ForceMatrix.Odd_p()),
								 kparam);
          }

          void preTune() {
            ForceMatrix.backup();
          }

          void postTune() {
            ForceMatrix.restore();
          }

          long long flops() const { 
	    return 72*kparam.X[0]*kparam.X[1]*kparam.X[2]*kparam.X[3];
	  }
      };


    template<class RealA, class RealB>
      class LongLinkTerm : public TunableLocalParity {

        private:
          const cudaGaugeField &link;
          const cudaGaugeField &naikOprod;
          const typename RealTypeId<RealA>::Type naik_coeff;
          cudaGaugeField &output;
          int X[4];
          const hisq_kernel_param_t &kparam;

          unsigned int minThreads() const { return X[0]*X[1]*X[2]*X[3]/2; }

        public:
          LongLinkTerm(const cudaGaugeField &link, const cudaGaugeField &naikOprod,
              const typename RealTypeId<RealA>::Type &naik_coeff,
              cudaGaugeField &output, const hisq_kernel_param_t &kparam) :
            link(link), naikOprod(naikOprod),  naik_coeff(naik_coeff), output(output),
            kparam(kparam)
        { for(int dir=0; dir<4; ++dir) X[dir] = kparam.X[dir]; }

          virtual ~LongLinkTerm() { ; }

          TuneKey tuneKey() const {
            std::stringstream vol, aux;
            vol << X[0] << "x";
            vol << X[1] << "x";
            vol << X[2] << "x";
            vol << X[3];    
            int threads = X[0]*X[1]*X[2]*X[3]/2;
            aux << "threads=" << threads << ",prec=" << link.Precision();
            return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
          }  

#define CALL_ARGUMENTS(typeA, typeB) <<<tp.grid,tp.block>>>		\
          ((typeB*)link.Even_p(), (typeB*)link.Odd_p(),			\
           (typeA*)naikOprod.Even_p(),  (typeA*)naikOprod.Odd_p(),		\
           naik_coeff,							\
           (typeA*)output.Even_p(), (typeA*)output.Odd_p(),			\
           kparam);		

          void apply(const hipStream_t &stream) {
            TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
            QudaReconstructType recon = link.Reconstruct();

            if(sizeof(RealA) == sizeof(float2)){
              if(recon == QUDA_RECONSTRUCT_NO){
                do_longlink_sp_18_kernel<float2,float2> CALL_ARGUMENTS(float2, float2);
              }else{
                do_longlink_sp_12_kernel<float2,float4> CALL_ARGUMENTS(float2, float4);
              }
            }else{
              if(recon == QUDA_RECONSTRUCT_NO){
                do_longlink_dp_18_kernel<double2,double2> CALL_ARGUMENTS(double2, double2);
              }else{
                do_longlink_dp_12_kernel<double2,double2> CALL_ARGUMENTS(double2, double2);
              }
            }
          }

#undef CALL_ARGUMENTS	

          void preTune() {
            output.backup();
          }

          void postTune() {
            output.restore();
          }

          long long flops() const { return 4968*kparam.X[0]*kparam.X[1]*kparam.X[2]*kparam.X[3]; }
      };




    template<class RealA, class RealB>
      class CompleteForce : public TunableLocalParity {

        private:
          const cudaGaugeField &link;
          const cudaGaugeField &oprod;
          cudaGaugeField &mom;
          int X[4];
          hisq_kernel_param_t kparam;

          unsigned int minThreads() const { return X[0]*X[1]*X[2]*X[3]/2; }

        public:
          CompleteForce(const cudaGaugeField &link, const cudaGaugeField &oprod, 
             cudaGaugeField &mom, const QudaGaugeParam &param) :
            link(link), oprod(oprod), mom(mom)
        {  

          for(int dir=0; dir<4; ++dir){
            X[dir] = param.X[dir];
            kparam.X[dir] = X[dir];
          }
          kparam.threads = X[0]*X[1]*X[2]*X[3]/2;
          kparam.setStride(param);
        }

          virtual ~CompleteForce() { ; }

          TuneKey tuneKey() const {
            std::stringstream vol, aux;
            vol << X[0] << "x";
            vol << X[1] << "x";
            vol << X[2] << "x";
            vol << X[3];    
            int threads = X[0]*X[1]*X[2]*X[3]/2;
            aux << "threads=" << threads << ",prec=" << link.Precision();
            return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
          }  

#define CALL_ARGUMENTS(typeA, typeB)  <<<tp.grid, tp.block>>>		\
          ((typeB*)link.Even_p(), (typeB*)link.Odd_p(),			\
           (typeA*)oprod.Even_p(), (typeA*)oprod.Odd_p(),			\
           (typeA*)mom.Even_p(), (typeA*)mom.Odd_p(),			\
           kparam);		

          void apply(const hipStream_t &stream) {
            TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
            QudaReconstructType recon = link.Reconstruct();

            if(sizeof(RealA) == sizeof(float2)){
              if(recon == QUDA_RECONSTRUCT_NO){
                do_complete_force_sp_18_kernel<float2,float2> CALL_ARGUMENTS(float2, float2);
              }else{
                do_complete_force_sp_12_kernel<float2,float4> CALL_ARGUMENTS(float2, float4);
              }
            }else{
              if(recon == QUDA_RECONSTRUCT_NO){
                do_complete_force_dp_18_kernel<double2,double2> CALL_ARGUMENTS(double2, double2);
              }else{
                do_complete_force_dp_12_kernel<double2,double2> CALL_ARGUMENTS(double2, double2);
              }
            }
          }

#undef CALL_ARGUMENTS	

          void preTune() {
            mom.backup();
          }

          void postTune() {
            mom.restore();
          }

          long long flops() const { 
	    return kparam.X[0]*kparam.X[1]*kparam.X[2]*kparam.X[3]*792;
	  }
      };


    static void 
      bind_tex_link(const cudaGaugeField& link, const cudaGaugeField& newOprod)
      {
        if(link.Precision() == QUDA_DOUBLE_PRECISION){
          hipBindTexture(0, thinLink0TexDouble, link.Even_p(), link.Bytes()/2);
          hipBindTexture(0, thinLink1TexDouble, link.Odd_p(), link.Bytes()/2);

          hipBindTexture(0, newOprod0TexDouble, newOprod.Even_p(), newOprod.Bytes()/2);
          hipBindTexture(0, newOprod1TexDouble, newOprod.Odd_p(), newOprod.Bytes()/2);
        }else{
          if(link.Reconstruct() == QUDA_RECONSTRUCT_NO){
            hipBindTexture(0, thinLink0TexSingle, link.Even_p(), link.Bytes()/2);      
            hipBindTexture(0, thinLink1TexSingle, link.Odd_p(), link.Bytes()/2);      
          }else{
            hipBindTexture(0, thinLink0TexSingle_recon, link.Even_p(), link.Bytes()/2);      
            hipBindTexture(0, thinLink1TexSingle_recon, link.Odd_p(), link.Bytes()/2);            
          }
          hipBindTexture(0, newOprod0TexSingle, newOprod.Even_p(), newOprod.Bytes()/2);
          hipBindTexture(0, newOprod1TexSingle, newOprod.Odd_p(), newOprod.Bytes()/2);

        }
      }

    static void 
      unbind_tex_link(const cudaGaugeField& link, const cudaGaugeField& newOprod)
      {
        if(link.Precision() == QUDA_DOUBLE_PRECISION){
          hipUnbindTexture(thinLink0TexDouble);
          hipUnbindTexture(thinLink1TexDouble);
          hipUnbindTexture(newOprod0TexDouble);
          hipUnbindTexture(newOprod1TexDouble);
        }else{
          if(link.Reconstruct() == QUDA_RECONSTRUCT_NO){
            hipUnbindTexture(thinLink0TexSingle);
            hipUnbindTexture(thinLink1TexSingle);      
          }else{
            hipUnbindTexture(thinLink0TexSingle_recon);
            hipUnbindTexture(thinLink1TexSingle_recon);      
          }
          hipUnbindTexture(newOprod0TexSingle);
          hipUnbindTexture(newOprod1TexSingle);
        }
      }

    template<class Real, class RealA, class RealB>
      static void
      do_hisq_staples_force_cuda( PathCoefficients<Real> act_path_coeff,
          const QudaGaugeParam& param,
          const cudaGaugeField &oprod, 
          const cudaGaugeField &link,
          cudaGaugeField &Pmu,
          cudaGaugeField &P3,
          cudaGaugeField &P5,
          cudaGaugeField &Pnumu,
          cudaGaugeField &Qmu,
          cudaGaugeField &Qnumu,
          cudaGaugeField &newOprod)
      {

        Real coeff;
        Real OneLink, Lepage, FiveSt, ThreeSt, SevenSt;
        Real mLepage, mFiveSt, mThreeSt;

        OneLink = act_path_coeff.one;
        ThreeSt = act_path_coeff.three; mThreeSt = -ThreeSt;
        FiveSt  = act_path_coeff.five; mFiveSt  = -FiveSt;
        SevenSt = act_path_coeff.seven; 
        Lepage  = act_path_coeff.lepage; mLepage  = -Lepage;



       	OneLinkTerm<RealA, RealB> oneLink(oprod, OneLink, newOprod, param);
        oneLink.apply(0);
        checkCudaError();


        int ghostDim[4]={
          commDimPartitioned(0),
          commDimPartitioned(1),
          commDimPartitioned(2),
          commDimPartitioned(3)
        };

        hisq_kernel_param_t kparam_1g, kparam_2g;

        for(int dir=0; dir<4; ++dir){
          kparam_1g.X[dir] = param.X[dir];
          kparam_2g.X[dir] = param.X[dir];
        }

        kparam_1g.setStride(param);
        kparam_2g.setStride(param);


#ifdef MULTI_GPU
        kparam_1g.D[0] = commDimPartitioned(0)?(param.X[0]+2):(param.X[0]);
        kparam_1g.D[1] = commDimPartitioned(1)?(param.X[1]+2):(param.X[1]);
        kparam_1g.D[2] = commDimPartitioned(2)?(param.X[2]+2):(param.X[2]);
        kparam_1g.D[3] = commDimPartitioned(3)?(param.X[3]+2):(param.X[3]);
        kparam_1g.D1h =  kparam_1g.D[0]/2;
        kparam_1g.base_idx[0]=commDimPartitioned(0)?1:2;
        kparam_1g.base_idx[1]=commDimPartitioned(1)?1:2;
        kparam_1g.base_idx[2]=commDimPartitioned(2)?1:2;
        kparam_1g.base_idx[3]=commDimPartitioned(3)?1:2;
        kparam_1g.threads = kparam_1g.D[0]*kparam_1g.D[1]*kparam_1g.D[2]*kparam_1g.D[3]/2;

        kparam_2g.D[0] = commDimPartitioned(0)?(param.X[0]+4):(param.X[0]);
        kparam_2g.D[1] = commDimPartitioned(1)?(param.X[1]+4):(param.X[1]);
        kparam_2g.D[2] = commDimPartitioned(2)?(param.X[2]+4):(param.X[2]);
        kparam_2g.D[3] = commDimPartitioned(3)?(param.X[3]+4):(param.X[3]);
        kparam_2g.D1h = kparam_2g.D[0]/2;
        kparam_2g.base_idx[0]=commDimPartitioned(0)?0:2;
        kparam_2g.base_idx[1]=commDimPartitioned(1)?0:2;
        kparam_2g.base_idx[2]=commDimPartitioned(2)?0:2;
        kparam_2g.base_idx[3]=commDimPartitioned(3)?0:2;
        kparam_2g.threads = kparam_2g.D[0]*kparam_2g.D[1]*kparam_2g.D[2]*kparam_2g.D[3]/2;


        for(int i=0;i < 4; i++){
          kparam_1g.ghostDim[i] = kparam_2g.ghostDim[i]=kparam_1g.ghostDim[i]=kparam_2g.ghostDim[i] = ghostDim[i];
        }
#else
        hisq_kernel_param_t kparam;
        kparam.D[0] = param.X[0];
        kparam.D[1] = param.X[1];
        kparam.D[2] = param.X[2];
        kparam.D[3] = param.X[3];
        kparam.D1h = param.X[0]/2;
        kparam.threads=param.X[0]*param.X[1]*param.X[2]*param.X[3]/2;
        kparam.base_idx[0]=0;
        kparam.base_idx[1]=0;
        kparam.base_idx[2]=0;
        kparam.base_idx[3]=0;
        kparam_2g.threads = kparam_1g.threads = kparam.threads;
  
        for(int i=0; i<4; ++i){
          kparam_2g.D[i] = kparam_1g.D[i] = kparam.D[i];
          kparam_2g.D1h  = kparam_1g.D1h  = kparam.D1h;
          kparam_2g.base_idx[i] = kparam_1g.base_idx[i] = 0;
          kparam_2g.ghostDim[i] = kparam_1g.ghostDim[i] = 0;
        }
#endif
        for(int sig=0; sig<8; sig++){
          for(int mu=0; mu<8; mu++){
            if ( (mu == sig) || (mu == OPP_DIR(sig))){
              continue;
            }
            //3-link
            //Kernel A: middle link

            MiddleLink<RealA,RealB> middleLink( link, oprod,  // read only
                sig, mu, mThreeSt,
                Pmu, P3, Qmu, // write only
                newOprod, kparam_2g);
            middleLink.apply(0);
            checkCudaError();

            for(int nu=0; nu < 8; nu++){
              if (nu == sig || nu == OPP_DIR(sig)
                  || nu == mu || nu == OPP_DIR(mu)){
                continue;
              }
              //5-link: middle link
              //Kernel B
              MiddleLink<RealA,RealB> middleLink( link, Pmu, Qmu, // read only
                  sig, nu, FiveSt,
                  Pnumu, P5, Qnumu, // write only
                  newOprod, kparam_1g);
              middleLink.apply(0);
              checkCudaError();

              for(int rho = 0; rho < 8; rho++){
                if (rho == sig || rho == OPP_DIR(sig)
                    || rho == mu || rho == OPP_DIR(mu)
                    || rho == nu || rho == OPP_DIR(nu)){
                  continue;
                }

                //7-link: middle link and side link
                if(FiveSt != 0)coeff = SevenSt/FiveSt; else coeff = 0;
                AllLink<RealA,RealB> allLink(link, Pnumu, Qnumu, sig, rho, SevenSt, coeff,
                    P5, newOprod, kparam_1g);

                allLink.apply(0);
                checkCudaError();

                //return;
              }//rho  		

              //5-link: side link
              if(ThreeSt != 0)coeff = FiveSt/ThreeSt; else coeff = 0;
              SideLink<RealA,RealB> sideLink(link, P5, Qmu, //read only
                  sig, nu, mFiveSt, coeff,
                  P3, // write only
                  newOprod, kparam_1g);
              sideLink.apply(0);
              checkCudaError();

            } //nu 

            //lepage
            if(Lepage != 0.){
              LepageMiddleLink<RealA,RealB> 
                lepageMiddleLink ( link, Pmu, Qmu, // read only
                    sig, mu, Lepage,
                    P5, // write only
                    newOprod, kparam_2g);
              lepageMiddleLink.apply(0);
              checkCudaError();

              if(ThreeSt != 0)coeff = Lepage/ThreeSt ; else coeff = 0;

              SideLink<RealA, RealB> sideLink(link, P5, Qmu, // read only
                  sig, mu, mLepage, coeff,
                  P3, //write only
                  newOprod, kparam_2g);

              sideLink.apply(0);
              checkCudaError();		

            } // Lepage != 0.0

            //3-link side link
            SideLinkShort<RealA,RealB> sideLinkShort(link, P3, // read only
                sig, mu, ThreeSt,
                newOprod, kparam_1g);
            sideLinkShort.apply(0);
            checkCudaError();			    

          }//mu
        }//sig

        return; 
      } // do_hisq_staples_force_cuda


#undef Pmu
#undef Pnumu
#undef P3
#undef P5
#undef Qmu
#undef Qnumu


    void hisqCompleteForceCuda(const QudaGaugeParam &param,
        const cudaGaugeField &oprod,
        const cudaGaugeField &link,
        cudaGaugeField* force, 
	long long* flops)
    {
      bind_tex_link(link, oprod);

      if(param.cuda_prec == QUDA_DOUBLE_PRECISION){
        CompleteForce<double2,double2> completeForce(link, oprod, *force, param);
        completeForce.apply(0);
	if(flops) *flops = completeForce.flops();
        checkCudaError();
      }else if(param.cuda_prec == QUDA_SINGLE_PRECISION){
        CompleteForce<float2,float2> completeForce(link, oprod, *force, param);
        completeForce.apply(0);
	if(flops) *flops = completeForce.flops();
        checkCudaError();
      }else{
          errorQuda("Unsupported precision");
      }


      unbind_tex_link(link, oprod);
      return;
    }


    void hisqLongLinkForceCuda(double coeff,
        const QudaGaugeParam &param,
        const cudaGaugeField &oldOprod,
        const cudaGaugeField &link,
        cudaGaugeField  *newOprod,
	long long* flops)
    {
      bind_tex_link(link, *newOprod);
      const int volume = param.X[0]*param.X[1]*param.X[2]*param.X[3];
      hisq_kernel_param_t kparam;
      for(int i=0; i<4; i++){
        kparam.X[i] = param.X[i];
        kparam.ghostDim[i] = commDimPartitioned(i);
      }
      kparam.threads = volume/2;
      kparam.setStride(param);

      if(param.cuda_prec == QUDA_DOUBLE_PRECISION){
        LongLinkTerm<double2,double2> longLink(link, oldOprod, coeff, *newOprod, kparam);
        longLink.apply(0);
	if(flops) (*flops) = longLink.flops();
        checkCudaError();
      }else if(param.cuda_prec == QUDA_SINGLE_PRECISION){
        LongLinkTerm<float2,float2> longLink(link, oldOprod, static_cast<float>(coeff), *newOprod, kparam);
        longLink.apply(0);
	if(flops) (*flops) = longLink.flops();
        checkCudaError();
      }else{
        errorQuda("Unsupported precision");
      }
      unbind_tex_link(link, *newOprod);
      return;
    }





    void
      hisqStaplesForceCuda(const double path_coeff_array[6],
          const QudaGaugeParam &param,
          const cudaGaugeField &oprod, 
          const cudaGaugeField &link, 
          cudaGaugeField* newOprod,
	  long long* flops)
      {

#ifdef MULTI_GPU
        int X[4] = {
          param.X[0]+4,  param.X[1]+4,  param.X[2]+4,  param.X[3]+4
        };
#else
        int X[4] = {
          param.X[0],  param.X[1],  param.X[2],  param.X[3]
        };
#endif	

        // create color matrix fields with zero padding
        int pad = 0;
        GaugeFieldParam gauge_param(X, param.cuda_prec, QUDA_RECONSTRUCT_NO, pad, QUDA_SCALAR_GEOMETRY);

        gauge_param.ghostExchange = QUDA_GHOST_EXCHANGE_NO;
        gauge_param.siteSubset = QUDA_FULL_SITE_SUBSET;
        gauge_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        cudaGaugeField Pmu(gauge_param);
        cudaGaugeField P3(gauge_param);
        cudaGaugeField P5(gauge_param);
        cudaGaugeField Pnumu(gauge_param);
        cudaGaugeField Qmu(gauge_param);
        cudaGaugeField Qnumu(gauge_param);

        bind_tex_link(link, *newOprod);

        hipEvent_t start, end;

        hipEventCreate(&start);
        hipEventCreate(&end);

        hipEventRecord(start);
        if (param.cuda_prec == QUDA_DOUBLE_PRECISION){

          PathCoefficients<double> act_path_coeff;
          act_path_coeff.one    = path_coeff_array[0];
          act_path_coeff.naik   = path_coeff_array[1];
          act_path_coeff.three  = path_coeff_array[2];
          act_path_coeff.five   = path_coeff_array[3];
          act_path_coeff.seven  = path_coeff_array[4];
          act_path_coeff.lepage = path_coeff_array[5];
          do_hisq_staples_force_cuda<double,double2,double2>( act_path_coeff,
              param,
              oprod,
              link, 
              Pmu,
              P3,
              P5,
              Pnumu,
              Qmu,
              Qnumu,
              *newOprod);


        }else if(param.cuda_prec == QUDA_SINGLE_PRECISION){	
          PathCoefficients<float> act_path_coeff;
          act_path_coeff.one    = path_coeff_array[0];
          act_path_coeff.naik   = path_coeff_array[1];
          act_path_coeff.three  = path_coeff_array[2];
          act_path_coeff.five   = path_coeff_array[3];
          act_path_coeff.seven  = path_coeff_array[4];
          act_path_coeff.lepage = path_coeff_array[5];

          do_hisq_staples_force_cuda<float,float2,float2>( act_path_coeff,
              param,
              oprod,
              link, 
              Pmu,
              P3,
              P5,
              Pnumu,
              Qmu,
              Qnumu,
              *newOprod);
        }else{
          errorQuda("Unsupported precision");
        }


        hipEventRecord(end);
        hipEventSynchronize(end);
        float runtime;
        hipEventElapsedTime(&runtime, start, end);
	
	if(flops){
	  int volume = param.X[0]*param.X[1]*param.X[2]*param.X[3];
	  // Middle Link, side link, short side link, AllLink, OneLink
	  *flops = (134784 + 24192 + 103680 + 864 + 397440 + 72);
	  			
	  if(path_coeff_array[5] != 0.) *flops += 28944; // Lepage contribution
	  *flops *= volume;
	}

        unbind_tex_link(link, *newOprod);

        hipEventDestroy(start);
        hipEventDestroy(end);

        return; 
      }

  } // namespace fermion_force
} // namespace quda

#endif // GPU_HISQ_FORCE
