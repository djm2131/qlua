#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <gauge_field_order.h>

namespace quda {

  template <typename Order, int nDim, int dim>
  struct ExtractGhostExArg {
    Order order;
    int X[nDim];
    int R[nDim];
    int surfaceCB[nDim];
    int A0[nDim];
    int A1[nDim];
    int B0[nDim];
    int B1[nDim];
    int C0[nDim];
    int C1[nDim];
    int fBody[nDim][nDim];
    int fBuf[nDim][nDim];
    int localParity[nDim];
    ExtractGhostExArg(const Order &order, const int *X_, const int *R_, 
		      const int *surfaceCB_, 
		      const int *A0_, const int *A1_, const int *B0_, const int *B1_, 
		      const int *C0_, const int *C1_, const int fBody_[nDim][nDim], 
		      const int fBuf_[nDim][nDim], const int *localParity_) 
  : order(order) { 
      for (int d=0; d<nDim; d++) {
	X[d] = X_[d];
	R[d] = R_[d];
	surfaceCB[d] = surfaceCB_[d];
	A0[d] = A0_[d];
	A1[d] = A1_[d];
	B0[d] = B0_[d];
	B1[d] = B1_[d];
	C0[d] = C0_[d];
	C1[d] = C1_[d];
	for (int e=0; e<nDim; e++) {
	  fBody[d][e] = fBody_[d][e];
	  fBuf[d][e] = fBuf_[d][e];
	}
	localParity[d] = localParity_[d]; 
      }
    }

  };

  template <typename Float, int length, int dim, typename Arg>
  __device__ __host__ void extractor(Arg &arg, int dir, int a, int b, 
				     int c, int d, int g, int parity) {
    typename mapper<Float>::type u[length];
    int srcIdx = (a*arg.fBody[dim][0] + b*arg.fBody[dim][1] + 
		  c*arg.fBody[dim][2] + d*arg.fBody[dim][3]) >> 1;
    
    int dstIdx = (a*arg.fBuf[dim][0] + b*arg.fBuf[dim][1] + 
		  c*arg.fBuf[dim][2] + (d-(dir?arg.X[dim]:arg.R[dim]))*arg.fBuf[dim][3]) >> 1;
    
    // load the ghost element from the bulk
    arg.order.load(u, srcIdx, g, parity); 

    // need dir dependence in write
    // srcIdx is used here to determine boundary condition
    arg.order.saveGhostEx(u, dstIdx, srcIdx, dir, dim, g, 
			  (parity+arg.localParity[dim])&1, arg.R);
  }


  template <typename Float, int length, int dim, typename Arg>
  __device__ __host__ void injector(Arg &arg, int dir, int a, int b, 
				    int c, int d, int g, int parity) {
    typename mapper<Float>::type u[length];
    int srcIdx = (a*arg.fBuf[dim][0] + b*arg.fBuf[dim][1] + 
		  c*arg.fBuf[dim][2] + (d-dir*(arg.X[dim]+arg.R[dim]))*arg.fBuf[dim][3]) >> 1;
    
    int dstIdx = (a*arg.fBody[dim][0] + b*arg.fBody[dim][1] + 
		  c*arg.fBody[dim][2] + d*arg.fBody[dim][3]) >> 1;
    
    // need dir dependence in read
    // dstIdx is used here to determine boundary condition
    arg.order.loadGhostEx(u, srcIdx, dstIdx, dir, dim, g, 
			  (parity+arg.localParity[dim])&1, arg.R);
    
    arg.order.save(u, dstIdx, g, parity); // save the ghost element into the bulk
  }
  
  /**
     Generic CPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
  */
  template <typename Float, int length, int nDim, int dim, typename Order, bool extract>
  void extractGhostEx(ExtractGhostExArg<Order,nDim,dim> arg) {  
    typedef typename mapper<Float>::type RegType;

    for (int parity=0; parity<2; parity++) {

      // the following 4-way loop means this is specialized for 4 dimensions 
      // dir = 0 backwards, dir = 1 forwards
      for (int dir = 0; dir<2; dir++) {

	int D0 = extract ? dir*arg.X[dim] + (1-dir)*arg.R[dim] : dir*(arg.X[dim] + arg.R[dim]); 
	  
	for (int d=D0; d<D0+arg.R[dim]; d++) {
	  for (int a=arg.A0[dim]; a<arg.A1[dim]; a++) { // loop over the interior surface
	    for (int b=arg.B0[dim]; b<arg.B1[dim]; b++) { // loop over the interior surface
	      for (int c=arg.C0[dim]; c<arg.C1[dim]; c++) { // loop over the interior surface
		for (int g=0; g<arg.order.geometry; g++) {

		  // we only do the extraction for parity we are currently working on
		  int oddness = (a+b+c+d) & 1;
		  if (oddness == parity) {
		    if (extract) extractor<Float,length,dim>(arg, dir, a, b, c, d, g, parity);
		    else injector<Float,length,dim>(arg, dir, a, b, c, d, g, parity);
		  } // oddness == parity
		} // g
	      } // c
	    } // b
	  } // a
	} // d
      } // dir
      
    } // parity

  }

  /**
     Generic GPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
     FIXME this implementation will have two-way warp divergence
  */
  
  /**
     Generic CPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
  */
  template <typename Float, int length, int nDim, int dim, typename Order, bool extract>
  __global__ void extractGhostExKernel(ExtractGhostExArg<Order,nDim,dim> arg) {  
    typedef typename mapper<Float>::type RegType;

    // parallelize over parity and dir using block or grid 
    /*for (int parity=0; parity<2; parity++) {*/
    {
      int parity = blockIdx.z;

      // the following 4-way loop means this is specialized for 4 dimensions 
      // dir = 0 backwards, dir = 1 forwards
      //for (int dir = 0; dir<2; dir++) {
      {
	int dir = blockIdx.y;

	// this will have two-warp divergence since we only do work on
	// one parity but parity alternates between threads
	// linear index used for writing into ghost buffer
	int X = blockIdx.x * blockDim.x + threadIdx.x; 	

	int dA = arg.A1[dim]-arg.A0[dim];
	int dB = arg.B1[dim]-arg.B0[dim];
	int dC = arg.C1[dim]-arg.C0[dim];
	int D0 = extract ? dir*arg.X[dim] + (1-dir)*arg.R[dim] : dir*(arg.X[dim] + arg.R[dim]); 

	if (X >= arg.R[dim]*dA*dB*dC*arg.order.geometry) return;

	// thread order is optimized to maximize coalescing
	// X = (((g*R + d) * dA + a)*dB + b)*dC + c
	int gdab = X / dC;
	int c    = arg.C0[dim] + X    - gdab*dC;
	int gda  = gdab / dB;
	int b    = arg.B0[dim] + gdab - gda *dB;
	int gd   = gda / dA;
	int a    = arg.A0[dim] + gda  - gd  *dA;
	int g    = gd / arg.R[dim];
	int d    = D0          + gd   - g   *arg.R[dim];

	// we only do the extraction for parity we are currently working on
	int oddness = (a+b+c+d) & 1;
	if (oddness == parity) {
	  if (extract) extractor<Float,length,dim>(arg, dir, a, b, c, d, g, parity);
	  else injector<Float,length,dim>(arg, dir, a, b, c, d, g, parity);
	} // oddness == parity
      } // dir
      
    } // parity

  }

  template <typename Float, int length, int nDim, int dim, typename Order>
  class ExtractGhostEx : Tunable {
    ExtractGhostExArg<Order,nDim,dim> arg;
    int size;
    bool extract;
    const GaugeField &meta;
    QudaFieldLocation location;

  private:
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0 ;}

    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return size; }

  public:
    ExtractGhostEx(ExtractGhostExArg<Order,nDim,dim> &arg, bool extract, 
		   const GaugeField &meta, QudaFieldLocation location)
      : arg(arg), extract(extract), meta(meta), location(location) {
      int dA = arg.A1[dim]-arg.A0[dim];
      int dB = arg.B1[dim]-arg.B0[dim];
      int dC = arg.C1[dim]-arg.C0[dim];
      size = arg.R[dim]*dA*dB*dC*arg.order.geometry;
      writeAuxString("prec=%lu,stride=%d,extract=%d,dimension=%d",
		     sizeof(Float),arg.order.stride, extract, dim);
    }
    virtual ~ExtractGhostEx() { ; }
  
    void apply(const hipStream_t &stream) {
      if (extract) {
	if (location==QUDA_CPU_FIELD_LOCATION) {
	  extractGhostEx<Float,length,nDim,dim,Order,true>(arg);
	} else {
	  TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	  tp.grid.y = 2;
	  tp.grid.z = 2;
	  extractGhostExKernel<Float,length,nDim,dim,Order,true> 
	    <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	}
      } else { // we are injecting
	if (location==QUDA_CPU_FIELD_LOCATION) {
	  extractGhostEx<Float,length,nDim,dim,Order,false>(arg);
	} else {
	  TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	  tp.grid.y = 2;
	  tp.grid.z = 2;
	  extractGhostExKernel<Float,length,nDim,dim,Order,false> 
	    <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	}
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    std::string paramString(const TuneParam &param) const { // Don't bother printing the grid dim.
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }

    long long flops() const { return 0; } 
    long long bytes() const { return 2 * 2 * 2 * size * arg.order.Bytes(); } // 2 for i/o    
  };


  /**
     Generic CPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
     @param E the extended gauge dimensions
     @param R array holding the radius of the extended region 
     @param extract Whether we are extracting or injecting the ghost zone
  */
  template <typename Float, int length, typename Order>
  void extractGhostEx(Order order, const int dim, const int *surfaceCB, const int *E, 
		      const int *R, bool extract, const GaugeField &u, QudaFieldLocation location) {
    const int nDim = 4;
    //loop variables: a, b, c with a the most signifcant and c the least significant
    //A0, B0, C0 the minimum value
    //A0, B0, C0 the maximum value

    int X[nDim]; // compute interior dimensions
    for (int d=0; d<nDim; d++) X[d] = E[d] - 2*R[d];

    //..........x..........y............z.............t
    int A0[nDim] = {R[3],      R[3],        R[3],         0};
    int A1[nDim] = {X[3]+R[3], X[3]+R[3],   X[3]+R[3],    X[2]+2*R[2]};
    
    int B0[nDim] = {R[2],      R[2],        0,            0};
    int B1[nDim] = {X[2]+R[2], X[2]+R[2],   X[1]+2*R[1],  X[1]+2*R[1]};
    
    int C0[nDim] = {R[1],      0,           0,            0};
    int C1[nDim] = {X[1]+R[1], X[0]+2*R[0], X[0]+2*R[0],  X[0]+2*R[0]};

    int fSrc[nDim][nDim] = {
      {E[2]*E[1]*E[0], E[1]*E[0], E[0],              1},
      {E[2]*E[1]*E[0], E[1]*E[0],    1,           E[0]},
      {E[2]*E[1]*E[0],      E[0],    1,      E[1]*E[0]},
      {E[1]*E[0],           E[0],    1, E[2]*E[1]*E[0]}
    };  
  
    int fBuf[nDim][nDim]={
      {E[2]*E[1], E[1], 1, E[3]*E[2]*E[1]},
      {E[2]*E[0], E[0], 1, E[3]*E[2]*E[0]}, 
      {E[1]*E[0], E[0], 1, E[3]*E[1]*E[0]},
      {E[1]*E[0], E[0], 1, E[2]*E[1]*E[0]}
    };

    //set the local processor parity 
    //switching odd and even ghost gauge when that dimension size is odd
    //only switch if X[dir] is odd and the gridsize in that dimension is greater than 1
    // FIXME - I don't understand this, shouldn't it be commDim(dim) == 0 ?
    int localParity[nDim];
    for (int d=0; d<nDim; d++) 
      localParity[dim] = ((X[dim] % 2 ==1) && (commDim(dim) > 1)) ? 1 : 0;
    //      localParity[dim] = (X[dim]%2==0 || commDim(dim)) ? 0 : 1;

    if (dim==0) {
      ExtractGhostExArg<Order,nDim,0> arg(order, X, R, surfaceCB, A0, A1, B0, B1, 
					  C0, C1, fSrc, fBuf, localParity);
      ExtractGhostEx<Float,length,nDim,0,Order> extractor(arg, extract, u, location);
      extractor.apply(0);
    } else if (dim==1) {
      ExtractGhostExArg<Order,nDim,1> arg(order, X, R, surfaceCB, A0, A1, B0, B1, 
					  C0, C1, fSrc, fBuf, localParity);
      ExtractGhostEx<Float,length,nDim,1,Order> extractor(arg, extract, u, location);
      extractor.apply(0);
    } else if (dim==2) {
      ExtractGhostExArg<Order,nDim,2> arg(order, X, R, surfaceCB, A0, A1, B0, B1, 
					  C0, C1, fSrc, fBuf, localParity);
      ExtractGhostEx<Float,length,nDim,2,Order> extractor(arg, extract, u, location);
      extractor.apply(0);
    } else if (dim==3) {
      ExtractGhostExArg<Order,nDim,3> arg(order, X, R, surfaceCB, A0, A1, B0, B1, 
					  C0, C1, fSrc, fBuf, localParity);
      ExtractGhostEx<Float,length,nDim,3,Order> extractor(arg, extract, u, location);
      extractor.apply(0);
    } else {
      errorQuda("Invalid dim=%d", dim);
    }

    if (location == QUDA_CUDA_FIELD_LOCATION) {
      hipDeviceSynchronize(); // need to sync before we commence any communication
      checkCudaError();
    }
  }

  /** This is the template driver for extractGhost */
  template <typename Float>
  void extractGhostEx(const GaugeField &u, int dim, const int *R, Float **Ghost, bool extract) {

    const int length = 18;

    QudaFieldLocation location = 
      (typeid(u)==typeid(cudaGaugeField)) ? QUDA_CUDA_FIELD_LOCATION : QUDA_CPU_FIELD_LOCATION;

    if (u.isNative()) {
      if (u.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	if (typeid(Float)==typeid(short) && u.LinkType() == QUDA_ASQTAD_FAT_LINKS) {
	  extractGhostEx<short,length>(FloatNOrder<short,length,2,19>(u, 0, (short**)Ghost), 
				       dim, u.SurfaceCB(), u.X(), R, extract, u, location);
	} else {
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type G;
	  extractGhostEx<Float,length>(G(u, 0, Ghost),
				       dim, u.SurfaceCB(), u.X(), R, extract, u, location);
	}
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_12) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type G;
	extractGhostEx<Float,length>(G(u, 0, Ghost),
				     dim, u.SurfaceCB(), u.X(), R, extract, u, location);
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_8) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type G;
	extractGhostEx<Float,length>(G(u, 0, Ghost), 
				     dim, u.SurfaceCB(), u.X(), R, extract, u, location);
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_13) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_13>::type G;
	extractGhostEx<Float,length>(G(u, 0, Ghost),
				     dim, u.SurfaceCB(), u.X(), R, extract, u, location);
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_9) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_13>::type G;
	extractGhostEx<Float,length>(G(u, 0, Ghost),
				     dim, u.SurfaceCB(), u.X(), R, extract, u, location);
      }
    } else if (u.Order() == QUDA_QDP_GAUGE_ORDER) {
      
#ifdef BUILD_QDP_INTERFACE
      extractGhostEx<Float,length>(QDPOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("QDP interface has not been built\n");
#endif
      
    } else if (u.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      extractGhostEx<Float,length>(QDPJITOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      extractGhostEx<Float,length>(CPSOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      extractGhostEx<Float,length>(MILCOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      extractGhostEx<Float,length>(BQCDOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      extractGhostEx<Float,length>(TIFROrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", u.Order());
    }

  }

  void extractExtendedGaugeGhost(const GaugeField &u, int dim, const int *R, 
				 void **ghost, bool extract) {

    if (u.Precision() == QUDA_DOUBLE_PRECISION) {
      extractGhostEx(u, dim, R, (double**)ghost, extract);
    } else if (u.Precision() == QUDA_SINGLE_PRECISION) {
      extractGhostEx(u, dim, R, (float**)ghost, extract);
    } else if (u.Precision() == QUDA_HALF_PRECISION) {
      extractGhostEx(u, dim, R, (short**)ghost, extract);      
    } else {
      errorQuda("Unknown precision type %d", u.Precision());
    }

  }

} // namespace quda
