#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <clover_field.h>
#include <gauge_field.h>
#include <gauge_field_order.h>

namespace CloverOrder {
  using namespace quda;
#include <clover_field_order.h>
} // CloverOrder


namespace quda {

#ifdef GPU_CLOVER_DIRAC

  template<typename Clover1, typename Clover2, typename Gauge>
    struct CloverTraceArg {
      Clover1 clover1;
      Clover2 clover2;
      Gauge gauge;
      int dir1;
      int dir2;

      CloverTraceArg(Clover1 &clover1, Clover2 &clover2, Gauge &gauge, int dir1, int dir2)
        : clover1(clover1), clover2(clover2), gauge(gauge), dir1(dir1), dir2(dir2) {}
    };


  template <typename Float, typename Clover1, typename Clover2, typename Gauge>
    __device__ __host__ void cloverSigmaTraceCompute(CloverTraceArg<Clover1, Clover2, Gauge>& arg, int x, int parity) 
    {

      Float A[72];
      typedef typename ComplexTypeId<Float>::Type Complex;

      Matrix<Complex,3> mat;  
      setZero(&mat);

      // load the clover term into memory

      int dir1 = arg.dir1;
      int dir2 = arg.dir2;


      Float sign = 1;
      if(dir2 < dir1){
        int tmp = dir2;
        dir2 = dir1;
        dir1 = tmp;
        sign = -1;
      }


      Float diag[2][6];
      complex<Float> tri[2][15];
      const int idtab[15]={0,1,3,6,10,2,4,7,11,5,8,12,9,13,14};
      complex<Float> ctmp;

      if(parity==0){
        arg.clover1.load(A,x,parity);
      }else{
        arg.clover2.load(A,x,parity);
      }

      for(int ch=0; ch<2; ++ch){
        // factor of two is inherent to QUDA clover storage
        for (int i=0; i<6; i++) diag[ch][i] = 2.0*A[ch*36+i];
        for (int i=0; i<15; i++) tri[ch][idtab[i]] = complex<Float>(2.0*A[ch*36+6+2*i], 2.0*A[ch*36+6+2*i+1]);
      }


      // X, Y
      if(dir1 == 0){
        if(dir2 == 1){
          for(int j=0; j<3; ++j){
            mat(j,j).y = diag[0][j+3] + diag[1][j+3] - diag[0][j] - diag[1][j];  
          }

          // triangular part
          int jk=0;
          for(int j=1; j<3; ++j){
            int jk2 = (j+3)*(j+2)/2 + 3;
            for(int k=0; k<j; ++k){
              ctmp = tri[0][jk2] + tri[1][jk2] - tri[0][jk] - tri[1][jk];

              mat(j,k).x = -ctmp.imag();
              mat(j,k).y =  ctmp.real();

              mat(k,j).x =  ctmp.imag();
              mat(k,j).y =  ctmp.real();

              jk++; jk2++; 
            }
          } // X Y


        }else if(dir2 == 2){

          for(int j=0; j<3; ++j){
            int jk = (j+3)*(j+2)/2;
            for(int k=0; k<3; ++k){
              int kj = (k+3)*(k+2)/2 + j;
              ctmp = conj(tri[0][kj]) - tri[0][jk] + conj(tri[1][kj]) - tri[1][jk];
              mat(j,k).x = ctmp.real();
              mat(j,k).y = ctmp.imag();
              jk++;
            }
          } // X Z

        }else if(dir2 == 3){
          for(int j=0; j<3; ++j){
            int jk = (j+3)*(j+2)/2;
            for(int k=0; k<3; ++k){
              int kj = (k+3)*(k+2)/2 + j;
              ctmp = conj(tri[0][kj]) + tri[0][jk] - conj(tri[1][kj]) - tri[1][jk]; 
              mat(j,k).x = -ctmp.imag();
              mat(j,k).y =  ctmp.real();
              jk++;
            }
          }

        } // dir2 == 3 // X T

      }else if(dir1 == 1){
        if(dir2 == 2){ // Y Z
          for(int j=0; j<3; ++j){
            int jk = (j+3)*(j+2)/2;
            for(int k=0; k<3; ++k){
              int kj = (k+3)*(k+2)/2 + j;
              ctmp = conj(tri[0][kj]) + tri[0][jk] + conj(tri[1][kj]) + tri[1][jk];
              mat(j,k).x =  ctmp.imag();
              mat(j,k).y = -ctmp.real();
              jk++;
            }
          }
        }else if(dir2 == 3){ // Y T
          for(int j=0; j<3; ++j){
            int jk = (j+3)*(j+2)/2;
            for(int k=0; k<3; ++k){
              int kj = (k+3)*(k+2)/2 + j;
              ctmp = conj(tri[0][kj]) - tri[0][jk] - conj(tri[1][kj]) + tri[1][jk];
              mat(j,k).x = ctmp.real();
              mat(j,k).y = ctmp.imag();
              jk++;
            }
          }
        } // dir2 == 3
      } // dir1 == 1
      else if(dir1 == 2){
        if(dir2 == 3){
          for(int j=0; j<3; ++j){
            mat(j,j).y = diag[0][j] - diag[0][j+3] - diag[1][j] + diag[1][j+3];
          }
          int jk=0;
          for(int j=1; j<3; ++j){
            int jk2 = (j+3)*(j+2)/2 + 3;
            for(int k=0; k<j; ++k){
              ctmp = tri[0][jk] - tri[0][jk2] - tri[1][jk] + tri[1][jk2];
              mat(j,k).x = -ctmp.imag();
              mat(j,k).y =  ctmp.real();

              mat(k,j).x = ctmp.imag();
              mat(k,j).y = ctmp.real();
              jk++; jk2++;
            }
          }
        }
      }
      // if we dir1 and dir2 were swapped, multiply by -1
      mat *= sign;

      arg.gauge.save((Float*)(mat.data), x, 0, parity);

      return;
    }

  template<typename Float, typename Clover1, typename Clover2, typename Gauge>
    void cloverSigmaTrace(CloverTraceArg<Clover1,Clover2,Gauge> arg)
    {
      for(int x=0; x<arg.clover1.volumeCB; x++){
        cloverSigmaTraceCompute<Float,Clover1,Clover2,Gauge>(arg, x, 1);
      }
      return;
    }


  template<typename Float, typename Clover1, typename Clover2, typename Gauge>
    __global__ void cloverSigmaTraceKernel(CloverTraceArg<Clover1,Clover2,Gauge> arg)
    {
      int idx = blockIdx.x*blockDim.x + threadIdx.x;
      if(idx >= arg.clover1.volumeCB) return;
      // odd parity
      cloverSigmaTraceCompute<Float,Clover1,Clover2,Gauge>(arg, idx, 1);
    }

  template<typename Float, typename Clover1, typename Clover2, typename Gauge>
    class CloverSigmaTrace : Tunable {
      CloverTraceArg<Clover1,Clover2,Gauge> arg;
      const GaugeField &meta;
      const QudaFieldLocation location;

      private:
      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool tuneSharedBytes() const { return false; } // Don't tune the shared memory
      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
      unsigned int minThreads() const { return arg.clover1.volumeCB; }

      public: 
      CloverSigmaTrace(CloverTraceArg<Clover1,Clover2,Gauge> &arg, const GaugeField &meta, QudaFieldLocation location)
        : arg(arg), meta(meta), location(location) {
	writeAuxString("stride=%d", arg.clover1.stride);
      }
      virtual ~CloverSigmaTrace() {;}

      void apply(const hipStream_t &stream){
        if (location == QUDA_CUDA_FIELD_LOCATION) {
	  TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
          cloverSigmaTraceKernel<Float,Clover1,Clover2,Gauge><<<tp.grid,tp.block,0>>>(arg);
        } else {
          cloverSigmaTrace<Float,Clover1,Clover2,Gauge>(arg);
        }
      }

      TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

      std::string paramString(const TuneParam &param) const { // Don't print the grid dim.
        std::stringstream ps;
        ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
        ps << "shared=" << param.shared_bytes;
        return ps.str();
      }

      long long flops() const { return 0; } // Fix this
      long long bytes() const { return (arg.clover1.Bytes() + arg.gauge.Bytes()) * arg.clover1.volumeCB; } 

    }; // CloverSigmaTrace


  template<typename Float, typename Clover1, typename Clover2, typename Gauge>
  void computeCloverSigmaTrace(Clover1 clover1, Clover2 clover2, Gauge gauge, int dir1, int dir2,
			       const GaugeField &meta, QudaFieldLocation location)
  {
    CloverTraceArg<Clover1, Clover2, Gauge> arg(clover1, clover2, gauge, dir1, dir2);
    CloverSigmaTrace<Float,Clover1,Clover2,Gauge> traceCompute(arg, meta, location);
    traceCompute.apply(0);
    return;
  }

  template<typename Float>
    void computeCloverSigmaTrace(GaugeField& gauge, const CloverField& clover, int dir1, int dir2,
        QudaFieldLocation location){

    if(clover.isNative()) {
      typedef typename CloverOrder::quda::clover_mapper<Float>::type C;
      if (gauge.isNative()) {
	if (gauge.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type G;
	  computeCloverSigmaTrace<Float>( C(clover,0), C(clover,1), G(gauge), dir1, dir2, gauge, location);
	} else if(gauge.Reconstruct() == QUDA_RECONSTRUCT_12) {
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type G;
	  computeCloverSigmaTrace<Float>( C(clover,0), C(clover,1), G(gauge), dir1, dir2, gauge, location);
	} else {
	  errorQuda("Reconstruction type %d not supported", gauge.Reconstruct());
	}
      } else {
	errorQuda("Gauge order %d not supported", gauge.Order());
      }
    } else {
      errorQuda("clover order %d not supported", clover.Order());
    } // clover order

  }

#endif

  void computeCloverSigmaTrace(GaugeField& output, const CloverField& clover, int dir1, int dir2, 
      QudaFieldLocation location){

#ifdef GPU_CLOVER_DIRAC
    if(clover.Precision() == QUDA_HALF_PRECISION){
      errorQuda("Half precision not supported\n");
    }  

    if(clover.Precision() == QUDA_SINGLE_PRECISION){
      computeCloverSigmaTrace<float>(output, clover, dir1, dir2, location);
    }else if(clover.Precision() == QUDA_DOUBLE_PRECISION){
      computeCloverSigmaTrace<double>(output, clover, dir1, dir2, location);
    }else{
      errorQuda("Precision %d not supported", clover.Precision());
    }

#else
    errorQuda("Clover has not been built");
#endif

  }     


} // namespace quda
