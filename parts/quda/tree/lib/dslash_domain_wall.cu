#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

namespace quda {

  namespace domainwall {

#undef GPU_STAGGERED_DIRAC
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>
    
    // Enable shared memory dslash for Fermi architecture
    //#define SHARED_WILSON_DSLASH
    //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#ifdef GPU_DOMAIN_WALL_DIRAC
#include <dw_dslash_def.h>        // Domain Wall kernels
#endif

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>
  }

  // declare the dslash events
#include <dslash_events.cuh>

  using namespace domainwall;

#ifdef GPU_DOMAIN_WALL_DIRAC
  template <typename sFloat, typename gFloat>
  class DomainWallDslashCuda : public DslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const double mferm;
    const double a;

    bool checkGrid(TuneParam &param) const {
      if (param.grid.x > deviceProp.maxGridSize[0] || param.grid.y > deviceProp.maxGridSize[1]) {
	warningQuda("Autotuner is skipping blockDim=(%u,%u,%u), gridDim=(%u,%u,%u) because lattice volume is too large",
                    param.block.x, param.block.y, param.block.z, 
                    param.grid.x, param.grid.y, param.grid.z);
	return false;
      } else {
	return true;
      }
    }

  protected:
    bool advanceBlockDim(TuneParam &param) const
    {
      const unsigned int max_shared = 16384; // FIXME: use deviceProp.sharedMemPerBlock;
      const int step[2] = { deviceProp.warpSize, 1 };
      bool advance[2] = { false, false };

      // first try to advance block.x
      param.block.x += step[0];
      if (param.block.x > deviceProp.maxThreadsDim[0] || 
	  sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
	advance[0] = false;
	param.block.x = step[0]; // reset block.x
      } else {
	advance[0] = true; // successfully advanced block.x
      }

      if (!advance[0]) {  // if failed to advance block.x, now try block.y
	param.block.y += step[1];

	if (param.block.y > in->X(4) || 
	    sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
	  advance[1] = false;
	  param.block.y = step[1]; // reset block.x
	} else {
	  advance[1] = true; // successfully advanced block.y
	}
      }

      if (advance[0] || advance[1]) {
	param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			   (in->X(4)+param.block.y-1) / param.block.y, 1);

	bool advance = true;
	if (!checkGrid(param)) advance = advanceBlockDim(param);
	return advance;
      } else {
	return false;
      }
    }

    unsigned int sharedBytesPerThread() const { return 0; }

  public:
    DomainWallDslashCuda(cudaColorSpinorField *out, const gFloat *gauge0, const gFloat *gauge1, 
			 const QudaReconstructType reconstruct, const cudaColorSpinorField *in,
			 const cudaColorSpinorField *x, const double mferm, 
			 const double a, const int dagger)
      : DslashCuda(out, in, x, reconstruct, dagger), gauge0(gauge0), 
	gauge1(gauge1), mferm(mferm), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x);
    }
    virtual ~DomainWallDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      DSLASH(domainWallDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	     (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
	     (sFloat*)in->V(), (float*)in->Norm(), mferm, (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a);
    }

    long long flops() const {
      long long flops = DslashCuda::flops();
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
      case EXTERIOR_KERNEL_ALL:
	break;
      case INTERIOR_KERNEL:
	int Ls = in->X(4);
	long long bulk = (Ls-2)*(in->VolumeCB()/Ls);
	long long wall = 2*(in->VolumeCB()/Ls);
	flops += 96ll*bulk + 120ll*wall;
	break;
      }
      return flops;
    }

    virtual long long bytes() const {
      bool isHalf = in->Precision() == sizeof(short) ? true : false;
      int spinor_bytes = 2 * in->Ncolor() * in->Nspin() * in->Precision() + (isHalf ? sizeof(float) : 0);
      long long bytes = DslashCuda::bytes();
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
      case EXTERIOR_KERNEL_ALL:
	break;
      case INTERIOR_KERNEL:
	bytes += 2 * spinor_bytes * in->VolumeCB();
	break;
      }
      return bytes;
    }
  };
#endif // GPU_DOMAIN_WALL_DIRAC

#include <dslash_policy.cuh>

  void domainWallDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
			    const cudaColorSpinorField *in, const int parity, const int dagger, 
			    const cudaColorSpinorField *x, const double &m_f, const double &k2, 
			    const int *commOverride, TimeProfile &profile, const QudaDslashPolicy &dslashPolicy)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

    dslashParam.parity = parity;

#ifdef GPU_DOMAIN_WALL_DIRAC
    //currently splitting in space-time is impelemented:
    int dirs = 4;
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
    for(int i = 0;i < dirs; i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }  

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge and spinor precision not supported");

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new DomainWallDslashCuda<double2,double2>(out, (double2*)gauge0, (double2*)gauge1, 
							 gauge.Reconstruct(), in, x, m_f, k2, dagger);
      regSize = sizeof(double);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new DomainWallDslashCuda<float4,float4>(out, (float4*)gauge0, (float4*)gauge1, 
						       gauge.Reconstruct(), in, x, m_f, k2, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new DomainWallDslashCuda<short4,short4>(out, (short4*)gauge0, (short4*)gauge1, 
						       gauge.Reconstruct(), in, x, m_f, k2, dagger);
    }

    // the parameters passed to dslashCuda must be 4-d volume and 3-d
    // faces because Ls is added as the y-dimension in thread space
    int ghostFace[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) ghostFace[i] = in->GhostFace()[i] / in->X(4);

#ifndef GPU_COMMS
    DslashPolicyImp* dslashImp = DslashFactory::create(dslashPolicy);
#else
    DslashPolicyImp* dslashImp = DslashFactory::create(QUDA_GPU_COMMS_DSLASH);
#endif

    (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), regSize, parity, dagger, in->Volume()/in->X(4), ghostFace, profile);
    delete dslashImp;

    delete dslash;
    unbindGaugeTex(gauge);

    checkCudaError();
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }

}
