#include "hip/hip_runtime.h"
#include <gauge_field_order.h>

namespace quda {

  /**
     Kernel argument struct
   */
  template <typename OutOrder, typename InOrder>
  struct CopyGaugeArg {
    OutOrder out;
    const InOrder in;
    int volume;
    int faceVolumeCB[QUDA_MAX_DIM];
    int nDim;
    int geometry;
    CopyGaugeArg(const OutOrder &out, const InOrder &in, int volume, 
		 const int *faceVolumeCB, int nDim, int geometry) 
      : out(out), in(in), volume(volume), nDim(nDim), geometry(geometry) {
      for (int d=0; d<nDim; d++) this->faceVolumeCB[d] = faceVolumeCB[d];
    }
  };

  /**
     Generic CPU gauge reordering and packing 
  */
  template <typename FloatOut, typename FloatIn, int length, typename OutOrder, typename InOrder>
  void copyGauge(CopyGaugeArg<OutOrder,InOrder> arg) {  
    typedef typename mapper<FloatIn>::type RegTypeIn;
    typedef typename mapper<FloatOut>::type RegTypeOut;

    for (int parity=0; parity<2; parity++) {

      for (int d=0; d<arg.geometry; d++) {
	for (int x=0; x<arg.volume/2; x++) {
	  RegTypeIn in[length];
	  RegTypeOut out[length];
	  arg.in.load(in, x, d, parity);
	  for (int i=0; i<length; i++) out[i] = in[i];
	  arg.out.save(out, x, d, parity);
	}
      }

    }
  }

  /**
     Check whether the field contains Nans
  */
  template <typename Float, int length, typename Arg>
  void checkNan(Arg arg) {  
    typedef typename mapper<Float>::type RegType;

    for (int parity=0; parity<2; parity++) {

      for (int d=0; d<arg.geometry; d++) {
	for (int x=0; x<arg.volume/2; x++) {
	  RegType u[length];
	  arg.in.load(u, x, d, parity);
	  for (int i=0; i<length; i++) 
	    if (isnan(u[i])) 
	      errorQuda("Nan detected at parity=%d, dir=%d, x=%d, i=%d", parity, d, x, i);
	}
      }

    }
  }


  /** 
      Generic CUDA gauge reordering and packing.  Adopts a similar form as
      the CPU version, using the same inlined functions.
  */
  template <typename FloatOut, typename FloatIn, int length, typename OutOrder, typename InOrder>
  __global__ void copyGaugeKernel(CopyGaugeArg<OutOrder,InOrder> arg) {  
    typedef typename mapper<FloatIn>::type RegTypeIn;
    typedef typename mapper<FloatOut>::type RegTypeOut;

    for (int parity=0; parity<2; parity++) {

      for (int d=0; d<arg.geometry; d++) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= arg.volume/2) return;

	RegTypeIn in[length];
	RegTypeOut out[length];
	arg.in.load(in, x, d, parity);
	for (int i=0; i<length; i++) out[i] = in[i];
	arg.out.save(out, x, d, parity);
      }
    }
  }

  /**
     Generic CPU gauge ghost reordering and packing 
  */
  template <typename FloatOut, typename FloatIn, int length, typename OutOrder, typename InOrder>
    void copyGhost(CopyGaugeArg<OutOrder,InOrder> arg) {  
    typedef typename mapper<FloatIn>::type RegTypeIn;
    typedef typename mapper<FloatOut>::type RegTypeOut;

    for (int parity=0; parity<2; parity++) {

      for (int d=0; d<arg.nDim; d++) {
	for (int x=0; x<arg.faceVolumeCB[d]; x++) {
	  RegTypeIn in[length];
	  RegTypeOut out[length];
	  arg.in.loadGhost(in, x, d, parity); // assumes we are loading 
	  for (int i=0; i<length; i++) out[i] = in[i];
	  arg.out.saveGhost(out, x, d, parity);
	}
      }

    }
  }

  /**
     Generic CUDA kernel for copying the ghost zone.  Adopts a similar form as
     the CPU version, using the same inlined functions.
  */
  template <typename FloatOut, typename FloatIn, int length, typename OutOrder, typename InOrder>
  __global__ void copyGhostKernel(CopyGaugeArg<OutOrder,InOrder> arg) {  
    typedef typename mapper<FloatIn>::type RegTypeIn;
    typedef typename mapper<FloatOut>::type RegTypeOut;

    int x = blockIdx.x * blockDim.x + threadIdx.x;

    for (int parity=0; parity<2; parity++) {
      for (int d=0; d<arg.nDim; d++) {
	if (x < arg.faceVolumeCB[d]) {
	  RegTypeIn in[length];
	  RegTypeOut out[length];
	  arg.in.loadGhost(in, x, d, parity); // assumes we are loading 
	  for (int i=0; i<length; i++) out[i] = in[i];
	  arg.out.saveGhost(out, x, d, parity);
	}
      }

    }
  }

  template <typename FloatOut, typename FloatIn, int length, typename OutOrder, typename InOrder, bool isGhost>
  class CopyGauge : Tunable {
    CopyGaugeArg<OutOrder,InOrder> arg;
    int size;
    const GaugeField &meta;

  private:
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0 ;}

    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return size; }

  public:
    CopyGauge(CopyGaugeArg<OutOrder,InOrder> &arg, const GaugeField &meta) : arg(arg), meta(meta) { 
      int faceMax = 0;
      for (int d=0; d<arg.nDim; d++) {
	faceMax = (arg.faceVolumeCB[d] > faceMax ) ? arg.faceVolumeCB[d] : faceMax;
      }
      size = isGhost ? faceMax : arg.volume/2;
      writeAuxString("out_stride=%d,in_stride=%d", arg.out.stride, arg.in.stride);
    }

    virtual ~CopyGauge() { ; }
  
    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (!isGhost) {
	copyGaugeKernel<FloatOut, FloatIn, length, OutOrder, InOrder> 
	  <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
      } else {
	copyGhostKernel<FloatOut, FloatIn, length, OutOrder, InOrder> 
	  <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    std::string paramString(const TuneParam &param) const { // Don't bother printing the grid dim.
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }

    long long flops() const { return 0; } 
    long long bytes() const { 
      int sites = 4*arg.volume/2;
      if (isGhost) {
	sites = 0;
	for (int d=0; d<4; d++) sites += arg.faceVolumeCB[d];
      }
      return 2 * sites * (  arg.in.Bytes() + arg.in.hasPhase*sizeof(FloatIn) 
			    + arg.out.Bytes() + arg.out.hasPhase*sizeof(FloatOut) ); 
    } 
  };


  template <typename FloatOut, typename FloatIn, int length, typename OutOrder, typename InOrder>
    void copyGauge(OutOrder outOrder, const InOrder inOrder, int volume, const int *faceVolumeCB, 
		   int nDim, int geometry, const GaugeField &out, QudaFieldLocation location, int type) {

    CopyGaugeArg<OutOrder,InOrder> arg(outOrder, inOrder, volume, faceVolumeCB, nDim, geometry);

    if (location == QUDA_CPU_FIELD_LOCATION) {
#ifdef HOST_DEBUG
      checkNan<FloatIn, length>(arg);
#endif

      if (type == 0 || type == 2) {
	copyGauge<FloatOut, FloatIn, length>(arg);
      }
#ifdef MULTI_GPU // only copy the ghost zone if doing multi-gpu
      if (type == 0 || type == 1) {
	if (geometry == QUDA_VECTOR_GEOMETRY) copyGhost<FloatOut, FloatIn, length>(arg);
	//else warningQuda("Cannot copy for %d geometry gauge field", geometry);
      }
#endif
    } else if (location == QUDA_CUDA_FIELD_LOCATION) {
      // first copy body
      if (type == 0 || type == 2) {
	CopyGauge<FloatOut, FloatIn, length, OutOrder, InOrder, 0> gaugeCopier(arg, out);
	gaugeCopier.apply(0);
      }
#ifdef MULTI_GPU
      if (type == 0 || type == 1) {
	if (geometry == QUDA_VECTOR_GEOMETRY) {
	  // now copy ghost
	  CopyGauge<FloatOut, FloatIn, length, OutOrder, InOrder, 1> ghostCopier(arg, out);
	  ghostCopier.apply(0);
	} else {
	  //warningQuda("Cannot copy for %d geometry gauge field", geometry);
	}
      }
#endif
    } else {
      errorQuda("Undefined field location %d for copyGauge", location);
    }

  }
  
  template <typename FloatOut, typename FloatIn, int length, typename InOrder>
  void copyGauge(const InOrder &inOrder, GaugeField &out, QudaFieldLocation location, 
		 FloatOut *Out, FloatOut **outGhost, int type) {
    int faceVolumeCB[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) faceVolumeCB[i] = out.SurfaceCB(i) * out.Nface(); 
    if (out.isNative()) {
      if (out.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	if (typeid(FloatOut)==typeid(short) && out.LinkType() == QUDA_ASQTAD_FAT_LINKS) {
	  copyGauge<short,FloatIn,length>
	    (FloatNOrder<short,length,2,19>(out, (short*)Out, (short**)outGhost), inOrder,
	     out.Volume(), faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
	} else {
	  typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_NO>::type G;
	  copyGauge<FloatOut,FloatIn,length>
	    (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	     out.Ndim(), out.Geometry(), out, location, type);
	}
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_12) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_12>::type G;
	copyGauge<FloatOut,FloatIn,length>
	  (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_8) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_8>::type G;
	copyGauge<FloatOut,FloatIn,length> 
	  (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
#ifdef GPU_STAGGERED_DIRAC
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_13) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_13>::type G;
        copyGauge<FloatOut,FloatIn,length>
	  (G(out, Out, outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(),  out.Geometry(), out, location, type);
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_9) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_9>::type G;
        copyGauge<FloatOut,FloatIn,length>
	  (G(out, Out, outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
#endif
      } else {
	errorQuda("Reconstruction %d and order %d not supported", out.Reconstruct(), out.Order());
      }
    } else if (out.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(QDPOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(), 
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(QDPJITOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(CPSOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(MILCOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(BQCDOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(TIFROrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", out.Order());
    }

  }

  template <typename FloatOut, typename FloatIn, int length>
    void copyGauge(GaugeField &out, const GaugeField &in, QudaFieldLocation location, 
		   FloatOut *Out, FloatIn *In, FloatOut **outGhost, FloatIn **inGhost, int type) {

    // reconstruction only supported on FloatN fields currently
    if (in.isNative()) {      
      if (in.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	if (typeid(FloatIn)==typeid(short) && in.LinkType() == QUDA_ASQTAD_FAT_LINKS) {
	  copyGauge<FloatOut,short,length> (FloatNOrder<short,length,2,19>
					    (in,(short*)In,(short**)inGhost),
					    out, location, Out, outGhost, type);
	} else {
	  typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_NO>::type G;
	  copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
	}
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_12) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_12>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_8) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_8>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
#ifdef GPU_STAGGERED_DIRAC
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_13) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_13>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_9) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_9>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
#endif
      } else {
	errorQuda("Reconstruction %d and order %d not supported", in.Reconstruct(), in.Order());
      }
    } else if (in.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(QDPOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(QDPJITOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(CPSOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(MILCOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(BQCDOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(TIFROrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", in.Order());
    }

  }

  void checkMomOrder(const GaugeField &u);

  template <typename FloatOut, typename FloatIn>
  void copyGauge(GaugeField &out, const GaugeField &in, QudaFieldLocation location, FloatOut *Out, 
		 FloatIn *In, FloatOut **outGhost, FloatIn **inGhost, int type) {

    if (in.Ncolor() != 3 && out.Ncolor() != 3) {
      errorQuda("Unsupported number of colors; out.Nc=%d, in.Nc=%d", out.Ncolor(), in.Ncolor());
    }
    
    if (out.Geometry() != in.Geometry()) {
      errorQuda("Field geometries %d %d do not match", out.Geometry(), in.Geometry());
    }

    if (in.LinkType() != QUDA_ASQTAD_MOM_LINKS && out.LinkType() != QUDA_ASQTAD_MOM_LINKS) {
      // we are doing gauge field packing
      copyGauge<FloatOut,FloatIn,18>(out, in, location, Out, In, outGhost, inGhost, type);
    } else {
      if (location != QUDA_CPU_FIELD_LOCATION) errorQuda("Location %d not supported", location);
      if (out.Geometry() != QUDA_VECTOR_GEOMETRY) errorQuda("Unsupported geometry %d", out.Geometry());

      checkMomOrder(in);
      checkMomOrder(out);
    
      int faceVolumeCB[QUDA_MAX_DIM];
      for (int d=0; d<in.Ndim(); d++) faceVolumeCB[d] = in.SurfaceCB(d) * in.Nface();

      // momentum only currently supported on MILC (10), TIFR (18) and Float2 (10) fields currently
	if (out.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    CopyGaugeArg<FloatNOrder<FloatOut,10,2,10>, FloatNOrder<FloatIn,10,2,10> >
	      arg(FloatNOrder<FloatOut,10,2,10>(out, Out), 
		  FloatNOrder<FloatIn,10,2,10>(in, In), in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
	  } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {
#ifdef BUILD_MILC_INTERFACE
	    CopyGaugeArg<FloatNOrder<FloatOut,10,2,10>, MILCOrder<FloatIn,10> >
	      arg(FloatNOrder<FloatOut,10,2,10>(out, Out), MILCOrder<FloatIn,10>(in, In), 
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
#else
	    errorQuda("MILC interface has not been built\n");
#endif
	    
	  } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {
#ifdef BUILD_TIFR_INTERFACE
	    CopyGaugeArg<FloatNOrder<FloatOut,18,2,11>, TIFROrder<FloatIn,18> >
	      arg(FloatNOrder<FloatOut,18,2,11>(out, Out), TIFROrder<FloatIn,18>(in, In), 
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,18>(arg);
#else
	    errorQuda("TIFR interface has not been built\n");
#endif
	    
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
	} else if (out.Order() == QUDA_MILC_GAUGE_ORDER) {
#ifdef BUILD_MILC_INTERFACE
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    CopyGaugeArg<MILCOrder<FloatOut,10>, FloatNOrder<FloatIn,10,2,10> >
	      arg(MILCOrder<FloatOut,10>(out, Out), FloatNOrder<FloatIn,10,2,10>(in, In),
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
	  } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {
	    CopyGaugeArg<MILCOrder<FloatOut,10>, MILCOrder<FloatIn,10> >
	      arg(MILCOrder<FloatOut,10>(out, Out), MILCOrder<FloatIn,10>(in, In),
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
#else
	  errorQuda("MILC interface has not been built\n");
#endif
	  
	} else if (out.Order() == QUDA_TIFR_GAUGE_ORDER) {
#ifdef BUILD_TIFR_INTERFACE
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    // FIX ME - 11 is a misnomer to avoid confusion in template instantiation
	    CopyGaugeArg<TIFROrder<FloatOut,18>, FloatNOrder<FloatIn,18,2,11> >
	      arg(TIFROrder<FloatOut,18>(out, Out), FloatNOrder<FloatIn,18,2,11>(in, In),
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,18>(arg);
	  } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {
	    CopyGaugeArg<TIFROrder<FloatOut,18>, TIFROrder<FloatIn,18> >
	      arg(TIFROrder<FloatOut,18>(out, Out), TIFROrder<FloatIn,18>(in, In),
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
#else
	  errorQuda("TIFR interface has not been built\n");
#endif
	} else {
	  errorQuda("Gauge field orders %d not supported", out.Order());
	}
    }
  }


} // namespace quda
