#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <staggered_oprod.h>

#include <tune_quda.h>
#include <quda_internal.h>
#include <gauge_field_order.h>
#include <quda_matrix.h>

namespace quda {

#ifdef GPU_STAGGERED_OPROD

  namespace { // anonymous
#include <texture.h>
  }

  template<int N>
    void createEventArray(hipEvent_t (&event)[N], unsigned int flags=hipEventDefault)
    {
      for(int i=0; i<N; ++i)
        hipEventCreate(&event[i],flags);
      return;
    }

  template<int N>
    void destroyEventArray(hipEvent_t (&event)[N])
    {
      for(int i=0; i<N; ++i)
        hipEventDestroy(event[i]);
    }


  static hipEvent_t packEnd;
  static hipEvent_t gatherEnd[4];
  static hipEvent_t scatterEnd[4];
  static hipEvent_t oprodStart;
  static hipEvent_t oprodEnd;


  void createStaggeredOprodEvents(){
#ifdef MULTI_GPU
    hipEventCreate(&packEnd, hipEventDisableTiming);
    createEventArray(gatherEnd, hipEventDisableTiming);
    createEventArray(scatterEnd, hipEventDisableTiming);
#endif
    hipEventCreate(&oprodStart, hipEventDisableTiming);
    hipEventCreate(&oprodEnd, hipEventDisableTiming);
    return;
  }

  void destroyStaggeredOprodEvents(){
#ifdef MULTI_GPU
    destroyEventArray(gatherEnd);
    destroyEventArray(scatterEnd);
    hipEventDestroy(packEnd);
#endif
    hipEventDestroy(oprodStart);
    hipEventDestroy(oprodEnd);
    return;
  }


  enum KernelType {OPROD_INTERIOR_KERNEL, OPROD_EXTERIOR_KERNEL};

  template<typename Complex, typename Output, typename InputA, typename InputB>
    struct StaggeredOprodArg {
      unsigned int length;
      int X[4];
      unsigned int parity;
      unsigned int dir;
      unsigned int ghostOffset;
      unsigned int displacement;
      KernelType kernelType;
      bool partitioned[4];
      InputA inA;
      InputB inB;
      Output outA;
      Output outB;
      cudaGaugeField& outFieldA;
      cudaGaugeField& outFieldB;
      typename RealTypeId<Complex>::Type coeff[2];
      
      StaggeredOprodArg(const unsigned int length,
          const int X[4],
          const unsigned int parity,
          const unsigned int dir,
          const unsigned int ghostOffset,
          const unsigned int displacement,   
          const KernelType& kernelType, 
          const double coeff[2],
          InputA& inA,
          InputB& inB,
          Output& outA,
          Output& outB,
          cudaGaugeField& outFieldA,
          cudaGaugeField& outFieldB) : length(length), parity(parity), ghostOffset(ghostOffset), 
      displacement(displacement), kernelType(kernelType), inA(inA), inB(inB), outA(outA), outB(outB),
      outFieldA(outFieldA), outFieldB(outFieldB)
      {
        this->coeff[0] = coeff[0];
        this->coeff[1] = coeff[1];
        for(int i=0; i<4; ++i) this->X[i] = X[i];
        for(int i=0; i<4; ++i) this->partitioned[i] = commDimPartitioned(i) ? true : false;
      }
  };

  enum IndexType {
    EVEN_X = 0,
    EVEN_Y = 1,
    EVEN_Z = 2,
    EVEN_T = 3
  };

  template <IndexType idxType>
    static __device__ __forceinline__ void coordsFromIndex(int& idx, int c[4],  
        const unsigned int cb_idx, const unsigned int parity, const int X[4])
    {
      const int &LX = X[0];
      const int &LY = X[1];
      const int &LZ = X[2];
      const int XYZ = X[2]*X[1]*X[0];
      const int XY = X[1]*X[0];

      idx = 2*cb_idx;

      int x, y, z, t;

      if (idxType == EVEN_X /*!(LX & 1)*/) { // X even
        //   t = idx / XYZ;
        //   z = (idx / XY) % Z;
        //   y = (idx / X) % Y;
        //   idx += (parity + t + z + y) & 1;
        //   x = idx % X;
        // equivalent to the above, but with fewer divisions/mods:
        int aux1 = idx / LX;
        x = idx - aux1 * LX;
        int aux2 = aux1 / LY;
        y = aux1 - aux2 * LY;
        t = aux2 / LZ;
        z = aux2 - t * LZ;
        aux1 = (parity + t + z + y) & 1;
        x += aux1;
        idx += aux1;
      } else if (idxType == EVEN_Y /*!(LY & 1)*/) { // Y even
        t = idx / XYZ;
        z = (idx / XY) % LZ;
        idx += (parity + t + z) & 1;
        y = (idx / LX) % LY;
        x = idx % LX;
      } else if (idxType == EVEN_Z /*!(LZ & 1)*/) { // Z even
        t = idx / XYZ;
        idx += (parity + t) & 1;
        z = (idx / XY) % LZ;
        y = (idx / LX) % LY;
        x = idx % LX;
      } else {
        idx += parity;
        t = idx / XYZ;
        z = (idx / XY) % LZ;
        y = (idx / LX) % LY;
        x = idx % LX;
      }

      c[0] = x;
      c[1] = y;
      c[2] = z;
      c[3] = t;
    }




  // Get the  coordinates for the exterior kernels
  template<int Nspin>
    __device__ void coordsFromIndex(int x[4], const unsigned int cb_idx, const int X[4], const unsigned int dir, const int displacement, const unsigned int parity)
    {

      if(Nspin == 1){
        unsigned int Xh[2] = {X[0]/2, X[1]/2};
        switch(dir){
          case 0:
            x[2] = cb_idx/Xh[1] % X[2];
            x[3] = cb_idx/(Xh[1]*X[2]) % X[3];
            x[0] = cb_idx/(Xh[1]*X[2]*X[3]);
            x[0] += (X[0] - displacement);
            x[1] = 2*(cb_idx % Xh[1]) + ((x[0]+x[2]+x[3]+parity)&1);
            break;

          case 1:
            x[2] = cb_idx/Xh[0] % X[2];
            x[3] = cb_idx/(Xh[0]*X[2]) % X[3];
            x[1] = cb_idx/(Xh[0]*X[2]*X[3]);
            x[1] += (X[1] - displacement);
            x[0] = 2*(cb_idx % Xh[0]) + ((x[1]+x[2]+x[3]+parity)&1);

            break;

          case 2:
            x[1] = cb_idx/Xh[0] % X[1];
            x[3] = cb_idx/(Xh[0]*X[1]) % X[3];
            x[2] = cb_idx/(Xh[0]*X[1]*X[3]);
            x[2] += (X[2] - displacement);
            x[0] = 2*(cb_idx % Xh[0]) + ((x[1]+x[2]+x[3]+parity)&1);

            break;

          case 3:
            x[1] = cb_idx/Xh[0] % X[1];
            x[2] = cb_idx/(Xh[0]*X[1]) % X[2];
            x[3] = cb_idx/(Xh[0]*X[1]*X[2]);
            x[3] += (X[3] - displacement);
            x[0] = 2*(cb_idx % Xh[0]) + ((x[1]+x[2]+x[3]+parity)&1);

            break;
        }
      }else if(Nspin == 3){
        // currently unsupported
      }
      return;
    }


  template<int Nspin, int Nface> 
    __device__  int ghostIndexFromCoords(const int x[4], const int X[4], const unsigned int dir, const int shift){
      return 0;
    }



  template<>
    __device__  int ghostIndexFromCoords<1,3>(
        const int x[4],
        const int X[4], 
        unsigned int dir, 
        const int shift)
    {
      /*
	FIXME the below can be extremely simplified.  Each of these
	essentially return cb_idx, and for the case the positive
	shift, there is an offset of size surfaceCB[dim]*nFace to
	start at the forward face instead of the backward face.

	The factors of 3 confused me when first reading this.  One
	factor comes from 3 colors, the other comes from the fact that
	the ghost zone is depth nFace=3 since this is for improved
	staggered fermions.  Float2 indexing is assumed here.
      */

      int ghost_idx;
      if(shift > 0){
        if((x[dir] + shift) >= X[dir]){
          switch(dir){
            case 0:
              ghost_idx = (3*3 + (x[0]-X[0]+shift))*(X[3]*X[2]*X[1])/2 + ((x[3]*X[2] + x[2])*X[1] + x[1])/2;
              break;          
            case 1:
              ghost_idx = (3*3 + (x[1]-X[1]+shift))*(X[3]*X[2]*X[0])/2 + (x[3]*X[2]*X[0] + x[2]*X[0] + x[0])/2;
              break;
            case 2:
              ghost_idx = (3*3 + (x[2]-X[2]+shift))*(X[3]*X[1]*X[0])/2 + (x[3]*X[1]*X[0] + x[1]*X[0] + x[0])/2;
              break;
            case 3:
              ghost_idx = (3*3 + (x[3]-X[3]+shift))*(X[2]*X[1]*X[0])/2 + (x[2]*X[1]*X[0] + x[1]*X[0] + x[0])/2;
              break;
            default:
              break;
          } // switch
        } // x[dir] + shift[dir] >= X[dir]
      }else{ // shift < 0
        if(static_cast<int>(x[dir]) + shift < 0){
          switch(dir){
            case 0:
              ghost_idx = (3 + shift)*(X[3]*X[2]*X[1])/2 + ((x[3]*X[2] + x[2])*X[1] + x[1])/2;
              break;
            case 1:
              ghost_idx = (3 + shift)*(X[3]*X[2]*X[0])/2 + ((x[3]*X[2] + x[2])*X[0] + x[0])/2;
              break;
            case 2:
              ghost_idx = (3 + shift)*(X[3]*X[1]*X[0])/2 + ((x[3]*X[1] + x[1])*X[0]  + x[0])/2;
              break;
            case 3:
              ghost_idx = (3 + shift)*(X[2]*X[1]*X[0])/2 + ((x[2]*X[1] + x[1])*X[0] + x[0])/2;
              break;
          } // switch(dir)
        }
      } // shift < 0

      return ghost_idx;
    }




  __device__ __forceinline__
    int neighborIndex(const unsigned int& cb_idx, const int shift[4],  const bool partitioned[4], const unsigned int& parity, 
        const int X[4]){

      int  full_idx;
      int x[4]; 


      coordsFromIndex<EVEN_X>(full_idx, x, cb_idx, parity, X);

#ifdef MULTI_GPU
      for(int dim = 0; dim<4; ++dim){
        if(partitioned[dim])
          if( (x[dim]+shift[dim])<0 || (x[dim]+shift[dim])>=X[dim]) return -1;
      }
#endif

      for(int dim=0; dim<4; ++dim){
        x[dim] = shift[dim] ? (x[dim]+shift[dim] + X[dim]) % X[dim] : x[dim];
      }
      return  (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    }



  template<typename Complex, typename Output, typename InputA, typename InputB>
  __global__ void interiorOprodKernel(StaggeredOprodArg<Complex, Output, InputA, InputB> arg)
    {
      unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
      const unsigned int gridSize = gridDim.x*blockDim.x;

      typedef typename RealTypeId<Complex>::Type real;
      Complex x[3];
      Complex y[3];
      Complex z[3];
      Matrix<Complex,3> result;
      Matrix<Complex,3> tempA, tempB; // input


      while(idx<arg.length){
        arg.inA.load(x, idx);
        for(int dir=0; dir<4; ++dir){
          int shift[4] = {0,0,0,0};
          shift[dir] = 1;
          const int first_nbr_idx = neighborIndex(idx, shift, arg.partitioned, arg.parity, arg.X);
          if(first_nbr_idx >= 0){
            arg.inB.load(y, first_nbr_idx);
            outerProd(y,x,&result);
            arg.outA.load(reinterpret_cast<real*>(tempA.data), idx, dir, arg.parity); 
            result = tempA + result*arg.coeff[0];
            arg.outA.save(reinterpret_cast<real*>(result.data), idx, dir, arg.parity); 

            shift[dir] = 3;
            const int third_nbr_idx = neighborIndex(idx, shift, arg.partitioned, arg.parity, arg.X);
            if(third_nbr_idx >= 0){
              arg.inB.load(z, third_nbr_idx);
              outerProd(z, x, &result);
              arg.outB.load(reinterpret_cast<real*>(tempB.data), idx, dir, arg.parity); 
              result = tempB + result*arg.coeff[1];
              arg.outB.save(reinterpret_cast<real*>(result.data), idx, dir, arg.parity); 
            }
          }
        } // dir
        idx += gridSize;
      }
      return;
    } // interiorOprodKernel



  template<typename Complex, typename Output, typename InputA, typename InputB> 
  __global__ void exteriorOprodKernel(StaggeredOprodArg<Complex, Output, InputA, InputB> arg)
    {
      unsigned int cb_idx = blockIdx.x*blockDim.x + threadIdx.x;
      const unsigned int gridSize = gridDim.x*blockDim.x;

      Complex a[3];
      Complex b[3];
      Matrix<Complex,3> result;
      Matrix<Complex,3> inmatrix; // input
      typedef typename RealTypeId<Complex>::Type real;


      Output& out = (arg.displacement == 1) ? arg.outA : arg.outB;
      real coeff = (arg.displacement == 1) ? arg.coeff[0] : arg.coeff[1];

      int x[4];
      while(cb_idx<arg.length){
        coordsFromIndex<1>(x, cb_idx, arg.X, arg.dir, arg.displacement, arg.parity); 
        const unsigned int bulk_cb_idx = ((((x[3]*arg.X[2] + x[2])*arg.X[1] + x[1])*arg.X[0] + x[0]) >> 1);

        out.load(reinterpret_cast<real*>(inmatrix.data), bulk_cb_idx, arg.dir, arg.parity); 
        arg.inA.load(a, bulk_cb_idx);

        const unsigned int ghost_idx = arg.ghostOffset + ghostIndexFromCoords<1,3>(x, arg.X, arg.dir, arg.displacement);
        arg.inB.loadGhost(b, ghost_idx, arg.dir);

        outerProd(b,a,&result);
        result = inmatrix + result*coeff; 
        out.save(reinterpret_cast<real*>(result.data), bulk_cb_idx, arg.dir, arg.parity); 

        cb_idx += gridSize;
      }
      return;
    }



  template<typename Complex, typename Output, typename InputA, typename InputB> 
    class StaggeredOprodField : public Tunable {

      private:
        StaggeredOprodArg<Complex,Output,InputA,InputB> arg;
        const GaugeField &meta;
        QudaFieldLocation location; // location of the lattice fields

        unsigned int sharedBytesPerThread() const { return 0; }
        unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

        unsigned int minThreads() const { return arg.outA.volumeCB; }
        bool tunedGridDim() const { return false; }

      public:
      StaggeredOprodField(const StaggeredOprodArg<Complex,Output,InputA,InputB> &arg,
			  const GaugeField &meta, QudaFieldLocation location)
	: arg(arg), meta(meta), location(location) {
   	  writeAuxString("threads=%d,prec=%lu,stride=%d",arg.length,sizeof(Complex)/2,arg.inA.Stride());
	  // this sets the communications pattern for the packing kernel
	  int comms[QUDA_MAX_DIM] = { commDimPartitioned(0), commDimPartitioned(1), commDimPartitioned(2), commDimPartitioned(3) };
	  setPackComms(comms);
	} 

       virtual ~StaggeredOprodField() {}

       void set(const StaggeredOprodArg<Complex,Output,InputA,InputB> &arg, QudaFieldLocation location){
          // This is a hack. Need to change this!
          this->arg.dir = arg.dir;
          this->arg.length = arg.length;
          this->arg.ghostOffset = arg.ghostOffset;
          this->arg.kernelType = arg.kernelType;
          this->location = location;
        } // set

        void apply(const hipStream_t &stream){
          if(location == QUDA_CUDA_FIELD_LOCATION){
            // Disable tuning for the time being
            TuneParam tp = tuneLaunch(*this, QUDA_TUNE_NO, getVerbosity());
            //if(arg.kernelType == OPROD_INTERIOR_KERNEL){
            interiorOprodKernel<<<tp.grid,tp.block,tp.shared_bytes, stream>>>(arg);
            //  dim3 blockDim(128, 1, 1);
            //  const int gridSize = (arg.length + (blockDim.x-1))/blockDim.x;
            //  dim3 gridDim(gridSize, 1, 1);               
            //  interiorOprodKernel<<<gridDim,blockDim,0, stream>>>(arg);
           // }else if(arg.kernelType == OPROD_EXTERIOR_KERNEL){
           //   const unsigned int volume = arg.X[0]*arg.X[1]*arg.X[2]*arg.X[3];
           //   exteriorOprodKernel<<<tp.grid,tp.block,tp.shared_bytes, stream>>>(arg);
           // }else{
           //   errorQuda("Kernel type not supported\n");
           // }
          }else{ // run the CPU code
            errorQuda("No CPU support for staggered outer-product calculation\n");
          }
        } // apply

        void preTune(){
          this->arg.outFieldA.backup();
          this->arg.outFieldB.backup();
        }
        void postTune(){
          this->arg.outFieldA.restore();
          this->arg.outFieldB.restore();
        }
  
        long long flops() const {
          return 0; // fix this
        }

        long long bytes() const { 
	  return 0; // fix this
        }

        TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux);}
  }; // StaggeredOprodField

  template<typename Complex, typename Output, typename InputA, typename InputB>
    void computeStaggeredOprodCuda(Output outA, Output outB, cudaGaugeField& outFieldA, cudaGaugeField& outFieldB, InputA& inA, InputB& inB, cudaColorSpinorField& src, 
        FaceBuffer& faceBuffer,  const unsigned int parity, const int faceVolumeCB[4], 
        const unsigned int ghostOffset[4], const double coeff[2])
    {

      hipEventRecord(oprodStart, streams[Nstream-1]);


      const int dim[4] = {src.X(0)*2, src.X(1), src.X(2), src.X(3)};
      // Create the arguments for the interior kernel 
      StaggeredOprodArg<Complex,Output,InputA,InputB> arg(outA.volumeCB, dim, parity, 0, 0, 1, OPROD_INTERIOR_KERNEL, coeff, inA, inB, outA, outB, outFieldA, 
							  outFieldB);


      StaggeredOprodField<Complex,Output,InputA,InputB> oprod(arg, outFieldA, QUDA_CUDA_FIELD_LOCATION);

#ifdef MULTI_GPU
      bool pack=false;
      for(int i=3; i>=0; i--){
        if(commDimPartitioned(i) && (i!=3 || getKernelPackT())){
          pack = true;
          break;
        }
      } // i=3,..,0

      // source, dir(+/-1), parity, dagger, stream_ptr
      // packing is all done in streams[Nstream-1]
      // always call pack since this also sets the stream pointer even if not packing
      //faceBuffer.pack(src, -1, 1-parity, 0, streams); 
      faceBuffer.pack(src, 1-parity, 0, streams);  // FIXME work around since uni-direction packing is broken
      if(pack){
        hipEventRecord(packEnd, streams[Nstream-1]);
      }

      for(int i=3; i>=0; i--){
        if(commDimPartitioned(i)){

          hipEvent_t &event = (i!=3 || getKernelPackT()) ? packEnd : oprodStart;
          hipStreamWaitEvent(streams[2*i], event, 0); // wait in stream 2*i for event to complete
      

          // Initialize the host transfer from the source spinor
          faceBuffer.gather(src, false, 2*i); 
          // record the end of the gathering 
          hipEventRecord(gatherEnd[i], streams[2*i]);
        } // comDim(i)
      } // i=3,..,0
#endif
      oprod.apply(streams[Nstream-1]); 

/*
      dim3 blockDim(128, 1, 1);
      const int gridSize = (arg.length + (blockDim.x-1))/blockDim.x;
      dim3 gridDim(gridSize, 1, 1);
      interiorOprodKernel<<<gridDim,blockDim,0,streams[Nstream-1]>>>(arg);

*/



#ifdef MULTI_GPU
      // compute gather completed 
      int gatherCompleted[5];
      int commsCompleted[5];
      int oprodCompleted[4];

      for(int i=0; i<4; ++i){
        gatherCompleted[i] = commsCompleted[i] = oprodCompleted[i] = 0;
      }
      gatherCompleted[4] = commsCompleted[4] = 1;

      // initialize commDimTotal 
      int commDimTotal = 0;
      for(int i=0; i<4; ++i){
        commDimTotal += commDimPartitioned(i);
      }
      commDimTotal *= 2;

      // initialize previousDir
      int previousDir[4];
      for(int i=3; i>=0; i--){
        if(commDimPartitioned(i)){
          int prev = 4;
          for(int j=3; j>i; j--){
            if(commDimPartitioned(j)){
              prev = j;
            }
          }
          previousDir[i] = prev;
        }
      } // set previous directions


      if(commDimTotal){
        arg.kernelType = OPROD_EXTERIOR_KERNEL;
        unsigned int completeSum=0;
        while(completeSum < commDimTotal){

          for(int i=3; i>=0; i--){
            if(!commDimPartitioned(i)) continue;

            if(!gatherCompleted[i] && gatherCompleted[previousDir[i]]){
              hipError_t event_test = hipEventQuery(gatherEnd[i]);

              if(event_test == hipSuccess){
                gatherCompleted[i] = 1;
                completeSum++;
                faceBuffer.commsStart(2*i);
              }
            }

            // Query if comms has finished 
            if(!commsCompleted[i] && commsCompleted[previousDir[i]] && gatherCompleted[i]){
              int comms_test = faceBuffer.commsQuery(2*i);
              if(comms_test){
                commsCompleted[i] = 1;
                completeSum++;
                faceBuffer.scatter(src, false, 2*i);
              }
            }

            // enqueue the boundary oprod kernel as soon as the scatters have been enqueud
            if(!oprodCompleted[i] && commsCompleted[i]){
              hipEventRecord(scatterEnd[i], streams[2*i]);
              hipStreamWaitEvent(streams[Nstream-1], scatterEnd[i],0);

              arg.dir = i;
              arg.ghostOffset = ghostOffset[i];
              const unsigned int volume = arg.X[0]*arg.X[1]*arg.X[2]*arg.X[3];
              // First, do the one hop term
              {

                arg.length = faceVolumeCB[i];
                arg.displacement = 1;
                dim3 blockDim(128, 1, 1);
                const int gridSize = (arg.length + (blockDim.x-1))/blockDim.x;
                dim3 gridDim(gridSize, 1, 1);               
                exteriorOprodKernel<<<gridDim, blockDim, 0, streams[Nstream-1]>>>(arg);              
              }
              // Now do the 3 hop term - Try putting this in a separate stream
              {

                arg.displacement = 3;                      
                arg.length = arg.displacement*faceVolumeCB[i];
                dim3 blockDim(128, 1, 1);
                const int gridSize = (arg.length + (blockDim.x-1))/blockDim.x;
                dim3 gridDim(gridSize, 1, 1);               
		exteriorOprodKernel<<<gridDim, blockDim, 0, streams[Nstream-1]>>>(arg);
              } 

              oprodCompleted[i] = 1;
            }

          } // i=3,..,0 
        } // completeSum < commDimTotal
      } // if commDimTotal
#endif
    } // computeStaggeredOprodCuda

#endif // GPU_STAGGERED_OPROD

  // At the moment, I pass an instance of FaceBuffer in. 
  // Soon, faceBuffer will be subsumed into cudaColorSpinorField.
  void computeStaggeredOprod(cudaGaugeField& outA, cudaGaugeField& outB, 
      cudaColorSpinorField& inEven,  
      cudaColorSpinorField& inOdd,
      FaceBuffer& faceBuffer,
      const unsigned int parity, const double coeff[2])
  {

#ifdef GPU_STAGGERED_OPROD 

    if(outA.Order() != QUDA_FLOAT2_GAUGE_ORDER)
      errorQuda("Unsupported output ordering: %d\n", outA.Order());    

    if(outB.Order() != QUDA_FLOAT2_GAUGE_ORDER)
      errorQuda("Unsupported output ordering: %d\n", outB.Order());    

    unsigned int ghostOffset[4] = {0,0,0,0};
#ifdef MULTI_GPU
    const unsigned int Npad = inEven.Ncolor()*inEven.Nspin()*2/inEven.FieldOrder();
    for(int dir=0; dir<4; ++dir){
      ghostOffset[dir] = Npad*(inEven.GhostOffset(dir) + inEven.Stride()); 
    }
#endif

    if(inEven.Precision() != outA.Precision()) errorQuda("Mixed precision not supported: %d %d\n", inEven.Precision(), outA.Precision());

    cudaColorSpinorField& inA = (parity&1) ? inOdd : inEven;
    cudaColorSpinorField& inB = (parity&1) ? inEven : inOdd;

    if(inEven.Precision() == QUDA_DOUBLE_PRECISION){
      // 3 sets number of faces in ghost_stride - we need to set this to 1 for naive staggered
      Spinor<double2, double2, double2, 3, 0, 0> spinorA(inA, 3); 
      Spinor<double2, double2, double2, 3, 0, 1> spinorB(inB, 3);
      computeStaggeredOprodCuda<double2>(FloatNOrder<double, 18, 2, 18>(outA), FloatNOrder<double, 18, 2, 18>(outB), 
          outA, outB, 
          spinorA, spinorB, inB, faceBuffer, parity, inB.GhostFace(), ghostOffset, coeff);
    }else if(inEven.Precision() == QUDA_SINGLE_PRECISION){

      Spinor<float2, float2, float2, 3, 0, 0> spinorA(inA, 3);
      Spinor<float2, float2, float2, 3, 0, 1> spinorB(inB, 3);
      computeStaggeredOprodCuda<float2>(FloatNOrder<float, 18, 2, 18>(outA), FloatNOrder<float, 18, 2, 18>(outB), 
          outA, outB,
          spinorA, spinorB, inB, faceBuffer, parity, inB.GhostFace(), ghostOffset, coeff);
    } else {
      errorQuda("Unsupported precision: %d\n", inEven.Precision());
    }

#else // GPU_STAGGERED_OPROD not defined
   errorQuda("Staggered Outer Product has not been built!"); 
#endif

    return;
  } // computeStaggeredOprod



} // namespace quda
