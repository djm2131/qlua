#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <typeinfo>

#include <color_spinor_field.h>
#include <blas_quda.h>

#include <string.h>
#include <iostream>
#include <misc_helpers.h>
#include <face_quda.h>
#include <dslash_quda.h>

#ifdef DEVICE_PACK
#define REORDER_LOCATION QUDA_CUDA_FIELD_LOCATION
#else
#define REORDER_LOCATION QUDA_CPU_FIELD_LOCATION
#endif

int zeroCopy = 0;

namespace quda {

  int cudaColorSpinorField::bufferIndex = 0;
  int cudaColorSpinorField::initGhostFaceBuffer = 0;
  void* cudaColorSpinorField::ghostFaceBuffer[2]; //gpu memory
  void* cudaColorSpinorField::fwdGhostFaceBuffer[2][QUDA_MAX_DIM]; //pointers to ghostFaceBuffer
  void* cudaColorSpinorField::backGhostFaceBuffer[2][QUDA_MAX_DIM]; //pointers to ghostFaceBuffer
  size_t cudaColorSpinorField::ghostFaceBytes = 0;

  /*cudaColorSpinorField::cudaColorSpinorField() : 
    ColorSpinorField(), v(0), norm(0), alloc(false), init(false) {

    }*/

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorParam &param) : 
    ColorSpinorField(param), alloc(false), init(true), texInit(false), 
    initComms(false), bufferMessageHandler(0), nFaceComms(0) {

    // this must come before create
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      v = param.v;
      norm = param.norm;
    }

    create(param.create);

    if  (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // dp nothing
    } else if (param.create == QUDA_COPY_FIELD_CREATE){
      errorQuda("not implemented");
    }
    checkCudaError();
  }

  cudaColorSpinorField::cudaColorSpinorField(const cudaColorSpinorField &src) : 
    ColorSpinorField(src), alloc(false), init(true), texInit(false), 
    initComms(false), bufferMessageHandler(0), nFaceComms(0) {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  // creates a copy of src, any differences defined in param
  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src, 
					     const ColorSpinorParam &param) :
    ColorSpinorField(src), alloc(false), init(true), texInit(false), 
    initComms(false), bufferMessageHandler(0), nFaceComms(0) {  

    // can only overide if we are not using a reference or parity special case
    if (param.create != QUDA_REFERENCE_FIELD_CREATE || 
	(param.create == QUDA_REFERENCE_FIELD_CREATE && 
	 src.SiteSubset() == QUDA_FULL_SITE_SUBSET && 
	 param.siteSubset == QUDA_PARITY_SITE_SUBSET && 
	 typeid(src) == typeid(cudaColorSpinorField) ) || 
         (param.create == QUDA_REFERENCE_FIELD_CREATE && param.eigv_dim > 0)) {
      reset(param);
    } else {
      errorQuda("Undefined behaviour"); // else silent bug possible?
    }

    // This must be set before create is called
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      if (typeid(src) == typeid(cudaColorSpinorField)) {
	v = (void*)src.V();
	norm = (void*)src.Norm();
      } else {
	errorQuda("Cannot reference a non-cuda field");
      }

      if (this->EigvDim() > 0) 
      {//setup eigenvector form the set
         if(eigv_dim != this->EigvDim()) errorQuda("\nEigenvector set does not match..\n") ;//for debug only.
         if(eigv_id > -1)
         {
           //printfQuda("\nSetting pointers for vector id %d\n", eigv_id); //for debug only.
           v    = (void*)((char*)v + eigv_id*bytes);         
           norm = (void*)((char*)norm + eigv_id*norm_bytes);         
         }
       //do nothing for the eigenvector subset...
      }
    }

    create(param.create);

    if (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      copySpinorField(src);
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else {
      errorQuda("CreateType %d not implemented", param.create);
    }

  }

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src) 
    : ColorSpinorField(src), alloc(false), init(true), texInit(false), 
      initComms(false), bufferMessageHandler(0), nFaceComms(0) {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  ColorSpinorField& cudaColorSpinorField::operator=(const ColorSpinorField &src) {
    if (typeid(src) == typeid(cudaColorSpinorField)) {
      *this = (dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cpuColorSpinorField)) {
      *this = (dynamic_cast<const cpuColorSpinorField&>(src));
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cudaColorSpinorField &src) {
    if (&src != this) {
      // keep current attributes unless unset
      if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
	destroy();
	destroyComms(); // not sure if this necessary
	ColorSpinorField::operator=(src);
	create(QUDA_COPY_FIELD_CREATE);
      }
      copySpinorField(src);
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cpuColorSpinorField &src) {
    // keep current attributes unless unset
    if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
      destroy();
      ColorSpinorField::operator=(src);
      create(QUDA_COPY_FIELD_CREATE);
    }
    loadSpinorField(src);
    return *this;
  }

  cudaColorSpinorField::~cudaColorSpinorField() {
    destroyComms();
    destroy();
  }

  bool cudaColorSpinorField::isNative() const {

    if (precision == QUDA_DOUBLE_PRECISION) {
      if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
    } else if (precision == QUDA_SINGLE_PRECISION) {
      if (nSpin == 4) {
	if (fieldOrder == QUDA_FLOAT4_FIELD_ORDER) return true;
      } else if (nSpin == 1) {
	if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
      }
    } else if (precision == QUDA_HALF_PRECISION) {
      if (nSpin == 4) {
	if (fieldOrder == QUDA_FLOAT4_FIELD_ORDER) return true;
      } else if (nSpin == 1) {
	if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
      }
    }

    return false;
  }

  void cudaColorSpinorField::create(const QudaFieldCreate create) {

    if (siteSubset == QUDA_FULL_SITE_SUBSET && siteOrder != QUDA_EVEN_ODD_SITE_ORDER) {
      errorQuda("Subset not implemented");
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      v = device_malloc(bytes);
      if (precision == QUDA_HALF_PRECISION) {
	norm = device_malloc(norm_bytes);
      }
      alloc = true;
    }

    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      if(eigv_dim != 0) errorQuda("Eigenvectors must be parity fields!");
      // create the associated even and odd subsets
      ColorSpinorParam param;
      param.siteSubset = QUDA_PARITY_SITE_SUBSET;
      param.nDim = nDim;
      memcpy(param.x, x, nDim*sizeof(int));
      param.x[0] /= 2; // set single parity dimensions
      param.create = QUDA_REFERENCE_FIELD_CREATE;
      param.v = v;
      param.norm = norm;
      even = new cudaColorSpinorField(*this, param);
      odd = new cudaColorSpinorField(*this, param);

      // need this hackery for the moment (need to locate the odd pointers half way into the full field)
      (dynamic_cast<cudaColorSpinorField*>(odd))->v = (void*)((char*)v + bytes/2);
      if (precision == QUDA_HALF_PRECISION) 
	(dynamic_cast<cudaColorSpinorField*>(odd))->norm = (void*)((char*)norm + norm_bytes/2);

      for(int i=0; i<nDim; ++i){
        if(commDimPartitioned(i)){
          (dynamic_cast<cudaColorSpinorField*>(odd))->ghost[i] =
	    static_cast<char*>((dynamic_cast<cudaColorSpinorField*>(odd))->ghost[i]) + bytes/2;
          if(precision == QUDA_HALF_PRECISION)
	    (dynamic_cast<cudaColorSpinorField*>(odd))->ghostNorm[i] =
	      static_cast<char*>((dynamic_cast<cudaColorSpinorField*>(odd))->ghostNorm[i]) + norm_bytes/2;
        }
      }

#ifdef USE_TEXTURE_OBJECTS
      dynamic_cast<cudaColorSpinorField*>(even)->destroyTexObject();
      dynamic_cast<cudaColorSpinorField*>(even)->createTexObject();
      dynamic_cast<cudaColorSpinorField*>(odd)->destroyTexObject();
      dynamic_cast<cudaColorSpinorField*>(odd)->createTexObject();
#endif
    }
    else{//siteSubset == QUDA_PARITY_SITE_SUBSET

      //! setup an object for selected eigenvector (the 1st one as a default):
      if ((eigv_dim > 0) && (create != QUDA_REFERENCE_FIELD_CREATE) && (eigv_id == -1)) 
      {
         //if(bytes > 1811939328) warningQuda("\nCUDA API probably won't be able to create texture object for the eigenvector set... Object size is : %u bytes\n", bytes);
         if (getVerbosity() == QUDA_DEBUG_VERBOSE) printfQuda("\nEigenvector set constructor...\n");
         // create the associated even and odd subsets
         ColorSpinorParam param;
         param.siteSubset = QUDA_PARITY_SITE_SUBSET;
         param.nDim = nDim;
         memcpy(param.x, x, nDim*sizeof(int));
         param.create = QUDA_REFERENCE_FIELD_CREATE;
         param.v = v;
         param.norm = norm;
         param.eigv_dim  = eigv_dim;
         //reserve eigvector set
         eigenvectors.reserve(eigv_dim);
         //setup volume, [real_]length and stride for a single eigenvector
         for(int id = 0; id < eigv_dim; id++)
         {
            param.eigv_id = id;
            eigenvectors.push_back(new cudaColorSpinorField(*this, param));

#ifdef USE_TEXTURE_OBJECTS //(a lot of texture objects...)
            dynamic_cast<cudaColorSpinorField*>(eigenvectors[id])->destroyTexObject();
            dynamic_cast<cudaColorSpinorField*>(eigenvectors[id])->createTexObject();
#endif
         }
      }
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (siteSubset != QUDA_FULL_SITE_SUBSET) {
	zeroPad();
      } else {
	(dynamic_cast<cudaColorSpinorField*>(even))->zeroPad();
	(dynamic_cast<cudaColorSpinorField*>(odd))->zeroPad();
      }
    }

#ifdef USE_TEXTURE_OBJECTS
    if((eigv_dim == 0) || (eigv_dim > 0 && eigv_id > -1))
       createTexObject();
#endif

    // initialize the ghost pointers 
    if(siteSubset == QUDA_PARITY_SITE_SUBSET) {
      for(int i=0; i<nDim; ++i){
        if(commDimPartitioned(i)){
          ghost[i] = (char*)v + (stride + ghostOffset[i])*nColor*nSpin*2*precision;
          if(precision == QUDA_HALF_PRECISION)
            ghostNorm[i] = (char*)norm + (stride + ghostNormOffset[i])*QUDA_SINGLE_PRECISION;
        }
      }
    }
    checkCudaError();
  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaColorSpinorField::createTexObject() {

    if (isNative()) {
      if (texInit) errorQuda("Already bound textures");
      
      // create the texture for the field components
      
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // half is short, double is int2
      
      // staggered fields in half and single are always two component
      if (nSpin == 1 && (precision == QUDA_HALF_PRECISION || precision == QUDA_SINGLE_PRECISION)) {
	desc.x = 8*precision;
	desc.y = 8*precision;
	desc.z = 0;
	desc.w = 0;
      } else { // all others are four component
	desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
      }
      
      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = v;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = bytes;
      
      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;
      
      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
      checkCudaError();
      
      // create the texture for the norm components
      if (precision == QUDA_HALF_PRECISION) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	desc.f = hipChannelFormatKindFloat;
	desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;
	
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = norm;
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = norm_bytes;
	
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	
	hipCreateTextureObject(&texNorm, &resDesc, &texDesc, NULL);
	checkCudaError();
      }
      
      texInit = true;
    }
  }

  void cudaColorSpinorField::destroyTexObject() {
    if (isNative() && texInit) {
      hipDestroyTextureObject(tex);
      if (precision == QUDA_HALF_PRECISION) hipDestroyTextureObject(texNorm);
      texInit = false;
      checkCudaError();
    }
  }
#endif

  void cudaColorSpinorField::destroy() {
    if (alloc) {
      device_free(v);
      if (precision == QUDA_HALF_PRECISION) device_free(norm);

      if (siteSubset != QUDA_FULL_SITE_SUBSET) {
        //! for deflated solvers:
        if (eigv_dim > 0) 
        {
          std::vector<ColorSpinorField*>::iterator vec;
          for(vec = eigenvectors.begin(); vec != eigenvectors.end(); vec++) delete *vec;
        } 
      }
      alloc = false;
    }

    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      delete even;
      delete odd;
    }

#ifdef USE_TEXTURE_OBJECTS
    if((eigv_dim == 0) || (eigv_dim > 0 && eigv_id > -1))
       destroyTexObject();
#endif

  }

  cudaColorSpinorField& cudaColorSpinorField::Even() const { 
    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      return *(dynamic_cast<cudaColorSpinorField*>(even)); 
    }

    errorQuda("Cannot return even subset of %d subset", siteSubset);
    exit(-1);
  }

  cudaColorSpinorField& cudaColorSpinorField::Odd() const {
    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      return *(dynamic_cast<cudaColorSpinorField*>(odd)); 
    }

    errorQuda("Cannot return odd subset of %d subset", siteSubset);
    exit(-1);
  }

  // cuda's floating point format, IEEE-754, represents the floating point
  // zero as 4 zero bytes
  void cudaColorSpinorField::zero() {
    hipMemsetAsync(v, 0, bytes, streams[Nstream-1]);
    if (precision == QUDA_HALF_PRECISION) hipMemsetAsync(norm, 0, norm_bytes, streams[Nstream-1]);
  }


  void cudaColorSpinorField::zeroPad() {
    size_t pad_bytes = (stride - volume) * precision * fieldOrder;
    int Npad = nColor * nSpin * 2 / fieldOrder;

    if (eigv_dim > 0 && eigv_id == -1){//we consider the whole eigenvector set:
      Npad      *= eigv_dim;
      pad_bytes /= eigv_dim;
    }

    size_t pitch = ((eigv_dim == 0 || eigv_id != -1) ? stride : eigv_stride)*fieldOrder*precision;
    char   *dst  = (char*)v + ((eigv_dim == 0 || eigv_id != -1) ? volume : eigv_volume)*fieldOrder*precision;
    if(pad_bytes) hipMemset2D(dst, pitch, 0, pad_bytes, Npad);

    //for (int i=0; i<Npad; i++) {
    //  if (pad_bytes) hipMemset((char*)v + (volume + i*stride)*fieldOrder*precision, 0, pad_bytes);
    //}
  }

  void cudaColorSpinorField::copy(const cudaColorSpinorField &src) {
    checkField(*this, src);
    copyCuda(*this, src);
  }

  void cudaColorSpinorField::copySpinorField(const ColorSpinorField &src) {
    
    // src is on the device and is native
    if (typeid(src) == typeid(cudaColorSpinorField) && 
	isNative() && dynamic_cast<const cudaColorSpinorField &>(src).isNative()) {
      copy(dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else if (typeid(src) == typeid(cpuColorSpinorField)) { // src is on the host
      loadSpinorField(src);
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
  } 

  void cudaColorSpinorField::loadSpinorField(const ColorSpinorField &src) {

    if (REORDER_LOCATION == QUDA_CPU_FIELD_LOCATION && 
	typeid(src) == typeid(cpuColorSpinorField)) {
      for(int b=0; b<2; ++b){
        resizeBufferPinned(bytes + norm_bytes, b);
        memset(bufferPinned[b], 0, bytes+norm_bytes); // FIXME (temporary?) bug fix for padding
      }
      copyGenericColorSpinor(*this, src, QUDA_CPU_FIELD_LOCATION, 
			     bufferPinned[bufferIndex], 0, (char*)bufferPinned[bufferIndex]+bytes, 0);

      hipMemcpy(v, bufferPinned[bufferIndex], bytes, hipMemcpyHostToDevice);
      hipMemcpy(norm, (char*)bufferPinned[bufferIndex]+bytes, norm_bytes, hipMemcpyHostToDevice);
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *Src, *srcNorm;
      if (!zeroCopy) {
	resizeBufferDevice(src.Bytes()+src.NormBytes());
	Src = bufferDevice;
	srcNorm = (char*)bufferDevice + src.Bytes();	
	hipMemcpy(Src, src.V(), src.Bytes(), hipMemcpyHostToDevice);
	hipMemcpy(srcNorm, src.Norm(), src.NormBytes(), hipMemcpyHostToDevice);
      } else {
	for(int b=0; b<2; ++b){
	 resizeBufferPinned(src.Bytes()+src.NormBytes(), b);
	}
	memcpy(bufferPinned[bufferIndex], src.V(), src.Bytes());
	memcpy((char*)bufferPinned[bufferIndex]+src.Bytes(), src.Norm(), src.NormBytes());

	hipHostGetDevicePointer(&Src, bufferPinned[bufferIndex], 0);
	srcNorm = (void*)((char*)Src + src.Bytes());
      }

      hipMemset(v, 0, bytes); // FIXME (temporary?) bug fix for padding
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION, 0, Src, 0, srcNorm);
    }

    checkCudaError();
    return;
  }


  void cudaColorSpinorField::saveSpinorField(ColorSpinorField &dest) const {

    if (REORDER_LOCATION == QUDA_CPU_FIELD_LOCATION && 
	typeid(dest) == typeid(cpuColorSpinorField)) {
      for(int b=0; b<2; ++b) resizeBufferPinned(bytes+norm_bytes,b);
      hipMemcpy(bufferPinned[bufferIndex], v, bytes, hipMemcpyDeviceToHost);
      hipMemcpy((char*)bufferPinned[bufferIndex]+bytes, norm, norm_bytes, hipMemcpyDeviceToHost);

      copyGenericColorSpinor(dest, *this, QUDA_CPU_FIELD_LOCATION, 
			     0, bufferPinned[bufferIndex], 0, (char*)bufferPinned[bufferIndex]+bytes);
    } else if (typeid(dest) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *dst, *dstNorm;
      if (!zeroCopy) {
	resizeBufferDevice(dest.Bytes()+dest.NormBytes());
	dst = bufferDevice;
	dstNorm = (char*)bufferDevice+dest.Bytes();
      } else {
	for(int b=0; b<2; ++b) resizeBufferPinned(dest.Bytes()+dest.NormBytes(),b);
	hipHostGetDevicePointer(&dst, bufferPinned[bufferIndex], 0);
	dstNorm = (char*)dst+dest.Bytes();
      }
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION, dst, v, dstNorm, 0);

      if (!zeroCopy) {
	hipMemcpy(dest.V(), dst, dest.Bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(dest.Norm(), dstNorm, dest.NormBytes(), hipMemcpyDeviceToHost);
      } else {
	memcpy(dest.V(), bufferPinned[bufferIndex], dest.Bytes());
	memcpy(dest.Norm(), (char*)bufferPinned[bufferIndex]+dest.Bytes(), dest.NormBytes());
      }
    }

    checkCudaError();
    return;
  }

  void cudaColorSpinorField::allocateGhostBuffer(int nFace) {
    int Nint = nColor * nSpin * 2; // number of internal degrees of freedom
    if (nSpin == 4) Nint /= 2; // spin projection for Wilson

    // compute size of buffer required
    size_t faceBytes = 0;
    for (int i=0; i<4; i++) {
      if(!commDimPartitioned(i)) continue;
      faceBytes += 2*nFace*ghostFace[i]*Nint*precision;
      // add extra space for the norms for half precision
      if (precision == QUDA_HALF_PRECISION) faceBytes += 2*nFace*ghostFace[i]*sizeof(float);
    }

    // only allocate if not already allocated or buffer required is bigger than previously
    if(initGhostFaceBuffer == 0 || faceBytes > ghostFaceBytes){    

      if (initGhostFaceBuffer){
        for(int b=0; b<2; ++b) device_free(ghostFaceBuffer[b]); 
      }

      if (faceBytes > 0) {
	for(int b=0; b<2; ++b) ghostFaceBuffer[b] = device_malloc(faceBytes);
	initGhostFaceBuffer = 1;
	ghostFaceBytes = faceBytes;
      }

    }

    size_t offset = 0;
    for (int i=0; i<4; i++) {
      if(!commDimPartitioned(i)) continue;
    
      for(int b=0; b<2; ++b) backGhostFaceBuffer[b][i] = (void*)(((char*)ghostFaceBuffer[b]) + offset);
      offset += nFace*ghostFace[i]*Nint*precision;
      if (precision == QUDA_HALF_PRECISION) offset += nFace*ghostFace[i]*sizeof(float);
      
      for(int b=0; b<2; ++b) fwdGhostFaceBuffer[b][i] = (void*)(((char*)ghostFaceBuffer[b]) + offset);
      offset += nFace*ghostFace[i]*Nint*precision;
      if (precision == QUDA_HALF_PRECISION) offset += nFace*ghostFace[i]*sizeof(float);
    }   
    
  }


  void cudaColorSpinorField::freeGhostBuffer(void)
  {
    if (!initGhostFaceBuffer) return;
  
    for(int b=0; b<2; ++b) device_free(ghostFaceBuffer[b]); 

    for(int i=0;i < 4; i++){
      if(!commDimPartitioned(i)) continue;
      for(int b=0; b<2; ++b){
        backGhostFaceBuffer[b][i] = NULL;
        fwdGhostFaceBuffer[b][i] = NULL;
      }
    }
    initGhostFaceBuffer = 0;  
  }

  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhost(const int nFace, const QudaParity parity, 
                                       const int dim, const QudaDirection dir,
				       const int dagger, hipStream_t *stream, 
				       void *buffer, double a, double b) 
  {
#ifdef MULTI_GPU
    int face_num;
    if(dir == QUDA_BACKWARDS){
      face_num = 0;
    }else if(dir == QUDA_FORWARDS){
      face_num = 1;
    }else{
      face_num = 2;
    }
    void *packBuffer = buffer ? buffer : ghostFaceBuffer[bufferIndex];
    packFace(packBuffer, *this, nFace, dagger, parity, dim, face_num, *stream, a, b); 
#else
    errorQuda("packGhost not built on single-GPU build");
#endif

  }
 
  // send the ghost zone to the host
  void cudaColorSpinorField::sendGhost(void *ghost_spinor, const int nFace, const int dim, 
				       const QudaDirection dir, const int dagger, 
				       hipStream_t *stream) {

#ifdef MULTI_GPU
    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom
    
    if (dim !=3 || getKernelPackT() || getTwistPack()) { // use kernels to pack into contiguous buffers then a single hipMemcpy

      size_t bytes = nFace*Nint*ghostFace[dim]*precision;
      if (precision == QUDA_HALF_PRECISION) bytes += nFace*ghostFace[dim]*sizeof(float);
      void* gpu_buf = 
	(dir == QUDA_BACKWARDS) ? this->backGhostFaceBuffer[bufferIndex][dim] : this->fwdGhostFaceBuffer[bufferIndex][dim];

      hipMemcpyAsync(ghost_spinor, gpu_buf, bytes, hipMemcpyDeviceToHost, *stream); 

    } else if(this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET){ // do multiple cudaMemcpys

      int Npad = Nint / Nvec; // number Nvec buffers we have
      int Nt_minus1_offset = (volume - nFace*ghostFace[3]); // N_t -1 = Vh-Vsh
      int offset = 0;
      if (nSpin == 1) {
	offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset;
      } else if (nSpin == 4) {    
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back  
	if (dir == QUDA_FORWARDS) upper = !upper;
	int lower_spin_offset = Npad*stride;
	if (upper) offset = (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
	else offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
      }
    
      // QUDA Memcpy NPad's worth. 
      //  -- Dest will point to the right beginning PAD. 
      //  -- Each Pad has size Nvec*Vsh Floats. 
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + offset*Nvec*precision;
      size_t len = nFace*ghostFace[3]*Nvec*precision;     
      size_t spitch = stride*Nvec*precision;
      hipMemcpy2DAsync(dst, len, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
	int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + nFace*Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
	hipMemcpyAsync(dst, src, nFace*ghostFace[3]*sizeof(float), hipMemcpyDeviceToHost, *stream); 
      }
    }else{
      int flavorVolume = volume / 2;
      int flavorTFace  = ghostFace[3] / 2;
      int Npad = Nint / Nvec; // number Nvec buffers we have
      int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
      int flavor2_Nt_minus1_offset = (volume - flavorTFace);
      int flavor1_offset = 0;
      int flavor2_offset = 0;
      // !dagger: send lower components backwards, send upper components forwards
      // dagger: send upper components backwards, send lower components forwards
      bool upper = dagger ? true : false; // Fwd is !Back
      if (dir == QUDA_FORWARDS) upper = !upper;
      int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
      if (upper){
        flavor1_offset = (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }else{
        flavor1_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }

      // QUDA Memcpy NPad's worth.
      //  -- Dest will point to the right beginning PAD.
      //  -- Each Pad has size Nvec*Vsh Floats.
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + flavor1_offset*Nvec*precision;
      size_t len = flavorTFace*Nvec*precision;
      size_t spitch = stride*Nvec*precision;//ndeg tm: stride=2*flavor_volume+pad
      size_t dpitch = 2*len;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);
      dst = (char*)ghost_spinor+len;
      src = (char*)v + flavor2_offset*Nvec*precision;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
        int Nt_minus1_offset = (flavorVolume - flavorTFace);
        int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
        size_t dpitch = flavorTFace*sizeof(float);
        size_t spitch = flavorVolume*sizeof(float);
	hipMemcpy2DAsync(dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToHost, *stream);
      }
    }
#else
    errorQuda("sendGhost not built on single-GPU build");
#endif

  }



  void cudaColorSpinorField::unpackGhost(const void* ghost_spinor, const int nFace, 
					 const int dim, const QudaDirection dir, 
					 const int dagger, hipStream_t* stream) 
  {
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint;
    int offset = length + ghostOffset[dim]*nColor*nSpin*2;
    offset += (dir == QUDA_BACKWARDS) ? 0 : len;

    void *dst = (char*)v + precision*offset;
    const void *src = ghost_spinor;

    hipMemcpyAsync(dst, src, len*precision, hipMemcpyHostToDevice, *stream);
    
    if (precision == QUDA_HALF_PRECISION) {
      // norm region of host ghost zone is at the end of the ghost_spinor

      int normlen = nFace*ghostFace[dim];
      int norm_offset = stride + ghostNormOffset[dim];
      norm_offset += (dir == QUDA_BACKWARDS) ? 0 : normlen;

      void *dst = static_cast<char*>(norm) + norm_offset*sizeof(float);
      const void *src = static_cast<const char*>(ghost_spinor)+nFace*Nint*ghostFace[dim]*precision; 
      hipMemcpyAsync(dst, src, normlen*sizeof(float), hipMemcpyHostToDevice, *stream);
    }
  }




   // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhostExtended(const int nFace, const int R[], const QudaParity parity,
                                       const int dim, const QudaDirection dir,
                                       const int dagger, hipStream_t *stream,
                                       void *buffer)
  {
#ifdef MULTI_GPU
    int face_num;
    if(dir == QUDA_BACKWARDS){
      face_num = 0;
    }else if(dir == QUDA_FORWARDS){
      face_num = 1;
    }else{
      face_num = 2;
    }
    void *packBuffer = buffer ? buffer : ghostFaceBuffer[bufferIndex];
    packFaceExtended(packBuffer, *this, nFace, R, dagger, parity, dim, face_num, *stream);
#else
    errorQuda("packGhostExtended not built on single-GPU build");
#endif

  }


  

  // copy data from host buffer into boundary region of device field
  void cudaColorSpinorField::unpackGhostExtended(const void* ghost_spinor, const int nFace, const QudaParity parity,
                                                 const int dim, const QudaDirection dir, 
                                                 const int dagger, hipStream_t* stream)
  {

     
     
    // First call the regular unpackGhost routine to copy data into the `usual' ghost-zone region 
    // of the data array 
    unpackGhost(ghost_spinor, nFace, dim, dir, dagger, stream);

    // Next step is to copy data from the ghost zone back to the interior region
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint;
    int offset = length + ghostOffset[dim]*nColor*nSpin*2;
    offset += (dir == QUDA_BACKWARDS) ? 0 : len;

#ifdef MULTI_GPU
    const int face_num = 2;
    const bool unpack = true;
    const int R[4] = {0,0,0,0};
    packFaceExtended(ghostFaceBuffer[bufferIndex], *this, nFace, R, dagger, parity, dim, face_num, *stream, unpack); 
#else
    errorQuda("unpackGhostExtended not built on single-GPU build");
#endif
  }



  hipStream_t *stream;

  void cudaColorSpinorField::createComms(int nFace) {

    if(bufferMessageHandler != bufferPinnedResizeCount) destroyComms();

    if (!initComms || nFaceComms != nFace) {

      // if we are requesting a new number of faces destroy and start over
      if(nFace != nFaceComms) destroyComms();

      if (siteSubset != QUDA_PARITY_SITE_SUBSET) 
	errorQuda("Only supports single parity fields");

#ifdef GPU_COMMS
      bool comms = false;
      for (int i=0; i<nDimComms; i++) if (commDimPartitioned(i)) comms = true;
#endif

      if (nFace > maxNface) 
	errorQuda("Requested number of faces %d in communicator is greater than supported %d",
		  nFace, maxNface);

      // faceBytes is the sum of all face sizes 
      size_t faceBytes = 0;
      
      // nbytes is the size in bytes of each face
      size_t nbytes[QUDA_MAX_DIM];
      
      // The number of degrees of freedom per site for the given
      // field.  Currently assumes spin projection of a Wilson-like
      // field (so half the number of degrees of freedom).
      int Ndof = (2 * nSpin * nColor) / (nSpin==4 ? 2 : 1);

      for (int i=0; i<nDimComms; i++) {
	nbytes[i] = maxNface*surfaceCB[i]*Ndof*precision;
	if (precision == QUDA_HALF_PRECISION) nbytes[i] += maxNface*surfaceCB[i]*sizeof(float);
	if (!commDimPartitioned(i)) continue;
	faceBytes += 2*nbytes[i];
      }
      
#ifndef GPU_COMMS
      // use static pinned memory for face buffers
      for(int b=0; b<2; ++b){
        resizeBufferPinned(2*faceBytes, b); // oversizes for GPU_COMMS case

        my_face[b] = bufferPinned[b];
        from_face[b] = static_cast<char*>(bufferPinned[b]) + faceBytes;
      }

      // assign pointers for each face - it's ok to alias for different Nface parameters
      size_t offset = 0;
#endif
      for (int i=0; i<nDimComms; i++) {
	if (!commDimPartitioned(i)) continue;
	
#ifdef GPU_COMMS
	for(int b=0; b<2; ++b){
	  my_back_face[b][i] = backGhostFaceBuffer[b][i];
	  from_back_face[b][i] = ghost[i];
	
	  if(precision == QUDA_HALF_PRECISION){
	    my_back_norm_face[b][i]  = static_cast<char*>(backGhostFaceBuffer[b][i]) + nFace*ghostFace[i]*Ndof*precision;
	    from_back_norm_face[b][i] = ghostNorm[i];
	  }
	} // loop over b

#else
        for(int b=0; b<2; ++b){
	  my_back_face[b][i] = static_cast<char*>(my_face[b]) + offset;
	  from_back_face[b][i] = static_cast<char*>(from_face[b]) + offset;
	}
	offset += nbytes[i];
#endif
	
#ifdef GPU_COMMS
	for(int b=0; b<2; ++b){
	  my_fwd_face[b][i] = fwdGhostFaceBuffer[b][i];	
	  from_fwd_face[b][i] = ghost[i] + nFace*ghostFace[i]*Ndof*precision;

	  if(precision == QUDA_HALF_PRECISION){
	    my_fwd_norm_face[b][i] = static_cast<char*>(fwdGhostFaceBuffer[b][i]) + nFace*ghostFace[i]*Ndof*precision;
	    from_fwd_norm_face[b][i] = static_cast<char*>(ghostNorm[i]) + nFace*ghostFace[i]*sizeof(float);
	  }
	} // loop over b
#else
	for(int b=0; b<2; ++b){
	  my_fwd_face[b][i] = static_cast<char*>(my_face[b]) + offset;
	  from_fwd_face[b][i] = static_cast<char*>(from_face[b]) + offset;
	}
	offset += nbytes[i];
#endif

      }

      // create a different message handler for each direction and Nface
      for(int b=0; b<2; ++b){
        mh_send_fwd[b] = new MsgHandle**[maxNface];
        mh_send_back[b] = new MsgHandle**[maxNface];
        mh_recv_fwd[b] = new MsgHandle**[maxNface];
        mh_recv_back[b] = new MsgHandle**[maxNface];
#ifdef GPU_COMMS
        if(precision == QUDA_HALF_PRECISION){
      	  mh_send_norm_fwd[b]  = new MsgHandle**[maxNface];
      	  mh_send_norm_back[b] = new MsgHandle**[maxNface];
     	  mh_recv_norm_fwd[b]  = new MsgHandle**[maxNface];
      	  mh_recv_norm_back[b] = new MsgHandle**[maxNface]; 
        }
#endif
      } // loop over b
      for (int j=0; j<maxNface; j++) {
	for(int b=0; b<2; ++b){
	  mh_send_fwd[b][j] = new MsgHandle*[2*nDimComms];
	  mh_send_back[b][j] = new MsgHandle*[2*nDimComms];
	  mh_recv_fwd[b][j] = new MsgHandle*[nDimComms];
	  mh_recv_back[b][j] = new MsgHandle*[nDimComms];
		
#ifdef GPU_COMMS
	  if(precision == QUDA_HALF_PRECISION){
	    mh_send_norm_fwd[b][j] = new MsgHandle*[2*nDimComms];
	    mh_send_norm_back[b][j] = new MsgHandle*[2*nDimComms];
	    mh_recv_norm_fwd[b][j] = new MsgHandle*[nDimComms];
	    mh_recv_norm_back[b][j] = new MsgHandle*[nDimComms];
	  }
#endif	
	} // loop over b


	for (int i=0; i<nDimComms; i++) {
	  if (!commDimPartitioned(i)) continue;
#ifdef GPU_COMMS
	  size_t nbytes_Nface = surfaceCB[i]*Ndof*precision*(j+1);
	  size_t nbytes_Nface_norm = surfaceCB[i]*(j+1)*sizeof(float);
	  if (i != 3 || getKernelPackT() || getTwistPack()) {
#else 
	    size_t nbytes_Nface = (nbytes[i] / maxNface) * (j+1);
#endif
	    for(int b=0; b<2; ++b){
	      mh_send_fwd[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(my_fwd_face[b][i], i, +1, nbytes_Nface) : NULL;
	      mh_send_back[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(my_back_face[b][i], i, -1, nbytes_Nface) : NULL;
	      mh_send_fwd[b][j][2*i+1] = mh_send_fwd[b][j][2*i]; // alias pointers
	      mh_send_back[b][j][2*i+1] = mh_send_back[b][j][2*i]; // alias pointers
	    }
#ifdef GPU_COMMS

	    if(precision == QUDA_HALF_PRECISION){
	      for(int b=0; b<2; ++b){
		mh_send_norm_fwd[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(my_fwd_norm_face[b][i], i, +1, nbytes_Nface_norm) : NULL;
		mh_send_norm_back[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(my_back_norm_face[b][i], i, -1, nbytes_Nface_norm) : NULL;
		mh_send_norm_fwd[b][j][2*i+1] = mh_send_norm_fwd[b][j][2*i];
		mh_send_norm_back[b][j][2*i+1] = mh_send_norm_back[b][j][2*i]; 	
	      }
	    }

	  } else if (this->TwistFlavor() == QUDA_TWIST_NONDEG_DOUBLET) {
	    errorQuda("GPU_COMMS for non-degenerate doublet only supported with time-dimension kernel packing enabled.");
	  } else {
	    /* 
	       use a strided communicator, here we can't really use
	       the previously declared my_fwd_face and my_back_face
	       pointers since they don't really map 1-to-1 so let's
	       just compute the required base pointers and pass these
	       directly into the communicator construction
	    */
	    
	    int Nblocks = Ndof / Nvec(); // number of Nvec buffers we have
	    // start of last time slice chunk we are sending forwards
	    int endOffset = (volume - (j+1)*ghostFace[i]); 

	    size_t offset[4];
	    void *base[4];
	    if (nSpin == 1) { // staggered is invariant with dagger
	      offset[2*0 + 0] = 0;
	      offset[2*1 + 0] = endOffset;
	      offset[2*0 + 1] = offset[2*0 + 0];
	      offset[2*1 + 1] = offset[2*1 + 0];
	    } else if (nSpin == 4) {    
	      // !dagger: send last components backwards, send first components forwards
	      offset[2*0 + 0] = Nblocks*stride;
	      offset[2*1 + 0] = endOffset;
	      //  dagger: send first components backwards, send last components forwards
	      offset[2*0 + 1] = 0;
	      offset[2*1 + 1] = Nblocks*stride + endOffset;
	    } else {
	      errorQuda("Unsupported number of spin components");
	    }

	    for (int k=0; k<4; k++) {
	      base[k] = static_cast<char*>(v) + offset[k]*Nvec()*precision; // total offset in bytes
	    }

	    size_t blksize  = (j+1)*ghostFace[i]*Nvec()*precision; // (j+1) is number of faces
	    size_t Stride = stride*Nvec()*precision;

	    if (blksize * Nblocks != nbytes_Nface) 
	      errorQuda("Total strided message size does not match expected size");

	    //printf("%d strided sends with Nface=%d Nblocks=%d blksize=%d Stride=%d\n", i, j+1, Nblocks, blksize, Stride);

            for(int b=0; b<2; ++b){
	      // only allocate a communicator for the present face (this needs cleaned up)
	      mh_send_fwd[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_strided_send_relative(base[2], i, +1, blksize, Nblocks, Stride) : NULL;
	      mh_send_back[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_strided_send_relative(base[0], i, -1, blksize, Nblocks, Stride) : NULL;
	      if (nSpin ==4) { // dagger communicators
	        mh_send_fwd[b][j][2*i+1] = (j+1 == nFace) ? comm_declare_strided_send_relative(base[3], i, +1, blksize, Nblocks, Stride) : NULL;
	        mh_send_back[b][j][2*i+1] = (j+1 == nFace) ? comm_declare_strided_send_relative(base[1], i, -1, blksize, Nblocks, Stride) : NULL;
	      } else {
	        mh_send_fwd[b][j][2*i+1] = mh_send_fwd[b][j][2*i+0];
	        mh_send_back[b][j][2*i+1] = mh_send_back[b][j][2*i+0];
	      }

            } // loop over b

          
	    if(precision == QUDA_HALF_PRECISION){
	      int Nt_minus1_offset = (volume - nFace*ghostFace[3]); // The space-time coordinate of the start of the last time slice
	      void *norm_fwd = static_cast<float*>(norm) + Nt_minus1_offset;
	      void *norm_back = norm; // the first time slice has zero offset
	      for(int b=0; b<2; ++b){
		mh_send_norm_fwd[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(norm_fwd, i, +1, surfaceCB[i]*(j+1)*sizeof(float)) : NULL;
		mh_send_norm_back[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(norm_back, i, -1, surfaceCB[i]*(j+1)*sizeof(float)) : NULL;
		mh_send_norm_fwd[b][j][2*i+1] = mh_send_norm_fwd[b][j][2*i];
		mh_send_norm_back[b][j][2*i+1] = mh_send_norm_back[b][j][2*i];  
	      }
	    }

	  }

	  if(precision == QUDA_HALF_PRECISION){
            for(int b=0; b<2; ++b){
	      mh_recv_norm_fwd[b][j][i] = (j+1 == nFace) ? comm_declare_receive_relative(from_fwd_norm_face[b][i], i, +1, nbytes_Nface_norm) : NULL;
	      mh_recv_norm_back[b][j][i] = (j+1 == nFace) ? comm_declare_receive_relative(from_back_norm_face[b][i], i, -1, nbytes_Nface_norm) : NULL;
            }
	  }
#endif // GPU_COMMS

	  for(int b=0; b<2; ++b){
	    mh_recv_fwd[b][j][i] = (j+1 == nFace) ? comm_declare_receive_relative(from_fwd_face[b][i], i, +1, nbytes_Nface) : NULL;
	    mh_recv_back[b][j][i] = (j+1 == nFace) ? comm_declare_receive_relative(from_back_face[b][i], i, -1, nbytes_Nface) : NULL;
	  }
	 


	} // loop over dimension
      }
     
      bufferMessageHandler = bufferPinnedResizeCount;
      initComms = true;
      nFaceComms = nFace;
    }
    checkCudaError();
  }
    
  void cudaColorSpinorField::destroyComms() {
    if (initComms) {
      for(int b=0; b<2; ++b){
      for (int j=0; j<maxNface; j++) {
	for (int i=0; i<nDimComms; i++) {
	  if (commDimPartitioned(i)) {
	    if (mh_recv_fwd[b][j][i]) comm_free(mh_recv_fwd[b][j][i]);
	    if (mh_recv_fwd[b][j][i]) comm_free(mh_recv_back[b][j][i]);
	    if (mh_send_fwd[b][j][2*i]) comm_free(mh_send_fwd[b][j][2*i]);
	    if (mh_send_back[b][j][2*i]) comm_free(mh_send_back[b][j][2*i]);
	    // only in a special case are these not aliasing pointers
#ifdef GPU_COMMS
	    if(precision == QUDA_HALF_PRECISION){
	      if (mh_recv_norm_fwd[b][j][i]) comm_free(mh_recv_norm_fwd[b][j][i]);
	      if (mh_recv_norm_back[b][j][i]) comm_free(mh_recv_norm_back[b][j][i]);
	      if (mh_send_norm_fwd[b][j][2*i]) comm_free(mh_send_norm_fwd[b][j][2*i]);
	      if (mh_send_norm_back[b][j][2*i]) comm_free(mh_send_norm_back[b][j][2*i]);
	    }

	    if (i == 3 && !getKernelPackT() && nSpin == 4) {
	      if (mh_send_fwd[b][j][2*i+1]) comm_free(mh_send_fwd[b][j][2*i+1]);
	      if (mh_send_back[b][j][2*i+1]) comm_free(mh_send_back[b][j][2*i+1]);
	    }
#endif // GPU_COMMS
	  }
	}
	delete []mh_recv_fwd[b][j];
	delete []mh_recv_back[b][j];
	delete []mh_send_fwd[b][j];
	delete []mh_send_back[b][j];
#ifdef GPU_COMMS
	if(precision == QUDA_HALF_PRECISION){
	  delete []mh_recv_norm_fwd[b][j];
	  delete []mh_recv_norm_back[b][j];
	  delete []mh_send_norm_fwd[b][j];
	  delete []mh_send_norm_back[b][j];
	}
#endif
      }    
      delete []mh_recv_fwd[b];
      delete []mh_recv_back[b];
      delete []mh_send_fwd[b];
      delete []mh_send_back[b];
      
      for (int i=0; i<nDimComms; i++) {
	my_fwd_face[b][i] = NULL;
	my_back_face[b][i] = NULL;
	from_fwd_face[b][i] = NULL;
	from_back_face[b][i] = NULL;      
      }
#ifdef GPU_COMMS
      if(precision == QUDA_HALF_PRECISION){
	delete []mh_recv_norm_fwd[b];
	delete []mh_recv_norm_back[b];
	delete []mh_send_norm_fwd[b];
	delete []mh_send_norm_back[b];
      }
	
      for(int i=0; i<nDimComms; i++){
	my_fwd_norm_face[b][i] = NULL;
	my_back_norm_face[b][i] = NULL;
	from_fwd_norm_face[b][i] = NULL;
	from_back_norm_face[b][i] = NULL;
      }
#endif      
      } // loop over b
      initComms = false;
      checkCudaError();
    }
  }

  void cudaColorSpinorField::streamInit(hipStream_t *stream_p){
    stream = stream_p;
  }

  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, hipStream_t *stream_p, 
				  bool zeroCopyPack, double a, double b) {
    allocateGhostBuffer(nFace);   // allocate the ghost buffer if not yet allocated  
    createComms(nFace); // must call this first

    stream = stream_p;
    
    const int dim=-1; // pack all partitioned dimensions
 
    if (zeroCopyPack) {
      void *my_face_d;
      hipHostGetDevicePointer(&my_face_d, my_face[bufferIndex], 0); // set the matching device pointer
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[0], my_face_d, a, b);
    } else {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger,  &stream[Nstream-1], 0, a, b);
    }
  }

  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, int stream_idx, 
				  bool zeroCopyPack, double a, double b) {
    allocateGhostBuffer(nFace);   // allocate the ghost buffer if not yet allocated  
    createComms(nFace); // must call this first

    const int dim=-1; // pack all partitioned dimensions
 
    if (zeroCopyPack) {
      void *my_face_d;
      hipHostGetDevicePointer(&my_face_d, my_face[bufferIndex], 0); // set the matching device pointer
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[stream_idx], my_face_d, a, b);
    } else {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger,  &stream[stream_idx], 0, a, b);
    }
  }

  void cudaColorSpinorField::packExtended(const int nFace, const int R[], const int parity, 
                                          const int dagger, const int dim,
                                          hipStream_t *stream_p, const bool zeroCopyPack){

    allocateGhostBuffer(nFace); // allocate the ghost buffer if not yet allocated
    createComms(nFace); // must call this first

    stream = stream_p;
 
    void *my_face_d = NULL;
    if(zeroCopyPack){ 
      hipHostGetDevicePointer(&my_face_d, my_face[bufferIndex], 0);
      packGhostExtended(nFace, R, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[0], my_face_d);
    }else{
      packGhostExtended(nFace, R, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[Nstream-1], my_face_d);
    }
  }
                                                      


  void cudaColorSpinorField::gather(int nFace, int dagger, int dir, hipStream_t* stream_p)
  {
    int dim = dir/2;

    // If stream_p != 0, use pack_stream, else use the stream array
    hipStream_t *pack_stream = (stream_p) ? stream_p : stream+dir;

    if(dir%2 == 0){
      // backwards copy to host
      sendGhost(my_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, pack_stream);
    } else {
      // forwards copy to host
      sendGhost(my_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, pack_stream);
    }
  }


  void cudaColorSpinorField::recvStart(int nFace, int dir, int dagger) {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;

    if (dir%2 == 0) { // sending backwards
      // Prepost receive
      comm_start(mh_recv_fwd[bufferIndex][nFace-1][dim]);
    } else { //sending forwards
      // Prepost receive
      comm_start(mh_recv_back[bufferIndex][nFace-1][dim]);
    }
#ifdef GPU_COMMS
    if(precision != QUDA_HALF_PRECISION) return;

    if (dir%2 == 0) { // sending backwards
      // Prepost receive
      comm_start(mh_recv_norm_fwd[bufferIndex][nFace-1][dim]);
    } else { //sending forwards
      // Prepost receive
      comm_start(mh_recv_norm_back[bufferIndex][nFace-1][dim]);
    }
#endif
  }

  void cudaColorSpinorField::sendStart(int nFace, int dir, int dagger) {
    int dim = dir / 2;
    if(!commDimPartitioned(dim)) return;

    if (dir%2 == 0) { // sending backwards
      comm_start(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]);
    } else { //sending forwards
      comm_start(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]);
    }
#ifdef GPU_COMMS
    if(precision != QUDA_HALF_PRECISION) return;
    if (dir%2 == 0) { // sending backwards
      comm_start(mh_send_norm_back[bufferIndex][nFace-1][2*dim+dagger]);
    } else { //sending forwards
      comm_start(mh_send_norm_fwd[bufferIndex][nFace-1][2*dim+dagger]);
    }
#endif
  }




 void cudaColorSpinorField::commsStart(int nFace, int dir, int dagger) {
    int dim = dir / 2;
    if(!commDimPartitioned(dim)) return;
    
    if (dir%2 == 0) { // sending backwards
      // Prepost receive
      comm_start(mh_recv_fwd[bufferIndex][nFace-1][dim]);
      comm_start(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]);
    } else { //sending forwards
      // Prepost receive
      comm_start(mh_recv_back[bufferIndex][nFace-1][dim]);
      // Begin forward send
      comm_start(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]);
    }
#ifdef GPU_COMMS
    if(precision != QUDA_HALF_PRECISION) return;		

    if (dir%2 == 0) { // sending backwards
      // Prepost receive
      comm_start(mh_recv_norm_fwd[bufferIndex][nFace-1][dim]);

      comm_start(mh_send_norm_back[bufferIndex][nFace-1][2*dim+dagger]);
    } else { //sending forwards
      // Prepost receive
      comm_start(mh_recv_norm_back[bufferIndex][nFace-1][dim]);
      // Begin forward send
      comm_start(mh_send_norm_fwd[bufferIndex][nFace-1][2*dim+dagger]);
    }
#endif
  }

  int cudaColorSpinorField::commsQuery(int nFace, int dir, int dagger) {
    int dim = dir / 2;
    if(!commDimPartitioned(dim)) return 0;

#ifdef GPU_COMMS
    if(precision != QUDA_HALF_PRECISION){
#endif
    if(dir%2==0) {
      if (comm_query(mh_recv_fwd[bufferIndex][nFace-1][dim]) && 
	  comm_query(mh_send_back[bufferIndex][nFace-1][2*dim+dagger])) return 1;
    } else {
      if (comm_query(mh_recv_back[bufferIndex][nFace-1][dim]) && 
	  comm_query(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger])) return 1;
    }
#ifdef GPU_COMMS
   }else{ // half precision
    if(dir%2==0) {
      if (comm_query(mh_recv_fwd[bufferIndex][nFace-1][dim]) && 
	  comm_query(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]) &&
	  comm_query(mh_recv_norm_fwd[bufferIndex][nFace-1][dim]) &&
	  comm_query(mh_send_norm_back[bufferIndex][nFace-1][2*dim+dagger])) return 1;
    } else {
      if (comm_query(mh_recv_back[bufferIndex][nFace-1][dim]) && 
	  comm_query(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]) &&
	  comm_query(mh_recv_norm_back[bufferIndex][nFace-1][dim]) && 
	  comm_query(mh_send_norm_fwd[bufferIndex][nFace-1][2*dim+dagger])) return 1;
    }
   } // half precision
#endif
    return 0;
  }

  void cudaColorSpinorField::commsWait(int nFace, int dir, int dagger) {
    int dim = dir / 2;
    if(!commDimPartitioned(dim)) return;

#ifdef GPU_COMMS
    if(precision != QUDA_HALF_PRECISION){
#endif
    if (dir%2==0) {
      comm_wait(mh_recv_fwd[bufferIndex][nFace-1][dim]);
      comm_wait(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]);
    } else {
      comm_wait(mh_recv_back[bufferIndex][nFace-1][dim]);
      comm_wait(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]);
    }
#ifdef GPU_COMMS
   } else { // half precision
      if (dir%2==0) {
	comm_wait(mh_recv_fwd[bufferIndex][nFace-1][dim]);
	comm_wait(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]);
	comm_wait(mh_recv_norm_fwd[bufferIndex][nFace-1][dim]);
	comm_wait(mh_send_norm_back[bufferIndex][nFace-1][2*dim+dagger]);
      } else {
	comm_wait(mh_recv_back[bufferIndex][nFace-1][dim]);
	comm_wait(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]);
	comm_wait(mh_recv_norm_back[bufferIndex][nFace-1][dim]);
	comm_wait(mh_send_norm_fwd[bufferIndex][nFace-1][2*dim+dagger]);
      }
    } // half precision
#endif

    return;
  }

  void cudaColorSpinorField::scatter(int nFace, int dagger, int dir, hipStream_t* stream_p)
  {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;

    // both scattering occurances now go through the same stream
    if (dir%2==0) {// receive from forwards
      unpackGhost(from_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, stream_p);
    } else { // receive from backwards
      unpackGhost(from_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, stream_p);
    }
  }



  void cudaColorSpinorField::scatter(int nFace, int dagger, int dir)
  {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;
    
    // both scattering occurances now go through the same stream
    if (dir%2==0) {// receive from forwards
      unpackGhost(from_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, &stream[2*dim/*+0*/]);
    } else { // receive from backwards
      unpackGhost(from_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, &stream[2*dim/*+1*/]);
    }
  }

  
  void cudaColorSpinorField::scatterExtended(int nFace, int parity, int dagger, int dir)
  {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;
    if (dir%2==0) {// receive from forwards
      unpackGhostExtended(from_fwd_face[bufferIndex][dim], nFace, static_cast<QudaParity>(parity), dim, QUDA_FORWARDS, dagger, &stream[2*dim/*+0*/]);
    } else { // receive from backwards
      unpackGhostExtended(from_back_face[bufferIndex][dim], nFace, static_cast<QudaParity>(parity),  dim, QUDA_BACKWARDS, dagger, &stream[2*dim/*+1*/]);
    }
  }
 

  // Return the location of the field
  QudaFieldLocation cudaColorSpinorField::Location() const { return QUDA_CUDA_FIELD_LOCATION; }

  std::ostream& operator<<(std::ostream &out, const cudaColorSpinorField &a) {
    out << (const ColorSpinorField&)a;
    out << "v = " << a.v << std::endl;
    out << "norm = " << a.norm << std::endl;
    out << "alloc = " << a.alloc << std::endl;
    out << "init = " << a.init << std::endl;
    return out;
  }

//! for deflated solvers:
  cudaColorSpinorField& cudaColorSpinorField::Eigenvec(const int idx) const {
    
    if (siteSubset == QUDA_PARITY_SITE_SUBSET && this->EigvId() == -1) {
      if (idx < this->EigvDim()) {//setup eigenvector form the set
        return *(dynamic_cast<cudaColorSpinorField*>(eigenvectors[idx])); 
      }
      else{
        errorQuda("Incorrect eigenvector index...");
      }
    }
    errorQuda("Eigenvector must be a parity spinor");
    exit(-1);
  }

//copyCuda currently cannot not work with set of spinor fields..
  void cudaColorSpinorField::CopyEigenvecSubset(cudaColorSpinorField &dst, const int range, const int first_element) const{
#if 0
    if(first_element < 0) errorQuda("\nError: trying to set negative first element.\n");
    if (siteSubset == QUDA_PARITY_SITE_SUBSET && this->EigvId() == -1) {
      if (first_element == 0 && range == this->EigvDim())
      {
        if(range != dst.EigvDim())errorQuda("\nError: eigenvector range to big.\n");
        checkField(dst, *this);
        copyCuda(dst, *this);
      }
      else if ((first_element+range) < this->EigvDim()) 
      {//setup eigenvector subset

        cudaColorSpinorField *eigv_subset;

        ColorSpinorParam param;

        param.nColor = nColor;
        param.nSpin = nSpin;
        param.twistFlavor = twistFlavor;
        param.precision = precision;
        param.nDim = nDim;
        param.pad = pad;
        param.siteSubset = siteSubset;
        param.siteOrder = siteOrder;
        param.fieldOrder = fieldOrder;
        param.gammaBasis = gammaBasis;
        memcpy(param.x, x, nDim*sizeof(int));
        param.create = QUDA_REFERENCE_FIELD_CREATE;
 
        param.eigv_dim  = range;
        param.eigv_id   = -1;
        param.v = (void*)((char*)v + first_element*eigv_bytes);
        param.norm = (void*)((char*)norm + first_element*eigv_norm_bytes);

        eigv_subset = new cudaColorSpinorField(param);

        //Not really needed:
        eigv_subset->eigenvectors.reserve(param.eigv_dim);
        for(int id = first_element; id < (first_element+range); id++)
        {
            param.eigv_id = id;
            eigv_subset->eigenvectors.push_back(new cudaColorSpinorField(*this, param));
        }
        checkField(dst, *eigv_subset);
        copyCuda(dst, *eigv_subset);

        delete eigv_subset;
      }
      else{
        errorQuda("Incorrect eigenvector dimension...");
      }
    }
    else{  
      errorQuda("Eigenvector must be a parity spinor");
      exit(-1);
    }
#endif
  }

  void cudaColorSpinorField::getTexObjectInfo() const
  {
#ifdef USE_TEXTURE_OBJECTS
    printfQuda("\nPrint texture info for the field:\n");
    std::cout << *this;
    hipResourceDesc resDesc;
    //memset(&resDesc, 0, sizeof(resDesc));
    hipGetTextureObjectResourceDesc(&resDesc, this->Tex());
    printfQuda("\nDevice pointer: %p\n", resDesc.res.linear.devPtr);
    printfQuda("\nVolume (in bytes): %d\n", resDesc.res.linear.sizeInBytes);
    if (resDesc.resType == hipResourceTypeLinear) printfQuda("\nResource type: linear \n");
    checkCudaError();
#endif
  }

} // namespace quda
