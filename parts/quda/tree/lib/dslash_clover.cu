#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

namespace quda {

  namespace clover {

#undef GPU_STAGGERED_DIRAC // do not delete - hack for Tesla architecture
#define GPU_DOMAIN_WALL_DIRAC // do not delete - work around for CUDA 6.5 alignment bug

#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

    // Enable shared memory dslash for Fermi architecture
    //#define SHARED_WILSON_DSLASH
    //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#ifdef GPU_CLOVER_DIRAC
#define DD_CLOVER 1
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#undef DD_CLOVER
#endif

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>

  } // end namespace clover

  // declare the dslash events
#include <dslash_events.cuh>

  using namespace clover;

#ifdef GPU_CLOVER_DIRAC
  template <typename sFloat, typename gFloat, typename cFloat>
  class CloverDslashCuda : public SharedDslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const cFloat *clover;
    const float *cloverNorm;
    const double a;

  protected:
    unsigned int sharedBytesPerThread() const
    {
      if (dslashParam.kernel_type == INTERIOR_KERNEL) {
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
      } else {
	return 0;
      }
    }
  public:
    CloverDslashCuda(cudaColorSpinorField *out,  const gFloat *gauge0, const gFloat *gauge1, 
		     const QudaReconstructType reconstruct, const cFloat *clover, 
		     const float *cloverNorm, int cl_stride, const cudaColorSpinorField *in, 
		     const cudaColorSpinorField *x, const double a, const int dagger)
      : SharedDslashCuda(out, in, x, reconstruct, dagger), gauge0(gauge0), gauge1(gauge1), clover(clover),
	cloverNorm(cloverNorm), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x);
      dslashParam.cl_stride = cl_stride;
    }
    virtual ~CloverDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    void apply(const hipStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
	errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      DSLASH(cloverDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	     (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, clover, cloverNorm, 
	     (sFloat*)in->V(), (float*)in->Norm(), (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a);
    }

    long long flops() const {
      int clover_flops = 504;
      long long flops = DslashCuda::flops();
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
	flops += clover_flops * in->GhostFace()[dslashParam.kernel_type];
	break;
      case EXTERIOR_KERNEL_ALL:
	flops += clover_flops * 2 * (in->GhostFace()[0]+in->GhostFace()[1]+in->GhostFace()[2]+in->GhostFace()[3]);
	break;
      case INTERIOR_KERNEL:
	flops += clover_flops * in->VolumeCB();	  

	// now correct for flops done by exterior kernel
	long long ghost_sites = 0;
	for (int d=0; d<4; d++) if (dslashParam.commDim[d]) ghost_sites += 2 * in->GhostFace()[d];
	flops -= clover_flops * ghost_sites;
	
	break;
      }
      return flops;
    }

    long long bytes() const {
      bool isHalf = in->Precision() == sizeof(short) ? true : false;
      int clover_bytes = 72 * in->Precision() + (isHalf ? 2*sizeof(float) : 0);

      long long bytes = DslashCuda::bytes();
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
	bytes += clover_bytes * 2 * in->GhostFace()[dslashParam.kernel_type];
	break;
      case EXTERIOR_KERNEL_ALL:
	bytes += clover_bytes * 2 * (in->GhostFace()[0]+in->GhostFace()[1]+in->GhostFace()[2]+in->GhostFace()[3]);
	break;
      case INTERIOR_KERNEL:
	bytes += clover_bytes*in->VolumeCB();

	// now correct for bytes done by exterior kernel
	long long ghost_sites = 0;
	for (int d=0; d<4; d++) if (dslashParam.commDim[d]) ghost_sites += 2*in->GhostFace()[d];
	bytes -= clover_bytes * ghost_sites;
	
	break;
      }

      return bytes;
    }

  };
#endif // GPU_CLOVER_DIRAC

#include <dslash_policy.cuh>

  void cloverDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const FullClover cloverInv,
			const cudaColorSpinorField *in, const int parity, const int dagger, 
			const cudaColorSpinorField *x, const double &a, const int *commOverride,
			TimeProfile &profile, const QudaDslashPolicy &dslashPolicy)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_CLOVER_DIRAC
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }

    void *cloverP, *cloverNormP;
    QudaPrecision clover_prec = bindCloverTex(cloverInv, parity, &cloverP, &cloverNormP);

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge and spinor precision not supported");

    if (in->Precision() != clover_prec)
      errorQuda("Mixing clover and spinor precision not supported");

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new CloverDslashCuda<double2, double2, double2>
	(out, (double2*)gauge0, (double2*)gauge1, gauge.Reconstruct(), 
	 (double2*)cloverP, (float*)cloverNormP, cloverInv.stride, in, x, a, dagger);
      regSize = sizeof(double);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new CloverDslashCuda<float4, float4, float4>
	(out, (float4*)gauge0, (float4*)gauge1, gauge.Reconstruct(), 
	 (float4*)cloverP, (float*)cloverNormP, cloverInv.stride, in, x, a, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new CloverDslashCuda<short4, short4, short4>
	(out, (short4*)gauge0, (short4*)gauge1, gauge.Reconstruct(), 
	 (short4*)cloverP, (float*)cloverNormP, cloverInv.stride, in, x, a, dagger);
    }

#ifndef GPU_COMMS
    DslashPolicyImp* dslashImp = DslashFactory::create(dslashPolicy);
#else
    DslashPolicyImp* dslashImp = DslashFactory::create(QUDA_GPU_COMMS_DSLASH);
#endif
    (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), regSize, parity, dagger, in->Volume(), in->GhostFace(), profile);
    delete dslashImp;

    delete dslash;
    unbindGaugeTex(gauge);
    unbindCloverTex(cloverInv);

    checkCudaError();
#else
    errorQuda("Clover dslash has not been built");
#endif

  }

}
