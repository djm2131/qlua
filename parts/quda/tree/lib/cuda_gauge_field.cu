#include "hip/hip_runtime.h"
#include <string.h>
#include <gauge_field.h>
#include <face_quda.h>
#include <typeinfo>
#include <misc_helpers.h>
#include <blas_quda.h>

namespace quda {

  cudaGaugeField::cudaGaugeField(const GaugeFieldParam &param) :
    GaugeField(param), gauge(0), even(0), odd(0), backed_up(false)
  {
    if ((order == QUDA_QDP_GAUGE_ORDER || order == QUDA_QDPJIT_GAUGE_ORDER) && 
        create != QUDA_REFERENCE_FIELD_CREATE) {
      errorQuda("QDP ordering only supported for reference fields");
    }

    if (order == QUDA_QDP_GAUGE_ORDER || order == QUDA_MILC_GAUGE_ORDER ||
	order == QUDA_TIFR_GAUGE_ORDER || order == QUDA_BQCD_GAUGE_ORDER ||
	order == QUDA_CPS_WILSON_GAUGE_ORDER) 
      errorQuda("Field ordering %d presently disabled for this type", order);

#ifdef MULTI_GPU
    if (link_type != QUDA_ASQTAD_MOM_LINKS &&
	ghostExchange == QUDA_GHOST_EXCHANGE_PAD && 
	isNative()) {
      bool pad_check = true;
      for (int i=0; i<nDim; i++)
	if (pad < nFace*surfaceCB[i]) pad_check = false;
      if (!pad_check)
	errorQuda("cudaGaugeField being constructed with insufficient padding\n");
    }
#endif

    if(create != QUDA_NULL_FIELD_CREATE &&  
        create != QUDA_ZERO_FIELD_CREATE && 
        create != QUDA_REFERENCE_FIELD_CREATE){
      errorQuda("ERROR: create type(%d) not supported yet\n", create);
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      gauge = device_malloc(bytes);  
      if (create == QUDA_ZERO_FIELD_CREATE) hipMemset(gauge, 0, bytes);
    } else { 
      gauge = param.gauge;
    }

    if ( !isNative() ) {
      for (int i=0; i<nDim; i++) {
        size_t nbytes = nFace * surface[i] * reconstruct * precision;
        ghost[i] = nbytes ? device_malloc(nbytes) : NULL;
      }        
    }

    if (ghostExchange == QUDA_GHOST_EXCHANGE_PAD) {
      if (create == QUDA_REFERENCE_FIELD_CREATE) exchangeGhost(); 
    }

    even = gauge;
    odd = (char*)gauge + bytes/2; 

#ifdef USE_TEXTURE_OBJECTS
    createTexObject(evenTex, even);
    createTexObject(oddTex, odd);
    if(reconstruct == QUDA_RECONSTRUCT_13 || reconstruct == QUDA_RECONSTRUCT_9)
    {  // Create texture objects for the phases
      const int isPhase = 1;
      createTexObject(evenPhaseTex, (char*)even + phase_offset, isPhase);
      createTexObject(oddPhaseTex, (char*)odd + phase_offset, isPhase);
    }
#endif

  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaGaugeField::createTexObject(hipTextureObject_t &tex, void *field, int isPhase) {

    if( isNative() ){
      // create the texture for the field components
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // half is short, double is int2

      if(isPhase){
        if(precision == QUDA_DOUBLE_PRECISION){
          desc.x = 8*sizeof(int);
          desc.y = 8*sizeof(int);
          desc.z = 0;
          desc.w = 0;
        }else{
          desc.x = 8*precision;
          desc.y = desc.z = desc.w = 0;
        }
      }else{
        // always four components regardless of precision
        if (precision == QUDA_DOUBLE_PRECISION) {
          desc.x = 8*sizeof(int);
          desc.y = 8*sizeof(int);
          desc.z = 8*sizeof(int);
          desc.w = 8*sizeof(int);
        } else {
          desc.x = 8*precision;
          desc.y = 8*precision;
          desc.z = (reconstruct == 18) ? 0 : 8*precision; // float2 or short2 for 18 reconstruct
          desc.w = (reconstruct == 18) ? 0 : 8*precision;
        }
      }

      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = field;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = isPhase ? phase_bytes/2 : (bytes-phase_bytes)/2;

      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;

      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
      checkCudaError();
    }
  }

  void cudaGaugeField::destroyTexObject() {
    if( isNative() ){
      hipDestroyTextureObject(evenTex);
      hipDestroyTextureObject(oddTex);
      if(reconstruct == QUDA_RECONSTRUCT_9 || reconstruct == QUDA_RECONSTRUCT_13){
        hipDestroyTextureObject(evenPhaseTex);
        hipDestroyTextureObject(oddPhaseTex);
      }
      checkCudaError();
    }
  }
#endif

  cudaGaugeField::~cudaGaugeField()
  {
#ifdef USE_TEXTURE_OBJECTS
    destroyTexObject();
#endif

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (gauge) device_free(gauge);
    }

    if ( !isNative() ) {
      for (int i=0; i<nDim; i++) {
        if (ghost[i]) device_free(ghost[i]);
      }
    }

  }

  // This does the exchange of the gauge field ghost zone and places it
  // into the ghost array.
  void cudaGaugeField::exchangeGhost() {
    if (ghostExchange != QUDA_GHOST_EXCHANGE_PAD)
      errorQuda("Cannot call exchangeGhost with ghostExchange=%d", 
		ghostExchange);

    if (geometry != QUDA_VECTOR_GEOMETRY) 
      errorQuda("Cannot exchange for %d geometry gauge field", geometry);

    void *ghost_[QUDA_MAX_DIM];
    void *send[QUDA_MAX_DIM];
    for (int d=0; d<nDim; d++) {
      ghost_[d] = isNative() ? device_malloc(nFace*surface[d]*reconstruct*precision) : ghost[d];
      send[d] = device_malloc(nFace*surface[d]*reconstruct*precision);
    }

    // get the links into contiguous buffers
    extractGaugeGhost(*this, send);

    // communicate between nodes
    FaceBuffer faceBuf(x, nDim, reconstruct, nFace, precision);
    faceBuf.exchangeLink(ghost_, send, QUDA_CUDA_FIELD_LOCATION);

    for (int d=0; d<nDim; d++) device_free(send[d]);

    if (isNative()) {
      // copy from ghost into the padded region in gauge
      copyGenericGauge(*this, *this, QUDA_CUDA_FIELD_LOCATION, 0, 0, 0, ghost_, 1);
      for (int d=0; d<nDim; d++) device_free(ghost_[d]);
    }
  }

  void cudaGaugeField::exchangeExtendedGhost(const int *R, bool no_comms_fill) {
    
    void *send[QUDA_MAX_DIM];
    void *recv[QUDA_MAX_DIM];
    void *send_d[QUDA_MAX_DIM];
    void *recv_d[QUDA_MAX_DIM];
    size_t bytes[QUDA_MAX_DIM];

    for (int d=0; d<nDim; d++) {
      if (!commDimPartitioned(d) && !no_comms_fill) continue;
      // store both parities and directions in each
      bytes[d] = surface[d] * R[d] * geometry * reconstruct * precision;
      send_d[d] = device_malloc(2 * bytes[d]);
      recv_d[d] = device_malloc(2 * bytes[d]);
    }

#ifndef GPU_COMMS
    void *send_h[QUDA_MAX_DIM];
    void *recv_h[QUDA_MAX_DIM];
    size_t total_bytes = 0;
    for (int d=0; d<nDim; d++) {
      if (!commDimPartitioned(d)) continue;
      total_bytes += 4*bytes[d];
    }
    resizeBufferPinned(total_bytes,0);


    size_t offset = 0;
    for (int d=0; d<nDim; d++) {
      if (!commDimPartitioned(d)) continue;

      recv_h[d] = static_cast<char*>(bufferPinned[0]) + offset;
      send_h[d] = static_cast<char*>(recv_h[d]) + 2*bytes[d];
      offset += 4*bytes[d];
    }
#endif

    // do the exchange
    MsgHandle *mh_recv_back[QUDA_MAX_DIM];
    MsgHandle *mh_recv_fwd[QUDA_MAX_DIM];
    MsgHandle *mh_send_fwd[QUDA_MAX_DIM];
    MsgHandle *mh_send_back[QUDA_MAX_DIM];

    for (int d=0; d<nDim; d++) {
      if (!commDimPartitioned(d)) continue;
#ifdef GPU_COMMS
      recv[d] = recv_d[d];
      send[d] = send_d[d];
#else
      recv[d] = recv_h[d];
      send[d] = send_h[d];
#endif

      // look into storing these for later
      mh_recv_back[d] = comm_declare_receive_relative(recv[d], d, -1, bytes[d]);
      mh_recv_fwd[d]  = comm_declare_receive_relative(static_cast<char*>(recv[d])+bytes[d], 
						      d, +1, bytes[d]);
      mh_send_back[d] = comm_declare_send_relative(send[d], d, -1, bytes[d]);
      mh_send_fwd[d]  = comm_declare_send_relative(static_cast<char*>(send[d])+bytes[d], 
						   d, +1, bytes[d]);
    }

    for (int d=0; d<nDim; d++) {
      if (!commDimPartitioned(d) && !no_comms_fill) continue;

      // FIXME why does this break if the order is switched?
      // prepost the receives
      if (commDimPartitioned(d)) {
	comm_start(mh_recv_fwd[d]);
	comm_start(mh_recv_back[d]);
      }

      //extract into a contiguous buffer
      extractExtendedGaugeGhost(*this, d, R, send_d, true);

      if (commDimPartitioned(d)) {
	
	// pipeline the forwards and backwards sending
#ifndef GPU_COMMS
	hipMemcpyAsync(send_h[d], send_d[d], bytes[d], hipMemcpyDeviceToHost, streams[0]);
	hipMemcpyAsync(static_cast<char*>(send_h[d])+bytes[d], 
			static_cast<char*>(send_d[d])+bytes[d], bytes[d], hipMemcpyDeviceToHost, streams[1]);
#endif      
	
#ifndef GPU_COMMS
	hipStreamSynchronize(streams[0]);
#endif
	comm_start(mh_send_back[d]);
	
#ifndef GPU_COMMS
	hipStreamSynchronize(streams[1]);
#endif
	comm_start(mh_send_fwd[d]);
	
	// forwards recv
	comm_wait(mh_send_back[d]);
	comm_wait(mh_recv_fwd[d]);
#ifndef GPU_COMMS
	hipMemcpyAsync(static_cast<char*>(recv_d[d])+bytes[d], 
			static_cast<char*>(recv_h[d])+bytes[d], bytes[d], hipMemcpyHostToDevice, streams[0]);
#endif      
	
	// backwards recv
	comm_wait(mh_send_fwd[d]);
	comm_wait(mh_recv_back[d]);
#ifndef GPU_COMMS
	hipMemcpyAsync(recv_d[d], recv_h[d], bytes[d], hipMemcpyHostToDevice, streams[1]);
#endif      
      } else { // if just doing a local exchange to fill halo then need to swap faces
	hipMemcpy(static_cast<char*>(recv_d[d])+bytes[d], send_d[d], bytes[d], hipMemcpyDeviceToDevice);
	hipMemcpy(recv_d[d], static_cast<char*>(send_d[d])+bytes[d], bytes[d], hipMemcpyDeviceToDevice);
      }

      // inject back into the gauge field
      extractExtendedGaugeGhost(*this, d, R, recv_d, false);
    }

    for (int d=0; d<nDim; d++) {
      if (!commDimPartitioned(d) && !no_comms_fill) continue;

      if (commDimPartitioned(d)) {
	comm_free(mh_send_fwd[d]);
	comm_free(mh_send_back[d]);
	comm_free(mh_recv_back[d]);
	comm_free(mh_recv_fwd[d]);
      }

      device_free(send_d[d]);
      device_free(recv_d[d]);
    }

  }

  void cudaGaugeField::setGauge(void *gauge_)
  {
    if(create != QUDA_REFERENCE_FIELD_CREATE) {
      errorQuda("Setting gauge pointer is only allowed when create="
          "QUDA_REFERENCE_FIELD_CREATE type\n");
    }
    gauge = gauge_;
  }

  void cudaGaugeField::copy(const GaugeField &src) {
    if (this == &src) return;

    checkField(src);

    if (link_type == QUDA_ASQTAD_FAT_LINKS) {
      fat_link_max = src.LinkMax();
      if (precision == QUDA_HALF_PRECISION && fat_link_max == 0.0) 
        errorQuda("fat_link_max has not been computed");
    } else {
      fat_link_max = 1.0;
    }

    if (typeid(src) == typeid(cudaGaugeField)) {
      // copy field and ghost zone into this field
      copyGenericGauge(*this, src, QUDA_CUDA_FIELD_LOCATION, gauge, 
          static_cast<const cudaGaugeField&>(src).gauge);
    } else if (typeid(src) == typeid(cpuGaugeField)) {
      LatticeField::resizeBufferPinned(bytes,0);

      // copy field and ghost zone into bufferPinned
      copyGenericGauge(*this, src, QUDA_CPU_FIELD_LOCATION, bufferPinned[0], 
		       static_cast<const cpuGaugeField&>(src).gauge); 

      // this copies over both even and odd
      hipMemcpy(gauge, bufferPinned[0], bytes, hipMemcpyHostToDevice);
    } else {
      errorQuda("Invalid gauge field type");
    }

    // if we have copied from a source without a pad then we need to exchange
    if (ghostExchange == QUDA_GHOST_EXCHANGE_PAD &&
	src.GhostExchange() != QUDA_GHOST_EXCHANGE_PAD) {
      exchangeGhost(); 
    }

    checkCudaError();
  }

  void cudaGaugeField::loadCPUField(const cpuGaugeField &cpu, const QudaFieldLocation &pack_location)
  {
    if (pack_location == QUDA_CUDA_FIELD_LOCATION) {
      if (cpu.Order() == QUDA_MILC_GAUGE_ORDER ||
	  cpu.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {
	resizeBufferPinned(cpu.Bytes(),0);
	memcpy(bufferPinned[0], cpu.Gauge_p(), cpu.Bytes());

	// run kernel directly using host-mapped input data
	void *bufferPinnedMapped;
	hipHostGetDevicePointer(&bufferPinnedMapped, bufferPinned[0], 0);
	copyGenericGauge(*this, cpu, QUDA_CUDA_FIELD_LOCATION, gauge, bufferPinnedMapped);
      } else {
	errorQuda("Not implemented for order %d", cpu.Order());
      }
    } else if (pack_location == QUDA_CPU_FIELD_LOCATION) {
      copy(cpu);
    } else {
      errorQuda("Invalid pack location %d", pack_location);
    }

  }

  /*
     Copies the device gauge field to the host.
     - no reconstruction support
     - device data is always Float2 ordered
     - host data is a 1-dimensional array (MILC ordered)
     - no support for half precision
     - input and output precisions must match
   */
  template<typename FloatN, typename Float>
    static void storeGaugeField(Float* cpuGauge, FloatN *gauge, int bytes, int volumeCB, 
        int stride, QudaPrecision prec) 
    {  
      hipStream_t streams[2];
      for (int i=0; i<2; i++) hipStreamCreate(&streams[i]);

      FloatN *even = gauge;
      FloatN *odd = (FloatN*)((char*)gauge + bytes/2);

      size_t datalen = 4*2*volumeCB*gaugeSiteSize*sizeof(Float); // both parities
      void *unpacked = device_malloc(datalen);
      void *unpackedEven = unpacked;
      void *unpackedOdd = (char*)unpacked + datalen/2;

      //unpack even data kernel
      link_format_gpu_to_cpu((void*)unpackedEven, (void*)even, volumeCB, stride, prec, streams[0]);
#ifdef GPU_DIRECT
      hipMemcpyAsync(cpuGauge, unpackedEven, datalen/2, hipMemcpyDeviceToHost, streams[0]);
#else
      hipMemcpy(cpuGauge, unpackedEven, datalen/2, hipMemcpyDeviceToHost);
#endif

      //unpack odd data kernel
      link_format_gpu_to_cpu((void*)unpackedOdd, (void*)odd, volumeCB, stride, prec, streams[1]);
#ifdef GPU_DIRECT
      hipMemcpyAsync(cpuGauge + 4*volumeCB*gaugeSiteSize, unpackedOdd, datalen/2, hipMemcpyDeviceToHost, streams[1]);  
      for(int i=0; i<2; i++) hipStreamSynchronize(streams[i]);
#else
      hipMemcpy(cpuGauge + 4*volumeCB*gaugeSiteSize, unpackedOdd, datalen/2, hipMemcpyDeviceToHost);  
#endif

      device_free(unpacked);
      for(int i=0; i<2; i++) hipStreamDestroy(streams[i]);
    }

  void cudaGaugeField::saveCPUField(cpuGaugeField &cpu, const QudaFieldLocation &pack_location) const
  {
    // FIXME use the generic copying for the below copying
    // do device-side reordering then copy
    if (pack_location == QUDA_CUDA_FIELD_LOCATION) {
      // check parameters are suitable for device-side packing
      if (precision != cpu.Precision())
        errorQuda("cpu precision %d and cuda precision %d must be the same", 
            cpu.Precision(), precision);

      if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Only no reconstruction supported");
      if (order != QUDA_FLOAT2_GAUGE_ORDER) errorQuda("Only QUDA_FLOAT2_GAUGE_ORDER supported");
      if (cpu.Order() != QUDA_MILC_GAUGE_ORDER) errorQuda("Only QUDA_MILC_GAUGE_ORDER supported");

      if (precision == QUDA_DOUBLE_PRECISION){
        storeGaugeField((double*)cpu.gauge, (double2*)gauge, bytes, volumeCB, stride, precision);
      } else if (precision == QUDA_SINGLE_PRECISION){
        storeGaugeField((float*)cpu.gauge, (float2*)gauge, bytes, volumeCB, stride, precision);
      } else {
        errorQuda("Half precision not supported");
      }

    } else if (pack_location == QUDA_CPU_FIELD_LOCATION) { // do copy then host-side reorder
      resizeBufferPinned(bytes,0);

      // this copies over both even and odd
      hipMemcpy(bufferPinned[0], gauge, bytes, hipMemcpyDeviceToHost);
      checkCudaError();

      copyGenericGauge(cpu, *this, QUDA_CPU_FIELD_LOCATION, cpu.gauge, bufferPinned[0]);
    } else {
      errorQuda("Invalid pack location %d", pack_location);
    }

  }

  void cudaGaugeField::backup() const {
    if (backed_up) errorQuda("Gauge field already backed up");
    backup_h = new char[bytes];
    hipMemcpy(backup_h, gauge, bytes, hipMemcpyDeviceToHost);
    checkCudaError();
    backed_up = true;
  }

  void cudaGaugeField::restore() {
    if (!backed_up) errorQuda("Cannot restore since not backed up");
    hipMemcpy(gauge, backup_h, bytes, hipMemcpyHostToDevice);
    delete []backup_h;
    checkCudaError();
    backed_up = false;
  }

  void cudaGaugeField::zero() {
    hipMemset(gauge, 0, bytes);
  }

  void setGhostSpinor(bool value);

  ColorSpinorParam colorSpinorParam(const cudaGaugeField &a) {
   if (a.FieldOrder() == QUDA_QDP_GAUGE_ORDER || 
        a.FieldOrder() == QUDA_QDPJIT_GAUGE_ORDER)
      errorQuda("Not implemented");

    int spin = 0;
    switch (a.Geometry()) {
      case QUDA_SCALAR_GEOMETRY:
        spin = 1;
        break;
      case QUDA_VECTOR_GEOMETRY:
        spin = a.Ndim();
        break;
      default:
        errorQuda("Unsupported field geometry %d", a.Geometry());
    }

    if (a.Precision() == QUDA_HALF_PRECISION) 
      errorQuda("Casting a cudaGaugeField into cudaColorSpinorField not possible in half precision");

    if (a.Reconstruct() == QUDA_RECONSTRUCT_13 || a.Reconstruct() == QUDA_RECONSTRUCT_9)
      errorQuda("Unsupported field reconstruct %d", a.Reconstruct());

    ColorSpinorParam spinor_param;
    spinor_param.nColor = a.Reconstruct()/2;
    spinor_param.nSpin = spin;
    spinor_param.nDim = a.Ndim();
    for (int d=0; d<a.Ndim(); d++) spinor_param.x[d] = a.X()[d];
    spinor_param.precision = a.Precision();
    spinor_param.pad = a.Pad();
    spinor_param.siteSubset = QUDA_FULL_SITE_SUBSET;
    spinor_param.siteOrder = QUDA_EVEN_ODD_SITE_ORDER;
    spinor_param.fieldOrder = (a.Precision() == QUDA_DOUBLE_PRECISION || spinor_param.nSpin == 1) ? 
    QUDA_FLOAT2_FIELD_ORDER : QUDA_FLOAT4_FIELD_ORDER; 
    spinor_param.gammaBasis = QUDA_UKQCD_GAMMA_BASIS;
    spinor_param.create = QUDA_REFERENCE_FIELD_CREATE;
    spinor_param.v = (void*)a.Gauge_p();
    return spinor_param;
  }

  // Return the L2 norm squared of the gauge field
  double norm2(const cudaGaugeField &a) {
    // quick hack to disable ghost zone creation which otherwise breaks this mapping on multi-gpu
    setGhostSpinor(false);
    cudaColorSpinorField b(colorSpinorParam(a));
    setGhostSpinor(true);

    return norm2(b);
  }

  // Return the L1 norm of the gauge field
  double norm1(const cudaGaugeField &a) {
    // quick hack to disable ghost zone creation which otherwise breaks this mapping on multi-gpu
    setGhostSpinor(false);
    cudaColorSpinorField b(colorSpinorParam(a));
    setGhostSpinor(true);

    return norm1Cuda(b);
  }

} // namespace quda
