#include "hip/hip_runtime.h"

#include <quda_internal.h>
#include <misc_helpers.h>
#define gaugeSiteSize 18
#define BLOCKSIZE 64



/*
 * MILC order, CPU->GPU
 *
 *This function converts format in CPU form 
 * into forms in GPU so as to enable coalesce access
 * The function only converts half(even or odd) of the links
 * Therefore the entire link conversion need to call this 
 * function twice
 *   
 * Without loss of generarity, the parity is assume to be even.
 * The actual data format in cpu is following
 * [a0a1 .... a17][b0b1...b17][c..][d...][a18a19 .....a35] ...[b0b1 ... b17] ...
 *  X links        Y links    T,Z links   X Links
 * where a0->a17 is the X link in the first site
 *       b0->b17 is the Y link in the first site
 *       c0->c17 is the Z link in the first site
 *       d0->d17 is the T link in the first site
 *       a18->a35 is the X link in the second site
 *       etc
 *
 * The GPU format of data looks like the following
 * [a0a1][a18a19]  ....[pad][a2a3][a20a21]..... [b0b1][b18b19]....
 *  X links                                      Y links      T,Z links
 *
 * N: # of FloatN in one gauge field
 *    9 for QUDA_RECONSTRUCT_NO, SP/DP
 *    6 for QUDA_RECONSTRUCT_12, DP
 *    3 for QUDA_RECONSTRUCT_12, SP
 */

namespace quda {

  template<int N, typename FloatN, typename Float2>
  __global__ void
  do_link_format_cpu_to_gpu(FloatN* dst, Float2* src,
			    int reconstruct,
			    int Vh, int pad, int ghostV, size_t threads)
  {
    int tid = blockIdx.x * blockDim.x +  threadIdx.x;
    int thread0_tid = blockIdx.x * blockDim.x;
    __shared__ FloatN buf[N*BLOCKSIZE];
  
    int dir;
    int j;
    int stride = Vh+pad;
    for(dir = 0; dir < 4; dir++){
#ifdef MULTI_GPU
      Float2* src_start = src + dir*9*(Vh+ghostV) + thread0_tid*9;   
#else
      Float2* src_start = src + dir*9*(Vh) + thread0_tid*9;   
#endif
      for(j=0; j < 9; j++){
	if(thread0_tid*9+j*blockDim.x+threadIdx.x >= 9*threads) break;
	if( N == 9){
	  ((Float2*)buf)[j*blockDim.x + threadIdx.x] =  src_start[j*blockDim.x + threadIdx.x];
	}else{ 
	  int idx = j*blockDim.x + threadIdx.x;
	  int modval = idx % 9;
	  int divval = idx / 9;
	  if(modval < 6){
	    ((Float2*)buf)[divval*6+modval] = src_start[idx];
	  }
	
	}
      }
    
      __syncthreads();
      if(tid < threads){
	FloatN* dst_start = (FloatN*)(dst+dir*N*stride);
	for(j=0; j < N; j++){
	  dst_start[tid + j*stride] = buf[N*threadIdx.x + j];
	}
      }
      __syncthreads();
    }//dir
  }


  /*
   *
   * N: # of FloatN in one gauge field
   *    9 for QUDA_RECONSTRUCT_NO, SP/DP
   *    6 for QUDA_RECONSTRUCT_12, DP
   *    3 for QUDA_RECONSTRUCT_12, SP
   *
   * FloatN: float2/double2
   * Float: float/double
   *
   * This is the reverse process for the function do_link_format_gpu_to_cpu()
   *
   */

  template<int N, typename FloatN, typename Float2>
  __global__ void
  do_link_format_cpu_to_gpu_milc(FloatN* dst, Float2* src,
				 int reconstruct,
				 int Vh, int pad, int ghostV, size_t threads)
  {
  
    __shared__ FloatN buf[N*BLOCKSIZE];
    int block_idx = blockIdx.x*blockDim.x/4;
    int local_idx = 16*(threadIdx.x/64) + threadIdx.x%16;
    int pos_idx = blockIdx.x * blockDim.x/4 + 16*(threadIdx.x/64) + threadIdx.x%16;
    int mydir = (threadIdx.x >> 4)% 4;
    int j;
    int stride = Vh+pad;
  
    for(j=0; j < 9; j++){
      if(block_idx*9*4 + j*blockDim.x+threadIdx.x >= 9*threads) break;
      if(N == 9){
	((Float2*)buf)[j*blockDim.x + threadIdx.x] = src[block_idx*9*4 + j*blockDim.x + threadIdx.x]; 
      }else{ 
	int idx = j*blockDim.x + threadIdx.x;
	int modval = idx % 9;
	int divval = idx / 9;
	if(modval < 6){
	  ((Float2*)buf)[divval*6+modval] = src[block_idx*9*4 + idx];
	}
      }
    }  
  
    __syncthreads();
  
    if(pos_idx >= threads/4) return;
  
    for(j=0; j < N; j++){
      if(N == 9){
	dst[pos_idx + mydir*N*stride + j*stride] = buf[local_idx*4*9+mydir*9+j];
      }else{
	dst[pos_idx + mydir*N*stride + j*stride] = buf[local_idx*4*N+mydir*N+j];      
      }
    }      
  }

  void 
  link_format_cpu_to_gpu(void* dst, void* src, 
			 int reconstruct, int Vh, int pad, 
			 int ghostV,
			 QudaPrecision prec, QudaGaugeFieldOrder cpu_order, 
			 hipStream_t stream)
  {
    dim3 blockDim(BLOCKSIZE);

    if(cpu_order ==  QUDA_QDP_GAUGE_ORDER){
#ifdef MULTI_GPU  
      size_t threads=Vh+ghostV;
#else
      size_t threads=Vh;
#endif    
      dim3 gridDim ((threads + BLOCKSIZE -1)/BLOCKSIZE);

      switch (prec){
      case QUDA_DOUBLE_PRECISION:
	switch( reconstruct){
	case QUDA_RECONSTRUCT_NO:
	  do_link_format_cpu_to_gpu<9><<<gridDim, blockDim, 0, stream>>>((double2*)dst, (double2*)src, reconstruct, Vh, pad, ghostV, threads);
	  break;
	case QUDA_RECONSTRUCT_12:
	  do_link_format_cpu_to_gpu<6><<<gridDim, blockDim, 0, stream>>>((double2*)dst, (double2*)src, reconstruct, Vh, pad, ghostV, threads);
	  break;
	default:
	  errorQuda("reconstruct type not supported\n");
	}
	break;    
      
      case QUDA_SINGLE_PRECISION:
	switch( reconstruct){
	case QUDA_RECONSTRUCT_NO:
	  do_link_format_cpu_to_gpu<9><<<gridDim, blockDim, 0, stream>>>((float2*)dst, (float2*)src, reconstruct,  Vh, pad, ghostV, threads);   
	  break;
	case QUDA_RECONSTRUCT_12:
	  do_link_format_cpu_to_gpu<3><<<gridDim, blockDim>>>((float4*)dst, (float2*)src, reconstruct, Vh, pad, ghostV, threads);   
	  break;
	default:
	  errorQuda("reconstruct type not supported\n");      
	}
	break;
      
      default:
	errorQuda("ERROR: half precision not support in %s\n", __FUNCTION__);
      }
    } else if (cpu_order == QUDA_MILC_GAUGE_ORDER){    
#ifdef MULTI_GPU  
      int threads=4*(Vh+ghostV);
#else
      int threads=4*Vh;
#endif  
      dim3 gridDim ((threads + BLOCKSIZE -1)/BLOCKSIZE);

      switch (prec){
      case QUDA_DOUBLE_PRECISION:
	switch( reconstruct){
	case QUDA_RECONSTRUCT_NO:
	  do_link_format_cpu_to_gpu_milc<9><<<gridDim, blockDim, 0, stream>>>((double2*)dst, (double2*)src, reconstruct, Vh, pad, ghostV, threads);
	  break;
	case QUDA_RECONSTRUCT_12:
	  do_link_format_cpu_to_gpu_milc<6><<<gridDim, blockDim, 0, stream>>>((double2*)dst, (double2*)src, reconstruct, Vh, pad, ghostV, threads);
	  break;
	default:
	  errorQuda("reconstruct type not supported\n");
	}
	break;    
      
      case QUDA_SINGLE_PRECISION:
	switch( reconstruct){
	case QUDA_RECONSTRUCT_NO:
	  do_link_format_cpu_to_gpu_milc<9><<<gridDim, blockDim, 0, stream>>>((float2*)dst, (float2*)src, reconstruct, Vh, pad, ghostV, threads);
	  break;
	case QUDA_RECONSTRUCT_12:
	  do_link_format_cpu_to_gpu_milc<3><<<gridDim, blockDim, 0, stream>>>((float4*)dst, (float2*)src, reconstruct, Vh, pad, ghostV, threads);
	  break;
	default:
	  errorQuda("reconstruct type not supported\n");      
	}
	break;
      
      default:
	errorQuda("ERROR: half precision not support in %s\n", __FUNCTION__);
      }
    
    }else{
      errorQuda("ERROR: invalid cpu ordering (%d)\n", cpu_order);
    }
  
    return;
  
  }
  /*
   * src format: the normal link format in GPU that has stride size @stride
   *	       the src is stored with 9 double2
   * dst format: an array of links where x,y,z,t links with the same node id is stored next to each other
   *             This format is used in destination in fatlink computation in cpu
   *    Without loss of generarity, the parity is assume to be even.
   * The actual data format in GPU is the following
   *    [a0a1][a18a19]  ....[pad][a2a3][a20a21]..... [b0b1][b18b19]....
   *    X links                                      Y links      T,Z links
   * The temporary data store in GPU shared memory and the CPU format of data are the following
   *    [a0a1 .... a17] [b0b1 .....b17] [c0c1 .....c17] [d0d1 .....d17] [a18a19....a35] ....
   *    |<------------------------site 0 ---------------------------->|<----- site 2 ----->
   *
   *
   * In loading phase the indices for all threads in the first block is the following (assume block size is 64)
   * (half warp works on one direction)
   * threadIdx.x	pos_idx		mydir
   * 0		0		0
   * 1		1		0
   * 2		2		0
   * 3		3		0			
   * 4		4		0		
   * 5		5		0
   * 6		6		0
   * 7		7		0
   * 8		8		0
   * 9		9		0
   * 10		10		0
   * 11		11		0
   * 12		12		0
   * 13		13		0
   * 14		14		0
   * 15		15		0
   * 16		0		1
   * 17		1		1
   * 18	       	2		1
   * 19		3		1
   * 20		4		1
   * 21		5		1
   * 22		6		1
   * 23		7		1
   * 24		8		1
   * 25		9		1
   * 26		10		1
   * 27		11		1
   * 28		12		1
   * 29		13		1
   * 30		14		1
   * 31		15		1
   * 32		0		2
   * 33		1		2
   * 34		2		2
   * 35		3		2
   * 36		4		2
   * 37		5		2
   * 38		6		2
   * 39		7		2
   * 40		8		2
   * 41		9		2
   * 42		10		2
   * 43		11		2
   * 44		12		2
   * 45		13		2
   * 46		14		2
   * 47		15		2
   * 48		0		3
   * 49		1		3
   * 50		2		3
   * 51		3		3
   * 52		4		3
   * 53		5		3
   * 54		6		3
   * 55		7		3
   * 56		8		3
   * 57		9		3
   * 58		10		3
   * 59		11		3
   * 60		12		3
   * 61		13		3
   * 62		14		3
   * 63		15		3
   *
   */

  template<typename FloatN>
  __global__ void
  do_link_format_gpu_to_cpu(FloatN* dst, FloatN* src,
			    int Vh, int stride)
  {
    __shared__ FloatN buf[gaugeSiteSize/2*BLOCKSIZE];
  
    int j;
  
    int block_idx = blockIdx.x*blockDim.x/4;
    int local_idx = 16*(threadIdx.x/64) + threadIdx.x%16;
    int pos_idx = blockIdx.x * blockDim.x/4 + 16*(threadIdx.x/64) + threadIdx.x%16;
    int mydir = (threadIdx.x >> 4)% 4;
    for(j=0; j < 9; j++){
      buf[local_idx*4*9+mydir*9+j] = src[pos_idx + mydir*9*stride + j*stride];
    }
    __syncthreads();
  
    for(j=0; j < 9; j++){
      dst[block_idx*9*4 + j*blockDim.x + threadIdx.x ] = buf[j*blockDim.x + threadIdx.x];    
    }  
  
  }



  void 
  link_format_gpu_to_cpu(void* dst, void* src, 
			 int Vh, int stride, QudaPrecision prec, hipStream_t stream)
  {
  
    dim3 blockDim(BLOCKSIZE);
    dim3 gridDim(4*Vh/blockDim.x); //every 4 threads process one site's x,y,z,t links
    //4*Vh must be multipl of BLOCKSIZE or the kernel does not work
    if ((4*Vh) % blockDim.x != 0){
      errorQuda("ERROR: 4*Vh(%d) is not multiple of blocksize(%d), exitting\n", Vh, blockDim.x);
    }
    if(prec == QUDA_DOUBLE_PRECISION){
      do_link_format_gpu_to_cpu<<<gridDim, blockDim, 0, stream>>>((double2*)dst, (double2*)src, Vh, stride);
    }else if(prec == QUDA_SINGLE_PRECISION){
      do_link_format_gpu_to_cpu<<<gridDim, blockDim, 0, stream>>>((float2*)dst, (float2*)src, Vh, stride);
    }else{
      printf("ERROR: half precision is not supported in %s\n",__FUNCTION__);
      exit(1);
    }
  
  }

#define READ_ST_STAPLE(staple, idx, mystride)	\
  Float2 P0 = staple[idx + 0*mystride];		\
  Float2 P1 = staple[idx + 1*mystride];		\
  Float2 P2 = staple[idx + 2*mystride];		\
  Float2 P3 = staple[idx + 3*mystride];		\
  Float2 P4 = staple[idx + 4*mystride];		\
  Float2 P5 = staple[idx + 5*mystride];		\
  Float2 P6 = staple[idx + 6*mystride];		\
  Float2 P7 = staple[idx + 7*mystride];		\
  Float2 P8 = staple[idx + 8*mystride];			

#define WRITE_ST_STAPLE(staple, idx, mystride)	\
  staple[idx + 0*mystride] = P0;		\
  staple[idx + 1*mystride] = P1;		\
  staple[idx + 2*mystride] = P2;		\
  staple[idx + 3*mystride] = P3;		\
  staple[idx + 4*mystride] = P4;		\
  staple[idx + 5*mystride] = P5;		\
  staple[idx + 6*mystride] = P6;		\
  staple[idx + 7*mystride] = P7;		\
  staple[idx + 8*mystride] = P8;			


  struct GhostStapleParam {
    const int in_stride;
    int X[4];
    GhostStapleParam(const int in_stride, const int X[4]) :
      in_stride(in_stride) { 
      for (int i=0 ;i<4; i++) this->X[i] = X[i];
    }
  };


  template<int dir, int whichway, typename Float2>
  __global__ void
  collectGhostStapleKernel(Float2 *out, Float2 *in, int parity, GhostStapleParam param)
  {

    int sid = blockIdx.x*blockDim.x + threadIdx.x;
    int z1 = sid / (param.X[0]>>1);
    int x1h = sid - z1*(param.X[0]>>1);
    int z2 = z1 / param.X[1];
    int x2 = z1 - z2*param.X[1];
    int x4 = z2 / param.X[2];
    int x3 = z2 - x4*param.X[2];
    int x1odd = (x2 + x3 + x4 + parity) & 1;
    int x1 = 2*x1h + x1odd;

    READ_ST_STAPLE(in, sid, param.in_stride);
    int ghost_face_idx;
  
    if ( dir == 0 && whichway == QUDA_BACKWARDS){
      if (x1 < 1){
	ghost_face_idx = (x4*(param.X[2]*param.X[1])+x3*param.X[1] +x2)>>1;
	WRITE_ST_STAPLE(out, ghost_face_idx, param.X[3]*param.X[2]*param.X[1]/2);
      }
    }

    if ( dir == 0 && whichway == QUDA_FORWARDS){
      if (x1 >= param.X[0] - 1){
	ghost_face_idx = (x4*(param.X[2]*param.X[1])+x3*param.X[1] +x2)>>1;
	WRITE_ST_STAPLE(out, ghost_face_idx, param.X[3]*param.X[2]*param.X[1]/2);
      }
    }
  
    if ( dir == 1 && whichway == QUDA_BACKWARDS){
      if (x2 < 1){
	ghost_face_idx = (x4*param.X[2]*param.X[0]+x3*param.X[0]+x1)>>1;
	WRITE_ST_STAPLE(out, ghost_face_idx, param.X[3]*param.X[2]*param.X[0]/2);
      }
    }

    if ( dir == 1 && whichway == QUDA_FORWARDS){
      if (x2 >= param.X[1] - 1){
	ghost_face_idx = (x4*param.X[2]*param.X[0]+x3*param.X[0]+x1)>>1;
	WRITE_ST_STAPLE(out, ghost_face_idx, param.X[3]*param.X[2]*param.X[0]/2);
      }
    }

    if ( dir == 2 && whichway == QUDA_BACKWARDS){
      if (x3 < 1){
	ghost_face_idx = (x4*param.X[1]*param.X[0]+x2*param.X[0]+x1)>>1;
	WRITE_ST_STAPLE(out, ghost_face_idx, param.X[3]*param.X[1]*param.X[0]/2);
      }
    }

    if ( dir == 2 && whichway == QUDA_FORWARDS){
      if (x3 >= param.X[2] - 1){
	ghost_face_idx = (x4*param.X[1]*param.X[0] + x2*param.X[0] + x1)>>1;
	WRITE_ST_STAPLE(out, ghost_face_idx, param.X[3]*param.X[1]*param.X[0]/2);
      }
    }

    if ( dir == 3 && whichway == QUDA_BACKWARDS){
      if (x4 < 1){
	ghost_face_idx = (x3*param.X[1]*param.X[0]+x2*param.X[0]+x1)>>1;
	WRITE_ST_STAPLE(out, ghost_face_idx, param.X[2]*param.X[1]*param.X[0]/2);
      }
    }
  
    if ( dir == 3 && whichway == QUDA_FORWARDS){
      if (x4 >= param.X[3] - 1){
	ghost_face_idx = (x3*param.X[1]*param.X[0]+x2*param.X[0]+x1)>>1;
	WRITE_ST_STAPLE(out, ghost_face_idx, param.X[2]*param.X[1]*param.X[0]/2);
      }
    }

  }

  
  //@dir can be 0, 1, 2, 3 (X,Y,Z,T directions)
  //@whichway can be QUDA_FORWARDS, QUDA_BACKWORDS
  void
  collectGhostStaple(int* X, void* even, void* odd, int volumeCB, int stride, QudaPrecision precision,
		     void* ghost_staple_gpu,		   
		     int dir, int whichway, hipStream_t* stream)
  {
    int Vsh_x, Vsh_y, Vsh_z, Vsh_t;
  
    Vsh_x = X[1]*X[2]*X[3]/2;
    Vsh_y = X[0]*X[2]*X[3]/2;
    Vsh_z = X[0]*X[1]*X[3]/2;
    Vsh_t = X[0]*X[1]*X[2]/2;  
    
    dim3 gridDim(volumeCB/BLOCKSIZE, 1, 1);
    dim3 blockDim(BLOCKSIZE, 1, 1);
    int Vsh[4] = {Vsh_x, Vsh_y, Vsh_z, Vsh_t};
    
    void* gpu_buf_even = ghost_staple_gpu;
    void* gpu_buf_odd = ((char*)ghost_staple_gpu) + Vsh[dir]*gaugeSiteSize*precision ;
    if (X[dir] % 2 ==1){ //need switch even/odd
      gpu_buf_odd = ghost_staple_gpu;
      gpu_buf_even = ((char*)ghost_staple_gpu) + Vsh[dir]*gaugeSiteSize*precision ;    
    }

    int even_parity = 0;
    int odd_parity = 1;
    GhostStapleParam param(stride, X);
  
    if (precision == QUDA_DOUBLE_PRECISION){
      switch(dir){
      case 0:
	switch(whichway){
	case QUDA_BACKWARDS:
	  collectGhostStapleKernel<0, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_even, (double2*)even, even_parity, param);
	  collectGhostStapleKernel<0, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_odd, (double2*)odd, odd_parity, param);
	  break;
	case QUDA_FORWARDS:
	  collectGhostStapleKernel<0, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_even, (double2*)even, even_parity, param);
	  collectGhostStapleKernel<0, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_odd, (double2*)odd, odd_parity, param);
	  break;
	default:
	  errorQuda("Invalid whichway");
	  break;
	}
	break;

      case 1:
	switch(whichway){
	case QUDA_BACKWARDS:
	  collectGhostStapleKernel<1, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_even, (double2*)even, even_parity, param);
	  collectGhostStapleKernel<1, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_odd, (double2*)odd, odd_parity, param);
	  break;
	case QUDA_FORWARDS:
	  collectGhostStapleKernel<1, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_even, (double2*)even, even_parity, param);
	  collectGhostStapleKernel<1, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_odd, (double2*)odd, odd_parity, param);
	  break;
	default:
	  errorQuda("Invalid whichway");
	  break;
	}
	break;
      
      case 2:
	switch(whichway){
	case QUDA_BACKWARDS:
	  collectGhostStapleKernel<2, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_even, (double2*)even, even_parity, param);
	  collectGhostStapleKernel<2, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_odd, (double2*)odd, odd_parity, param);
	  break;
	case QUDA_FORWARDS:
	  collectGhostStapleKernel<2, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_even, (double2*)even, even_parity, param);
	  collectGhostStapleKernel<2, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_odd, (double2*)odd, odd_parity, param);
	  break;
	default:
	  errorQuda("Invalid whichway");
	  break;
	}
	break;
      
      case 3:
	switch(whichway){
	case QUDA_BACKWARDS:
	  collectGhostStapleKernel<3, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_even, (double2*)even, even_parity, param);
	  collectGhostStapleKernel<3, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_odd, (double2*)odd, odd_parity, param);
	  break;
	case QUDA_FORWARDS:
	  collectGhostStapleKernel<3, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_even, (double2*)even, even_parity, param);
	  collectGhostStapleKernel<3, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((double2*)gpu_buf_odd, (double2*)odd, odd_parity, param);
	  break;
	default:
	  errorQuda("Invalid whichway");
	  break;
	}
	break;      
      }
    }else if(precision == QUDA_SINGLE_PRECISION){
      switch(dir){
      case 0:
	switch(whichway){
	case QUDA_BACKWARDS:
	  collectGhostStapleKernel<0, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_even, (float2*)even, even_parity, param);
	  collectGhostStapleKernel<0, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_odd, (float2*)odd, odd_parity, param);
	  break;
	case QUDA_FORWARDS:
	  collectGhostStapleKernel<0, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_even, (float2*)even, even_parity, param);
	  collectGhostStapleKernel<0, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_odd, (float2*)odd, odd_parity, param);
	  break;
	default:
	  errorQuda("Invalid whichway");
	  break;
	}
	break;

      case 1:
	switch(whichway){
	case QUDA_BACKWARDS:
	  collectGhostStapleKernel<1, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_even, (float2*)even, even_parity, param);
	  collectGhostStapleKernel<1, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_odd, (float2*)odd, odd_parity, param);
	  break;
	case QUDA_FORWARDS:
	  collectGhostStapleKernel<1, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_even, (float2*)even, even_parity, param);
	  collectGhostStapleKernel<1, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_odd, (float2*)odd, odd_parity, param);
	  break;
	default:
	  errorQuda("Invalid whichway");
	  break;
	}
	break;
      
      case 2:
	switch(whichway){
	case QUDA_BACKWARDS:
	  collectGhostStapleKernel<2, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_even, (float2*)even, even_parity, param);
	  collectGhostStapleKernel<2, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_odd, (float2*)odd, odd_parity, param);
	  break;
	case QUDA_FORWARDS:
	  collectGhostStapleKernel<2, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_even, (float2*)even, even_parity, param);
	  collectGhostStapleKernel<2, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_odd, (float2*)odd, odd_parity, param);
	  break;
	default:
	  errorQuda("Invalid whichway");
	  break;
	}
	break;
      
      case 3:
	switch(whichway){
	case QUDA_BACKWARDS:
	  collectGhostStapleKernel<3, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_even, (float2*)even, even_parity, param);
	  collectGhostStapleKernel<3, QUDA_BACKWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_odd, (float2*)odd, odd_parity, param);
	  break;
	case QUDA_FORWARDS:
	  collectGhostStapleKernel<3, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_even, (float2*)even, even_parity, param);
	  collectGhostStapleKernel<3, QUDA_FORWARDS><<<gridDim, blockDim, 0, *stream>>>((float2*)gpu_buf_odd, (float2*)odd, odd_parity, param);
	  break;
	default:
	  errorQuda("Invalid whichway");
	  break;
	}
	break;
      }
    }else{
      printf("ERROR: invalid  precision for %s\n", __FUNCTION__);
      exit(1);
    }

  }

} // namespace quda

#undef gaugeSiteSize 
#undef BLOCKSIZE 
