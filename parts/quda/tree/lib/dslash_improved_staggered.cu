#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

//these are access control for staggered action
#ifdef GPU_STAGGERED_DIRAC
#if (__COMPUTE_CAPABILITY__ >= 300) // Kepler works best with texture loads only
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#else // Fermi
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#endif
#endif // GPU_STAGGERED_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

namespace quda {

  namespace improvedstaggered {
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

#undef GPU_NDEG_TWISTED_MASS_DIRAC
#undef GPU_CLOVER_DIRAC
#undef GPU_DOMAIN_WALL_DIRAC
#define DD_IMPROVED 1
#include <staggered_dslash_def.h> // staggered Dslash kernels
#undef DD_IMPROVED

#include <dslash_quda.cuh>
  } // end namespace improvedstaggered

  // declare the dslash events
#include <dslash_events.cuh>

  using namespace improvedstaggered;

  template<typename T> struct RealType {};
  template<> struct RealType<double2> { typedef double type; };
  template<> struct RealType<float2> { typedef float type; };
  template<> struct RealType<float4> { typedef float type; };
  template<> struct RealType<short2> { typedef short type; };
  template<> struct RealType<short4> { typedef short type; };

#ifdef GPU_STAGGERED_DIRAC
  template <typename sFloat, typename fatGFloat, typename longGFloat, typename phaseFloat>
  class StaggeredDslashCuda : public DslashCuda {

  private:
    const fatGFloat *fat0, *fat1;
    const longGFloat *long0, *long1;
    const phaseFloat *phase0, *phase1;
    const double a;

  protected:
    unsigned int sharedBytesPerThread() const
    {
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return 6 * reg_size;
    }

  public:
    StaggeredDslashCuda(cudaColorSpinorField *out, const fatGFloat *fat0, const fatGFloat *fat1,
			const longGFloat *long0, const longGFloat *long1,
			const phaseFloat *phase0, const phaseFloat *phase1, 
			const QudaReconstructType reconstruct, const cudaColorSpinorField *in,
			const cudaColorSpinorField *x, const double a, const int dagger)
      : DslashCuda(out, in, x, reconstruct, dagger), fat0(fat0), fat1(fat1), long0(long0), 
	long1(long1), phase0(phase0), phase1(phase1), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x);
    }

    virtual ~StaggeredDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
      IMPROVED_STAGGERED_DSLASH(gridDim, tp.block, tp.shared_bytes, stream, dslashParam,
				(sFloat*)out->V(), (float*)out->Norm(), 
				fat0, fat1, long0, long1, phase0, phase1, 
				(sFloat*)in->V(), (float*)in->Norm(), 
				(sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a); 
    }

    int Nface() { return 6; } 

    /*
      per direction / dimension flops
      SU(3) matrix-vector flops = (8 Nc - 2) * Nc
      xpay = 2 * 2 * Nc * Ns
      
      So for the full dslash we have      
      flops = (2 * 2 * Nd * (8*Nc-2) * Nc)  +  ((2 * 2 * Nd - 1) * 2 * Nc * Ns)
      flops_xpay = flops + 2 * 2 * Nc * Ns
      
      For Asqtad this should give 1146 for Nc=3,Ns=2 and 1158 for the axpy equivalent
    */
    virtual long long flops() const {
      int mv_flops = (8 * in->Ncolor() - 2) * in->Ncolor(); // SU(3) matrix-vector flops
      int ghost_flops = (3 + 1) * (mv_flops + 2*in->Ncolor()*in->Nspin());
      int xpay_flops = 2 * 2 * in->Ncolor() * in->Nspin(); // multiply and add per real component
      int num_dir = 2 * 4; // dir * dim

      long long flops;
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
	flops = ghost_flops * 2 * in->GhostFace()[dslashParam.kernel_type];
	break;
      case EXTERIOR_KERNEL_ALL:
	{
	  long long ghost_sites = 2 * (in->GhostFace()[0]+in->GhostFace()[1]+in->GhostFace()[2]+in->GhostFace()[3]);
	  flops = ghost_flops * ghost_sites;
	  break;
	}
      case INTERIOR_KERNEL:
	{
	  long long sites = in->VolumeCB();
	  flops = (2*num_dir*mv_flops +                   // SU(3) matrix-vector multiplies
		   (2*num_dir-1)*2*in->Ncolor()*in->Nspin()) * sites;   // accumulation
	  if (x) flops += xpay_flops * sites; // axpy is always on interior

	  // now correct for flops done by exterior kernel
	  long long ghost_sites = 0;
	  for (int d=0; d<4; d++) if (dslashParam.commDim[d]) ghost_sites += 2 * in->GhostFace()[d];
	  flops -= ghost_flops * ghost_sites;
	  
	  break;
	}
      }
      return flops;
    }

    virtual long long bytes() const {
      int gauge_bytes_fat = QUDA_RECONSTRUCT_NO * in->Precision();
      int gauge_bytes_long = reconstruct * in->Precision();
      bool isHalf = in->Precision() == sizeof(short) ? true : false;
      int spinor_bytes = 2 * in->Ncolor() * in->Nspin() * in->Precision() + (isHalf ? sizeof(float) : 0);
      int ghost_bytes = 3 * (spinor_bytes + gauge_bytes_long) + (spinor_bytes + gauge_bytes_fat) + spinor_bytes;
      int num_dir = 2 * 4; // set to 4 dimensions since we take care of 5-d fermions in derived classes where necessary

      long long bytes;
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
	bytes = ghost_bytes * 2 * in->GhostFace()[dslashParam.kernel_type];
	break;
      case EXTERIOR_KERNEL_ALL:
	{
	  long long ghost_sites = 2 * (in->GhostFace()[0]+in->GhostFace()[1]+in->GhostFace()[2]+in->GhostFace()[3]);
	  bytes = ghost_bytes * ghost_sites;
	  break;
	}
      case INTERIOR_KERNEL:
	{
	  long long sites = in->VolumeCB();
	  bytes = (num_dir*(gauge_bytes_fat + gauge_bytes_long) + // gauge reads
		   num_dir*2*spinor_bytes +                       // spinor reads
		   spinor_bytes)*sites;                           // spinor write
	  if (x) bytes += spinor_bytes;

	  // now correct for bytes done by exterior kernel
	  long long ghost_sites = 0;
	  for (int d=0; d<4; d++) if (dslashParam.commDim[d]) ghost_sites += 2*in->GhostFace()[d];
	  bytes -= ghost_bytes * ghost_sites;
	  
	  break;
	}
      }
      return bytes;
    }

  };
#endif // GPU_STAGGERED_DIRAC

#include <dslash_policy.cuh>

  void improvedStaggeredDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &fatGauge, 
				   const cudaGaugeField &longGauge, const cudaColorSpinorField *in,
				   const int parity, const int dagger, const cudaColorSpinorField *x,
				   const double &k, const int *commOverride, TimeProfile &profile, const QudaDslashPolicy &dslashPolicy)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_STAGGERED_DIRAC

#ifdef MULTI_GPU
    for(int i=0;i < 4; i++){
      if(commDimPartitioned(i) && (fatGauge.X()[i] < 6)){
	errorQuda("ERROR: partitioned dimension with local size less than 6 is not supported in staggered dslash\n");
      }    
    }
#endif

    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code

    dslashParam.parity = parity;
    dslashParam.gauge_stride = fatGauge.Stride();
    dslashParam.long_gauge_stride = longGauge.Stride();
    dslashParam.fat_link_max = fatGauge.LinkMax();

    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }

    void *fatGauge0, *fatGauge1;
    void* longGauge0, *longGauge1;
    bindFatGaugeTex(fatGauge, parity, &fatGauge0, &fatGauge1);
    bindLongGaugeTex(longGauge, parity, &longGauge0, &longGauge1);
    void *longPhase0 = (char*)longGauge0 + longGauge.PhaseOffset();
    void *longPhase1 = (char*)longGauge1 + longGauge.PhaseOffset();   

    if (in->Precision() != fatGauge.Precision() || in->Precision() != longGauge.Precision()){
      errorQuda("Mixing gauge and spinor precision not supported"
		"(precision=%d, fatlinkGauge.precision=%d, longGauge.precision=%d",
		in->Precision(), fatGauge.Precision(), longGauge.Precision());
    }

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new StaggeredDslashCuda<double2, double2, double2, double>
	(out, (double2*)fatGauge0, (double2*)fatGauge1,
	 (double2*)longGauge0, (double2*)longGauge1,
	 (double*)longPhase0, (double*)longPhase1, 
	 longGauge.Reconstruct(), in, x, k, dagger);
      regSize = sizeof(double);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new StaggeredDslashCuda<float2, float2, float4, float>
	(out, (float2*)fatGauge0, (float2*)fatGauge1,
	 (float4*)longGauge0, (float4*)longGauge1, 
	 (float*)longPhase0, (float*)longPhase1,
	 longGauge.Reconstruct(), in, x, k, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {	
      dslash = new StaggeredDslashCuda<short2, short2, short4, short>
	(out, (short2*)fatGauge0, (short2*)fatGauge1,
	 (short4*)longGauge0, (short4*)longGauge1, 
	 (short*)longPhase0, (short*)longPhase1,
	 longGauge.Reconstruct(), in, x, k, dagger);
    }

#ifndef GPU_COMMS
    DslashPolicyImp* dslashImp = DslashFactory::create(dslashPolicy);
#else
    DslashPolicyImp* dslashImp = DslashFactory::create(QUDA_GPU_COMMS_DSLASH);
#endif
    (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), regSize, parity, dagger, in->Volume(), in->GhostFace(), profile);
    delete dslashImp;

    delete dslash;
    unbindFatGaugeTex(fatGauge);
    unbindLongGaugeTex(longGauge);

    checkCudaError();

#else
    errorQuda("Staggered dslash has not been built");
#endif  // GPU_STAGGERED_DIRAC
  }

}
