#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <gauge_field.h>
#include <tune_quda.h>

#include <tune_quda.h>
#include <quda_matrix.h>

#ifdef GPU_HISQ_FORCE

// work around for CUDA 7.0 bug on OSX
#if defined(__APPLE__) && CUDA_VERSION >= 7000 && CUDA_VERSION < 7050
#define EXPONENT_TYPE Real
#else
#define EXPONENT_TYPE int
#endif

namespace quda{
namespace {
  #include <svd_quda.h>
}

namespace { // anonymous
#include <svd_quda.h>
}

#define HISQ_UNITARIZE_PI 3.14159265358979323846
#define HISQ_UNITARIZE_PI23 HISQ_UNITARIZE_PI*2.0/3.0

// constants - File scope only
__constant__ double DEV_HISQ_UNITARIZE_EPS;
__constant__ double DEV_HISQ_FORCE_FILTER;
__constant__ double DEV_MAX_DET_ERROR;
__constant__ bool DEV_REUNIT_ALLOW_SVD;
__constant__ bool DEV_REUNIT_SVD_ONLY;
__constant__ double DEV_REUNIT_SVD_REL_ERROR;
__constant__ double DEV_REUNIT_SVD_ABS_ERROR;

static double HOST_HISQ_UNITARIZE_EPS;
static double HOST_HISQ_FORCE_FILTER;
static double HOST_MAX_DET_ERROR;
static bool   HOST_REUNIT_ALLOW_SVD;
static bool   HOST_REUNIT_SVD_ONLY;
static double HOST_REUNIT_SVD_REL_ERROR;
static double HOST_REUNIT_SVD_ABS_ERROR;


 
  namespace fermion_force{


    void setUnitarizeForceConstants(double unitarize_eps_h, double hisq_force_filter_h, 
				    double max_det_error_h, bool allow_svd_h, bool svd_only_h,
				    double svd_rel_error_h, double svd_abs_error_h)
    {

      // not_set is only initialised once
      static bool not_set=true;
		
      if(not_set){

	hipMemcpyToSymbol(HIP_SYMBOL(DEV_HISQ_UNITARIZE_EPS), &unitarize_eps_h, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(DEV_HISQ_FORCE_FILTER), &hisq_force_filter_h, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(DEV_MAX_DET_ERROR), &max_det_error_h, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(DEV_REUNIT_ALLOW_SVD), &allow_svd_h, sizeof(bool));
	hipMemcpyToSymbol(HIP_SYMBOL(DEV_REUNIT_SVD_ONLY), &svd_only_h, sizeof(bool));
	hipMemcpyToSymbol(HIP_SYMBOL(DEV_REUNIT_SVD_REL_ERROR), &svd_rel_error_h, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(DEV_REUNIT_SVD_ABS_ERROR), &svd_abs_error_h, sizeof(double));

	HOST_HISQ_UNITARIZE_EPS = unitarize_eps_h;
	HOST_HISQ_FORCE_FILTER = hisq_force_filter_h;
	HOST_MAX_DET_ERROR = max_det_error_h;     
	HOST_REUNIT_ALLOW_SVD = allow_svd_h;
	HOST_REUNIT_SVD_ONLY = svd_only_h;
	HOST_REUNIT_SVD_REL_ERROR = svd_rel_error_h;
	HOST_REUNIT_SVD_ABS_ERROR = svd_abs_error_h;
	not_set = false;
      }
      checkCudaError();
      return;
    }


    template<class Real>
    class DerivativeCoefficients{
    private:
      Real b[6]; 
      __device__ __host__       
      Real computeC00(const Real &, const Real &, const Real &);
      __device__ __host__
      Real computeC01(const Real &, const Real &, const Real &);
      __device__ __host__
      Real computeC02(const Real &, const Real &, const Real &);
      __device__ __host__
      Real computeC11(const Real &, const Real &, const Real &);
      __device__ __host__
      Real computeC12(const Real &, const Real &, const Real &);
      __device__ __host__
      Real computeC22(const Real &, const Real &, const Real &);

    public:
      __device__ __host__ void set(const Real & u, const Real & v, const Real & w);
      __device__ __host__
      Real getB00() const { return b[0]; }
      __device__ __host__
      Real getB01() const { return b[1]; }
      __device__ __host__
      Real getB02() const { return b[2]; }
      __device__ __host__
      Real getB11() const { return b[3]; }
      __device__ __host__
      Real getB12() const { return b[4]; }
      __device__ __host__
      Real getB22() const { return b[5]; }
    };

    template<class Real>
    __device__ __host__
    Real DerivativeCoefficients<Real>::computeC00(const Real & u, const Real & v, const Real & w){
      Real result = -pow(w,static_cast<EXPONENT_TYPE>(3)) * pow(u,static_cast<EXPONENT_TYPE>(6))
	+ 3*v*pow(w,static_cast<EXPONENT_TYPE>(3))*pow(u,static_cast<EXPONENT_TYPE>(4))
	+ 3*pow(v,static_cast<EXPONENT_TYPE>(4))*w*pow(u,static_cast<EXPONENT_TYPE>(4))
	-   pow(v,static_cast<EXPONENT_TYPE>(6))*pow(u,static_cast<EXPONENT_TYPE>(3))
	- 4*pow(w,static_cast<EXPONENT_TYPE>(4))*pow(u,static_cast<EXPONENT_TYPE>(3))
	- 12*pow(v,static_cast<EXPONENT_TYPE>(3))*pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(3))
	+ 16*pow(v,static_cast<EXPONENT_TYPE>(2))*pow(w,static_cast<EXPONENT_TYPE>(3))*pow(u,static_cast<EXPONENT_TYPE>(2))
	+ 3*pow(v,static_cast<EXPONENT_TYPE>(5))*w*pow(u,static_cast<EXPONENT_TYPE>(2))
	- 8*v*pow(w,static_cast<EXPONENT_TYPE>(4))*u
	- 3*pow(v,static_cast<EXPONENT_TYPE>(4))*pow(w,static_cast<EXPONENT_TYPE>(2))*u
	+ pow(w,static_cast<EXPONENT_TYPE>(5))
	+ pow(v,static_cast<EXPONENT_TYPE>(3))*pow(w,static_cast<EXPONENT_TYPE>(3));

      return result;
    }

    template<class Real>
    __device__ __host__
    Real DerivativeCoefficients<Real>::computeC01(const Real & u, const Real & v, const Real & w){
      Real result =  - pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(7))
	- pow(v,static_cast<EXPONENT_TYPE>(2))*w*pow(u,static_cast<EXPONENT_TYPE>(6))
	+ pow(v,static_cast<EXPONENT_TYPE>(4))*pow(u,static_cast<EXPONENT_TYPE>(5))   // This was corrected!
	+ 6*v*pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(5))
	- 5*pow(w,static_cast<EXPONENT_TYPE>(3))*pow(u,static_cast<EXPONENT_TYPE>(4))    // This was corrected!
	- pow(v,static_cast<EXPONENT_TYPE>(3))*w*pow(u,static_cast<EXPONENT_TYPE>(4))
	- 2*pow(v,static_cast<EXPONENT_TYPE>(5))*pow(u,static_cast<EXPONENT_TYPE>(3))
	- 6*pow(v,static_cast<EXPONENT_TYPE>(2))*pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(3))
	+ 10*v*pow(w,static_cast<EXPONENT_TYPE>(3))*pow(u,static_cast<EXPONENT_TYPE>(2))
	+ 6*pow(v,static_cast<EXPONENT_TYPE>(4))*w*pow(u,static_cast<EXPONENT_TYPE>(2))
	- 3*pow(w,static_cast<EXPONENT_TYPE>(4))*u
	- 6*pow(v,static_cast<EXPONENT_TYPE>(3))*pow(w,static_cast<EXPONENT_TYPE>(2))*u
	+ 2*pow(v,static_cast<EXPONENT_TYPE>(2))*pow(w,static_cast<EXPONENT_TYPE>(3));
      return result;
    }

    template<class Real>
    __device__ __host__
    Real DerivativeCoefficients<Real>::computeC02(const Real & u, const Real & v, const Real & w){
      Real result =   pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(5))
	+ pow(v,static_cast<EXPONENT_TYPE>(2))*w*pow(u,static_cast<EXPONENT_TYPE>(4))
	- pow(v,static_cast<EXPONENT_TYPE>(4))*pow(u,static_cast<EXPONENT_TYPE>(3))
	- 4*v*pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(3))
	+ 4*pow(w,static_cast<EXPONENT_TYPE>(3))*pow(u,static_cast<EXPONENT_TYPE>(2))
	+ 3*pow(v,static_cast<EXPONENT_TYPE>(3))*w*pow(u,static_cast<EXPONENT_TYPE>(2))
	- 3*pow(v,static_cast<EXPONENT_TYPE>(2))*pow(w,static_cast<EXPONENT_TYPE>(2))*u
	+ v*pow(w,static_cast<EXPONENT_TYPE>(3));
      return result;
    }

    template<class Real>
    __device__ __host__
    Real DerivativeCoefficients<Real>::computeC11(const Real & u, const Real & v, const Real & w){
      Real result = - w*pow(u,static_cast<EXPONENT_TYPE>(8))
	- pow(v,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(7))
	+ 7*v*w*pow(u,static_cast<EXPONENT_TYPE>(6))
	+ 4*pow(v,static_cast<EXPONENT_TYPE>(3))*pow(u,static_cast<EXPONENT_TYPE>(5))
	- 5*pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(5))
	- 16*pow(v,static_cast<EXPONENT_TYPE>(2))*w*pow(u,static_cast<EXPONENT_TYPE>(4))
	- 4*pow(v,static_cast<EXPONENT_TYPE>(4))*pow(u,static_cast<EXPONENT_TYPE>(3))
	+ 16*v*pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(3))
	- 3*pow(w,static_cast<EXPONENT_TYPE>(3))*pow(u,static_cast<EXPONENT_TYPE>(2))
	+ 12*pow(v,static_cast<EXPONENT_TYPE>(3))*w*pow(u,static_cast<EXPONENT_TYPE>(2))
	- 12*pow(v,static_cast<EXPONENT_TYPE>(2))*pow(w,static_cast<EXPONENT_TYPE>(2))*u
	+ 3*v*pow(w,static_cast<EXPONENT_TYPE>(3));
      return result;
    }

    template<class Real>
    __device__ __host__
    Real DerivativeCoefficients<Real>::computeC12(const Real & u, const Real & v, const Real & w){
      Real result =  w*pow(u,static_cast<EXPONENT_TYPE>(6))
	+ pow(v,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(5)) // Fixed this!
	- 5*v*w*pow(u,static_cast<EXPONENT_TYPE>(4))  // Fixed this!
	- 2*pow(v,static_cast<EXPONENT_TYPE>(3))*pow(u,static_cast<EXPONENT_TYPE>(3))
	+ 4*pow(w,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(3))
	+ 6*pow(v,static_cast<EXPONENT_TYPE>(2))*w*pow(u,static_cast<EXPONENT_TYPE>(2))
	- 6*v*pow(w,static_cast<EXPONENT_TYPE>(2))*u
	+ pow(w,static_cast<EXPONENT_TYPE>(3));
      return result;
    }

    template<class Real>
    __device__ __host__
    Real DerivativeCoefficients<Real>::computeC22(const Real & u, const Real & v, const Real & w){
      Real result = - w*pow(u,static_cast<EXPONENT_TYPE>(4))
	- pow(v,static_cast<EXPONENT_TYPE>(2))*pow(u,static_cast<EXPONENT_TYPE>(3))
	+ 3*v*w*pow(u,static_cast<EXPONENT_TYPE>(2))
	- 3*pow(w,static_cast<EXPONENT_TYPE>(2))*u;
      return result;
    }

    template <class Real>
    __device__ __host__
    void  DerivativeCoefficients<Real>::set(const Real & u, const Real & v, const Real & w){
      const Real & denominator = 2.0*pow(w*(u*v-w),static_cast<EXPONENT_TYPE>(3));
      b[0] = computeC00(u,v,w)/denominator;
      b[1] = computeC01(u,v,w)/denominator;
      b[2] = computeC02(u,v,w)/denominator;
      b[3] = computeC11(u,v,w)/denominator;
      b[4] = computeC12(u,v,w)/denominator;
      b[5] = computeC22(u,v,w)/denominator;
      return;
    }


    template<class Cmplx>
    __device__ __host__
    void accumBothDerivatives(Matrix<Cmplx,3>* result, const Matrix<Cmplx,3> & left, const Matrix<Cmplx,3> & right, const Matrix<Cmplx,3> & outer_prod)
    {
      const typename RealTypeId<Cmplx>::Type temp = 2.0*getTrace(left*outer_prod).x;
      for(int k=0; k<3; ++k){
	for(int l=0; l<3; ++l){
	  // Need to write it this way to get it to work 
	  // on the CPU. Not sure why.
	  result->operator()(k,l).x += temp*right(k,l).x;
	  result->operator()(k,l).y += temp*right(k,l).y;
	}
      }
      return;
    }


    template<class Cmplx>
    __device__ __host__
    void accumDerivatives(Matrix<Cmplx,3>* result, const Matrix<Cmplx,3> & left, const Matrix<Cmplx,3> & right, const Matrix<Cmplx,3> & outer_prod)
    {
      Cmplx temp = getTrace(left*outer_prod);
      for(int k=0; k<3; ++k){
	for(int l=0; l<3; ++l){
	  result->operator()(k,l) = temp*right(k,l);
	}
      }
      return;
    }


    template<class T>
    __device__ __host__
    T getAbsMin(const T* const array, int size){
      T min = fabs(array[0]);
      for(int i=1; i<size; ++i){
        T abs_val = fabs(array[i]);
        if((abs_val) < min){ min = abs_val; }   
      }
      return min;
    }


    template<class Real>
    __device__ __host__
    inline bool checkAbsoluteError(Real a, Real b, Real epsilon)
    {
      if( fabs(a-b) <  epsilon) return true;
      return false;
    }


    template<class Real>
    __device__ __host__ 
    inline bool checkRelativeError(Real a, Real b, Real epsilon)
    {
      if( fabs((a-b)/b)  < epsilon ) return true;
      return false;
    }
    



    // Compute the reciprocal square root of the matrix q
    // Also modify q if the eigenvalues are dangerously small.
    template<class Cmplx> 
    __device__  __host__ 
    void reciprocalRoot(Matrix<Cmplx,3>* res, DerivativeCoefficients<typename RealTypeId<Cmplx>::Type>* deriv_coeffs, 
			typename RealTypeId<Cmplx>::Type f[3], Matrix<Cmplx,3> & q, int *unitarization_failed){

      Matrix<Cmplx,3> qsq, tempq;

      typename RealTypeId<Cmplx>::Type c[3];
      typename RealTypeId<Cmplx>::Type g[3];

#ifdef __CUDA_ARCH__
#define REUNIT_SVD_ONLY DEV_REUNIT_SVD_ONLY
#else
#define REUNIT_SVD_ONLY HOST_REUNIT_SVD_ONLY
#endif
      if(!REUNIT_SVD_ONLY){
	qsq = q*q;
	tempq = qsq*q;

	c[0] = getTrace(q).x;
	c[1] = getTrace(qsq).x/2.0;
	c[2] = getTrace(tempq).x/3.0;

	g[0] = g[1] = g[2] = c[0]/3.;
	typename RealTypeId<Cmplx>::Type r,s,theta;
	s = c[1]/3. - c[0]*c[0]/18;
	r = c[2]/2. - (c[0]/3.)*(c[1] - c[0]*c[0]/9.);

#ifdef __CUDA_ARCH__
#define HISQ_UNITARIZE_EPS DEV_HISQ_UNITARIZE_EPS
#else
#define HISQ_UNITARIZE_EPS HOST_HISQ_UNITARIZE_EPS
#endif

	typename RealTypeId<Cmplx>::Type cosTheta = r/sqrt(s*s*s);
	if(fabs(s) < HISQ_UNITARIZE_EPS){
	  cosTheta = 1.;
	  s = 0.0; 
	}
	if(fabs(cosTheta)>1.0){ r>0 ? theta=0.0 : theta=HISQ_UNITARIZE_PI/3.0; }
	else{ theta = acos(cosTheta)/3.0; }

	s = 2.0*sqrt(s);
	for(int i=0; i<3; ++i){
	  g[i] += s*cos(theta + (i-1)*HISQ_UNITARIZE_PI23);
	}

      } // !REUNIT_SVD_ONLY?

	//
	// Compare the product of the eigenvalues computed thus far to the 
	// absolute value of the determinant. 
	// If the determinant is very small or the relative error is greater than some predefined value 
	// then recompute the eigenvalues using a singular-value decomposition.
	// Note that this particular calculation contains multiple branches, 
	// so it doesn't appear to be particularly well-suited to the GPU 
	// programming model. However, the analytic calculation of the 
	// unitarization is extremely fast, and if the SVD routine is not called 
	// too often, we expect pretty good performance.
	//

#ifdef __CUDA_ARCH__
#define REUNIT_ALLOW_SVD DEV_REUNIT_ALLOW_SVD
#define REUNIT_SVD_REL_ERROR DEV_REUNIT_SVD_REL_ERROR
#define REUNIT_SVD_ABS_ERROR DEV_REUNIT_SVD_ABS_ERROR
#else // cpu
#define REUNIT_ALLOW_SVD HOST_REUNIT_ALLOW_SVD
#define REUNIT_SVD_REL_ERROR HOST_REUNIT_SVD_REL_ERROR
#define REUNIT_SVD_ABS_ERROR HOST_REUNIT_SVD_ABS_ERROR
#endif

      if(REUNIT_ALLOW_SVD){
	bool perform_svd = true;
	if(!REUNIT_SVD_ONLY){
	  const typename RealTypeId<Cmplx>::Type det = getDeterminant(q).x;
	  if( fabs(det) >= REUNIT_SVD_ABS_ERROR){  
	    if( checkRelativeError(g[0]*g[1]*g[2],det,REUNIT_SVD_REL_ERROR) ) perform_svd = false;
	  }
	}	

	if(perform_svd){	
	  Matrix<Cmplx,3> tmp2;
	  // compute the eigenvalues using the singular value decomposition
	  computeSVD<Cmplx>(q,tempq,tmp2,g);
	  // The array g contains the eigenvalues of the matrix q
	  // The determinant is the product of the eigenvalues, and I can use this
	  // to check the SVD
	  const typename RealTypeId<Cmplx>::Type determinant = getDeterminant(q).x;
	  const typename RealTypeId<Cmplx>::Type gprod = g[0]*g[1]*g[2];
	  // Check the svd result for errors
#ifdef __CUDA_ARCH__
#define MAX_DET_ERROR DEV_MAX_DET_ERROR
#else
#define MAX_DET_ERROR HOST_MAX_DET_ERROR
#endif
	  if(fabs(gprod - determinant) > MAX_DET_ERROR){
	    printf("Warning: Error in determinant computed by SVD : %g > %g\n", fabs(gprod-determinant), MAX_DET_ERROR);
	    printLink(q);

#ifdef __CUDA_ARCH__
	    atomicAdd(unitarization_failed,1);
#else
	    (*unitarization_failed)++;
#endif
	  } 
	} // perform_svd?

      } // REUNIT_ALLOW_SVD?

#ifdef __CUDA_ARCH__
#define HISQ_FORCE_FILTER DEV_HISQ_FORCE_FILTER
#else
#define HISQ_FORCE_FILTER HOST_HISQ_FORCE_FILTER
#endif	
      typename RealTypeId<Cmplx>::Type delta = getAbsMin(g,3);
      if(delta < HISQ_FORCE_FILTER){
	for(int i=0; i<3; ++i){ 
	  g[i]     += HISQ_FORCE_FILTER; 
	  q(i,i).x += HISQ_FORCE_FILTER;
	}
	qsq = q*q; // recalculate Q^2
      }


      // At this point we have finished with the c's 
      // use these to store sqrt(g)
      for(int i=0; i<3; ++i) c[i] = sqrt(g[i]);

      // done with the g's, use these to store u, v, w
      g[0] = c[0]+c[1]+c[2];
      g[1] = c[0]*c[1] + c[0]*c[2] + c[1]*c[2];
      g[2] = c[0]*c[1]*c[2];
        
      // set the derivative coefficients!
      deriv_coeffs->set(g[0], g[1], g[2]);

      const typename RealTypeId<Cmplx>::Type & denominator  = g[2]*(g[0]*g[1]-g[2]); 
      c[0] = (g[0]*g[1]*g[1] - g[2]*(g[0]*g[0]+g[1]))/denominator;
      c[1] = (-g[0]*g[0]*g[0] - g[2] + 2.*g[0]*g[1])/denominator;
      c[2] =  g[0]/denominator;

      tempq = c[1]*q + c[2]*qsq;
      // Add a real scalar
      tempq(0,0).x += c[0];
      tempq(1,1).x += c[0];
      tempq(2,2).x += c[0];

      f[0] = c[0];
      f[1] = c[1];
      f[2] = c[2];

      *res = tempq;
      return;
    }



    // "v" denotes a "fattened" link variable
    template<class Cmplx>
    __device__ __host__
    void getUnitarizeForceSite(const Matrix<Cmplx,3> & v, const Matrix<Cmplx,3> & outer_prod, Matrix<Cmplx,3>* result, int *unitarization_failed)
    {
      typename RealTypeId<Cmplx>::Type f[3]; 
      typename RealTypeId<Cmplx>::Type b[6];

      Matrix<Cmplx,3> v_dagger = conj(v);  // okay!
      Matrix<Cmplx,3> q   = v_dagger*v;    // okay!

      Matrix<Cmplx,3> rsqrt_q;

      DerivativeCoefficients<typename RealTypeId<Cmplx>::Type> deriv_coeffs;

      reciprocalRoot<Cmplx>(&rsqrt_q, &deriv_coeffs, f, q, unitarization_failed); // approx 529 flops (assumes no SVD)

      // Pure hack here
      b[0] = deriv_coeffs.getB00();
      b[1] = deriv_coeffs.getB01();
      b[2] = deriv_coeffs.getB02();
      b[3] = deriv_coeffs.getB11();
      b[4] = deriv_coeffs.getB12();
      b[5] = deriv_coeffs.getB22();


      Matrix<Cmplx,3> & local_result = *result;

      local_result = rsqrt_q*outer_prod;

      // We are now finished with rsqrt_q
      Matrix<Cmplx,3> qv_dagger  = q*v_dagger;
      Matrix<Cmplx,3> vv_dagger  = v*v_dagger; 
      Matrix<Cmplx,3> vqv_dagger = v*qv_dagger;
      Matrix<Cmplx,3> temp = f[1]*vv_dagger + f[2]*vqv_dagger;


      temp = f[1]*v_dagger + f[2]*qv_dagger;
      Matrix<Cmplx,3> conj_outer_prod = conj(outer_prod);


      temp = f[1]*v + f[2]*v*q;
      local_result = local_result + outer_prod*temp*v_dagger + f[2]*q*outer_prod*vv_dagger;

      local_result = local_result + v_dagger*conj_outer_prod*conj(temp) + f[2]*qv_dagger*conj_outer_prod*v_dagger;


      // now done with vv_dagger, I think
      Matrix<Cmplx,3> qsqv_dagger = q*qv_dagger;
      Matrix<Cmplx,3> pv_dagger   = b[0]*v_dagger + b[1]*qv_dagger + b[2]*qsqv_dagger;
      accumBothDerivatives(&local_result, v, pv_dagger, outer_prod); // 41 flops

      Matrix<Cmplx,3> rv_dagger = b[1]*v_dagger + b[3]*qv_dagger + b[4]*qsqv_dagger;
      Matrix<Cmplx,3> vq = v*q;
      accumBothDerivatives(&local_result, vq, rv_dagger, outer_prod); // 41 flops

      Matrix<Cmplx,3> sv_dagger = b[2]*v_dagger + b[4]*qv_dagger + b[5]*qsqv_dagger;
      Matrix<Cmplx,3> vqsq = vq*q;
      accumBothDerivatives(&local_result, vqsq, sv_dagger, outer_prod); // 41 flops
      return;
      // 4528 flops - 17 matrix multiplies (198 flops each) + reciprocal root (approx 529 flops) + accumBothDerivatives (41 each) + miscellaneous
    } // get unit force term



    template<class Cmplx>
    __global__ void getUnitarizeForceField(const int threads, const Cmplx* link_even, const Cmplx* link_odd,
					   const Cmplx* old_force_even, const Cmplx* old_force_odd,
					   Cmplx* force_even, Cmplx* force_odd,
					   int* unitarization_failed)
    {
       
      int mem_idx = blockIdx.x*blockDim.x + threadIdx.x;
      // The number of GPU threads is equal to the local volume
      const int HALF_VOLUME = threads/2;
      if(mem_idx >= threads) return;
	
      Cmplx* force;
      const Cmplx* link;
      const Cmplx* old_force;

      force = force_even;
      link = link_even;
      old_force = old_force_even;
      if(mem_idx >= HALF_VOLUME){
	      mem_idx = mem_idx - HALF_VOLUME;
	      force = force_odd;
	      link = link_odd;
	      old_force = old_force_odd;
      }


      // This part of the calculation is always done in double precision
      Matrix<double2,3> v, result, oprod;
           
      for(int dir=0; dir<4; ++dir){
	loadLinkVariableFromArray(old_force, dir, mem_idx, HALF_VOLUME, &oprod);
	loadLinkVariableFromArray(link, dir, mem_idx, HALF_VOLUME, &v);

	getUnitarizeForceSite<double2>(v, oprod, &result, unitarization_failed); 

	writeLinkVariableToArray(result, dir, mem_idx, HALF_VOLUME, force); 
      } // 4*4528 flops per site
      return;
    } // getUnitarizeForceField


    void unitarizeForceCPU(cpuGaugeField& cpuOldForce, cpuGaugeField& cpuGauge, cpuGaugeField* cpuNewForce)
    {
      
      int num_failures = 0;	
      Matrix<double2,3> old_force, new_force, v;

      // I can change this code to make it much more compact

      const QudaGaugeFieldOrder order = cpuGauge.Order();

      if(order == QUDA_MILC_GAUGE_ORDER){
        for(int i=0; i<cpuGauge.Volume(); ++i){
	  for(int dir=0; dir<4; ++dir){
	    if(cpuGauge.Precision() == QUDA_SINGLE_PRECISION){
	      copyArrayToLink(&old_force, ((float*)(cpuOldForce.Gauge_p()) + (i*4 + dir)*18)); 
	      copyArrayToLink(&v, ((float*)(cpuGauge.Gauge_p()) + (i*4 + dir)*18)); 
	      getUnitarizeForceSite<double2>(v, old_force, &new_force, &num_failures);
	      copyLinkToArray(((float*)(cpuNewForce->Gauge_p()) + (i*4 + dir)*18), new_force); 
	    }else if(cpuGauge.Precision() == QUDA_DOUBLE_PRECISION){
	      copyArrayToLink(&old_force, ((double*)(cpuOldForce.Gauge_p()) + (i*4 + dir)*18)); 
	      copyArrayToLink(&v, ((double*)(cpuGauge.Gauge_p()) + (i*4 + dir)*18)); 
	      getUnitarizeForceSite<double2>(v, old_force, &new_force, &num_failures);
	      copyLinkToArray(((double*)(cpuNewForce->Gauge_p()) + (i*4 + dir)*18), new_force); 
	    } // precision?
	  } // dir
        } // i
      }else if(order == QUDA_QDP_GAUGE_ORDER){
        for(int dir=0; dir<4; ++dir){
          for(int i=0; i<cpuGauge.Volume(); ++i){
	    if(cpuGauge.Precision() == QUDA_SINGLE_PRECISION){
	      copyArrayToLink(&old_force, ((float**)(cpuOldForce.Gauge_p()))[dir] + i*18);
	      copyArrayToLink(&v, ((float**)(cpuGauge.Gauge_p()))[dir] + i*18);
	      getUnitarizeForceSite<double2>(v, old_force, &new_force, &num_failures);
	      copyLinkToArray(((float**)(cpuNewForce->Gauge_p()))[dir] + i*18, new_force);
	    }else if(cpuGauge.Precision() == QUDA_DOUBLE_PRECISION){
	      copyArrayToLink(&old_force, ((double**)(cpuOldForce.Gauge_p()))[dir] + i*18);
	      copyArrayToLink(&v, ((double**)(cpuGauge.Gauge_p()))[dir] + i*18);
	      getUnitarizeForceSite<double2>(v, old_force, &new_force, &num_failures);
	      copyLinkToArray(((double**)(cpuNewForce->Gauge_p()))[dir] + i*18, new_force);
	    }
          }
        }
      }else{
        errorQuda("Only MILC and QDP gauge orders supported\n");
      }
      return;
    } // unitarize_force_cpu

    class UnitarizeForceCuda : public Tunable {
    private:
      const cudaGaugeField &oldForce;
      const cudaGaugeField &gauge;
      cudaGaugeField &newForce;
      int *fails;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

      // don't tune the grid dimension
      bool tuneGridDim() const { return false; }
      unsigned int minThreads() const { return gauge.Volume(); }

    public:
      UnitarizeForceCuda(const cudaGaugeField& oldForce, const cudaGaugeField& gauge,  
			 cudaGaugeField& newForce, int* fails) : 
	oldForce(oldForce), gauge(gauge), newForce(newForce), fails(fails) { 
	writeAuxString("threads=%d,prec=%lu,stride=%d", 
		       gauge.Volume(), gauge.Precision(), gauge.Stride());
      }
      virtual ~UnitarizeForceCuda() { ; }

      void apply(const hipStream_t &stream) {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

	if(gauge.Precision() == QUDA_SINGLE_PRECISION){
	  getUnitarizeForceField<<<tp.grid,tp.block>>>(gauge.Volume(), (const float2*)gauge.Even_p(), (const float2*)gauge.Odd_p(),
						       (const float2*)oldForce.Even_p(), (const float2*)oldForce.Odd_p(),
						       (float2*)newForce.Even_p(), (float2*)newForce.Odd_p(), 
						       fails);
	}else if(gauge.Precision() == QUDA_DOUBLE_PRECISION){
	  getUnitarizeForceField<<<tp.grid,tp.block>>>(gauge.Volume(), (const double2*)gauge.Even_p(), (const double2*)gauge.Odd_p(),
						       (const double2*)oldForce.Even_p(), (const double2*)oldForce.Odd_p(),
						       (double2*)newForce.Even_p(), (double2*)newForce.Odd_p(), 
						       fails);      
	}
      }
      
      void preTune() { ; }
      void postTune() { hipMemset(fails, 0, sizeof(int)); } // reset fails counter
      
      long long flops() const { return 4ll*4528*gauge.Volume(); }
      
      TuneKey tuneKey() const { return TuneKey(gauge.VolString(), typeid(*this).name(), aux); }
    }; // UnitarizeForceCuda

    void unitarizeForceCuda(cudaGaugeField &cudaOldForce,
                            cudaGaugeField &cudaGauge, cudaGaugeField *cudaNewForce, int* unitarization_failed, long long *flops) {

      UnitarizeForceCuda unitarizeForce(cudaOldForce, cudaGauge, *cudaNewForce, unitarization_failed);
      unitarizeForce.apply(0);
      hipDeviceSynchronize(); // need to synchronize to ensure failure write has completed
      if(flops) *flops = unitarizeForce.flops(); 
      checkCudaError();
    }
    
    
  } // namespace fermion_force

//#endif
} // namespace quda


#endif
