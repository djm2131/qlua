#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <iostream>
#include <random_quda.h>
#include <hip/hip_runtime.h>
#include <quda_internal.h>

#include <comm_quda.h>
#include <index_helper.cuh>


namespace quda {

#ifdef GPU_GAUGE_ALG
  
#define BLOCKSDIVUP(a, b)  (((a)+(b)-1)/(b))


dim3 GetBlockDim(size_t threads, size_t size){
    int blockx = BLOCKSDIVUP(size, threads);
    dim3 blocks(blockx,1,1);
    return blocks;
}




#  define CUDA_SAFE_CALL_NO_SYNC( call) {                               \
        hipError_t err = call;                                           \
        if( hipSuccess != err) {                                       \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString( err) );     \
            exit(EXIT_FAILURE);                                         \
        } }

#  define CUDA_SAFE_CALL( call)     CUDA_SAFE_CALL_NO_SYNC(call);   

/**
    @brief CUDA kernel to initialize CURAND RNG states
    @param state CURAND RNG state array
    @param seed initial seed for RNG
    @param rng_size size of the CURAND RNG state array
    @param node_offset this parameter is used to skip ahead the index in the sequence, usefull for multigpu. 
*/
__global__ void 
kernel_random(cuRNGState *state, int seed, int rng_size, int node_offset ){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < rng_size){
        /* Each thread gets same seed, a different sequence number, no offset */
        hiprand_init(seed, id + node_offset, 0, &state[id]);
    }
}

struct rngArg{
    int comm_dim[4];
    int comm_coord[4];
    int X[4];
};


__global__ void 
kernel_random(cuRNGState *state, int seed, int rng_size, int node_offset, rngArg arg ){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < rng_size){
        /* Each thread gets same seed, a different sequence number, no offset */
    #ifndef MULTI_GPU
        hiprand_init(seed, id + node_offset, 0, &state[id]);
    #else

    int x[4];
    getCoords(x, id, arg.X, 0);
    for(int i=0; i<4;i++) x[i] += arg.comm_coord[i] * arg.X[i];
    int idd = ((((x[3] * arg.comm_dim[2] * arg.X[2] + x[2]) * arg.comm_dim[1] * arg.X[1]) + x[1] ) * arg.comm_dim[0] * arg.X[0] + x[0]) >> 1 ;
    hiprand_init(seed, idd, 0, &state[id]);
    #endif
    }
}

/**
    @brief Call CUDA kernel to initialize CURAND RNG states
    @param state CURAND RNG state array
    @param seed initial seed for RNG
    @param rng_size size of the CURAND RNG state array
    @param node_offset this parameter is used to skip ahead the index in the sequence, usefull for multigpu. 
*/
void launch_kernel_random(cuRNGState *state, int seed, int rng_size, int node_offset, int X[4]){  
    dim3 nthreads(128,1,1);
    dim3 nblocks = GetBlockDim(nthreads.x, rng_size);
    //CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>( kernel_random),	hipFuncCachePreferL1));
    #ifndef MULTI_GPU
    kernel_random<<<nblocks,nthreads>>>(state, seed, rng_size, node_offset);
    #else
    rngArg arg;
    for(int i=0; i < 4; i++){
        arg.comm_dim[i] = comm_dim(i);
        arg.comm_coord[i] = comm_coord(i);
        arg.X[i] = X[i];
    }
    kernel_random<<<nblocks,nthreads>>>(state, seed, rng_size, 0, arg);
    #endif
    hipDeviceSynchronize();
}

RNG::RNG(int rng_sizes, int seedin){
    rng_size = rng_sizes;
    seed = seedin;
    state = NULL;
    node_offset = 0;
    #ifdef MULTI_GPU
    for(int i=0; i<4;i++) X[i]=0;
    node_offset = comm_rank() * rng_sizes;
    #endif
#if defined(XORWOW)
    printfQuda("Using hiprandStateXORWOW\n");
#elif defined(RG32k3a)
    printfQuda("Using hiprandStateMRG32k3a\n");
#else
    printfQuda("Using hiprandStateMRG32k3a\n");
#endif
} 
RNG::RNG(int rng_sizes, int seedin, int XX[4]){
    rng_size = rng_sizes;
    seed = seedin;
    state = NULL;
    node_offset = 0;
    #ifdef MULTI_GPU
    for(int i=0; i<4;i++) X[i]=XX[i];
    node_offset = comm_rank() * rng_sizes;
    #endif
#if defined(XORWOW)
    printfQuda("Using hiprandStateXORWOW\n");
#elif defined(RG32k3a)
    printfQuda("Using hiprandStateMRG32k3a\n");
#else
    printfQuda("Using hiprandStateMRG32k3a\n");
#endif
} 




/**
    @brief Initialize CURAND RNG states
*/
void RNG::Init(){
	AllocateRNG();
	launch_kernel_random(state, seed, rng_size, node_offset, X);
}		
					

/**
    @brief Allocate Device memory for CURAND RNG states
*/
void RNG::AllocateRNG(){
    if(rng_size>0 && state == NULL){
        state = (cuRNGState*)device_malloc(rng_size * sizeof(cuRNGState));
        CUDA_SAFE_CALL(hipMemset( state , 0 , rng_size * sizeof(cuRNGState) ));
        printfQuda("Allocated array of random numbers with rng_size: %.2f MB\n", rng_size * sizeof(cuRNGState)/(float)(1048576));
    }
    else{
        errorQuda("Array of random numbers not allocated, array size: %d !\nExiting...\n",rng_size);
    }
}
/**
    @brief Release Device memory for CURAND RNG states
*/
void RNG::Release(){
    if(rng_size>0 && state != NULL){
        device_free(state);
        printfQuda("Free array of random numbers with rng_size: %.2f MB\n", rng_size * sizeof(cuRNGState)/(float)(1048576));
        rng_size = 0;
        state = NULL;
    }
}


/*! @brief Restore CURAND array states initialization */
void RNG::restore(){    
  hipError_t err = hipMemcpy(state, backup_state, rng_size * sizeof(cuRNGState), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    host_free(backup_state);
    printfQuda("ERROR: Failed to restore hiprand rng states array\n");
    errorQuda("Aborting");
  }
  host_free(backup_state);
}
/*! @brief Backup CURAND array states initialization */
void RNG::backup(){ 
  backup_state = (cuRNGState*) safe_malloc(rng_size * sizeof(cuRNGState));   
  hipError_t err = hipMemcpy(backup_state, state, rng_size * sizeof(cuRNGState), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    host_free(backup_state);
    printfQuda("ERROR: Failed to backup hiprand rng states array\n");
    errorQuda("Aborting");
  }
}

#endif // GPU_GAUGE_ALG

}
