#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>

#define QudaSumFloat double
#define QudaSumFloat2 double2
#define QudaSumFloat3 double3

#define REDUCE_MAX_BLOCKS 65536

#define checkSpinor(a, b)						\
  {									\
    if (a.Precision() != b.Precision())					\
      errorQuda("precisions do not match: %d %d", a.Precision(), b.Precision()); \
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %d %d", a.Length(), b.Length());	\
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

#define checkLength(a, b)						\
  {									\
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %d %d", a.Length(), b.Length());	\
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

static struct {
  const char *vol_str;
  const char *aux_str;
  char aux_tmp[quda::TuneKey::aux_n];
} blasStrings;

// These are used for reduction kernels
static QudaSumFloat *d_reduce=0;
static QudaSumFloat *h_reduce=0;
static QudaSumFloat *hd_reduce=0;
static hipEvent_t reduceEnd;
    
namespace quda {

  hipStream_t* getBlasStream();
    
  void* getDeviceReduceBuffer() { return d_reduce; }
  void* getMappedHostReduceBuffer() { return hd_reduce; }
  void* getHostReduceBuffer() { return h_reduce; }

  void initReduce()
  { 

    const int MaxReduce = 12;
    // reduction buffer size
    size_t bytes = MaxReduce*3*REDUCE_MAX_BLOCKS*sizeof(QudaSumFloat); // Factor of N for composite reductions


    if (!d_reduce) d_reduce = (QudaSumFloat *) device_malloc(bytes);
    
    // these arrays are actually oversized currently (only needs to be QudaSumFloat3)
    
    // if the device supports host-mapped memory then use a host-mapped array for the reduction
    if (!h_reduce) {
      // only use zero copy reductions when using 64-bit
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
      if(deviceProp.canMapHostMemory) {
	h_reduce = (QudaSumFloat *) mapped_malloc(bytes);	
	hipHostGetDevicePointer(&hd_reduce, h_reduce, 0); // set the matching device pointer
      } else 
#endif 
      {
	h_reduce = (QudaSumFloat *) pinned_malloc(bytes);
	hd_reduce = d_reduce;
      }
      memset(h_reduce, 0, bytes); // added to ensure that valgrind doesn't report h_reduce is unitialised
    }
    
    hipEventCreateWithFlags(&reduceEnd, hipEventDisableTiming);
    
    checkCudaError();
  }

  void endReduce(void)
  {
    if (d_reduce) {
      device_free(d_reduce);
      d_reduce = 0;
    }
    if (h_reduce) {
      host_free(h_reduce);
      h_reduce = 0;
    }
    hd_reduce = 0;
    
    hipEventDestroy(reduceEnd);
  }

  namespace reduce {

#include <texture.h>
#include <reduce_core.h>
#include <reduce_mixed_core.h>
    
  } // namespace reduce

  /**
     Base class from which all reduction functors should derive.
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct ReduceFunctor {
    
    //! pre-computation routine called before the "M-loop"
    virtual __device__ void pre() { ; }
    
    //! where the reduction is usually computed and any auxiliary operations
    virtual __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, 
				       FloatN &z, FloatN &w, FloatN &v) = 0;
    
    //! post-computation routine called after the "M-loop"
    virtual __device__ void post(ReduceType &sum) { ; }
    
  };

  /**
     Return the L1 norm of x
  */
  __device__ double norm1_(const double2 &a) { return fabs(a.x) + fabs(a.y); }
  __device__ double norm1_(const float2 &a) { return (double)fabs(a.x) + (double)fabs(a.y); }
  __device__ double norm1_(const float4 &a) { return (double)fabs(a.x) + (double)fabs(a.y) + (double)fabs(a.z) + (double)fabs(a.w); }

  template <typename ReduceType, typename Float2, typename FloatN>
  struct Norm1 : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Norm1(const Float2 &a, const Float2 &b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,FloatN  &w, FloatN &v) { sum += norm1_(x); }
    static int streams() { return 1; } //! total number of input and output streams
    static int flops() { return 2; } //! flops per element
  };

  double norm1Cuda(const cudaColorSpinorField &x) {
    cudaColorSpinorField &y = (cudaColorSpinorField&)x; // FIXME
#ifdef HOST_DEBUG
    return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,Norm1,0,0,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
#else
    errorQuda("L1 norm kernel only built when HOST_DEBUG is enabled");
    return 0.0;
#endif
  }

  /**
     Return the L2 norm of x
  */
  __device__ double norm2_(const double2 &a) { return a.x*a.x + a.y*a.y; }
  __device__ double norm2_(const float2 &a) { return (double)a.x*(double)a.x + (double)a.y*(double)a.y; }
  __device__ double norm2_(const float4 &a) { return (double)a.x*(double)a.x + (double)a.y*(double)a.y +
      (double)a.z*(double)a.z + (double)a.w*(double)a.w; }

  template <typename ReduceType, typename Float2, typename FloatN>
  struct Norm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Norm2(const Float2 &a, const Float2 &b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,FloatN  &w, FloatN &v) { sum += norm2_(x); }
    static int streams() { return 1; } //! total number of input and output streams
    static int flops() { return 2; } //! flops per element
  };

  double normCuda(const cudaColorSpinorField &x) {
    cudaColorSpinorField &y = (cudaColorSpinorField&)x; // FIXME
    return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,Norm2,0,0,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
  }

  /**
     Return the real dot product of x and y
  */
    __device__ double dot_(const double2 &a, const double2 &b) { return a.x*b.x + a.y*b.y; }
    __device__ double dot_(const float2 &a, const float2 &b) { return (double)a.x*(double)b.x + (double)a.y*(double)b.y; }
    __device__ double dot_(const float4 &a, const float4 &b) { return (double)a.x*(double)b.x + (double)a.y*(double)b.y + (double)a.z*(double)b.z + (double)a.w*(double)b.w; }

  template <typename ReduceType, typename Float2, typename FloatN>
  struct Dot : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Dot(const Float2 &a, const Float2 &b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { sum += dot_(x,y); }
    static int streams() { return 2; } //! total number of input and output streams
    static int flops() { return 2; } //! flops per element
  };

  double reDotProductCuda(cudaColorSpinorField &x, cudaColorSpinorField &y) {
    return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
  }

 
  void reDotProductCuda(double* result, std::vector<cudaColorSpinorField*>& x, std::vector<cudaColorSpinorField*>& y){
#ifndef SSTEP
    errorQuda("S-step code not built\n");
#else 
    switch(x.size()){
      case 1:
        reduce::multiReduceCuda<1,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 2:
        reduce::multiReduceCuda<2,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break; 
      case 3:
        reduce::multiReduceCuda<3,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 4:
        reduce::multiReduceCuda<4,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 5:
        reduce::multiReduceCuda<5,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 6:
        reduce::multiReduceCuda<6,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 7:
        reduce::multiReduceCuda<7,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 8:
        reduce::multiReduceCuda<8,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 9:
        reduce::multiReduceCuda<9,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 10:
        reduce::multiReduceCuda<10,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 11:
        reduce::multiReduceCuda<11,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 12:
        reduce::multiReduceCuda<12,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 13:
        reduce::multiReduceCuda<13,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 14:
        reduce::multiReduceCuda<14,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 15:
        reduce::multiReduceCuda<15,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 16:
        reduce::multiReduceCuda<16,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      default:
        errorQuda("Unsupported vector size");
        break;
    }
#endif // SSTEP
  } 


    /* 
      returns the real component of the dot product of a and b 
      and the norm of a
    */
  __device__ double2 dotNormA_(const double2 &a, const double2 &b)
  { return make_double2(a.x*b.x + a.y*b.y, a.x*a.x + a.y*a.y); }
 
  __device__ double2 dotNormA_(const float2 &a, const float2 &b)
    { return make_double2((double)a.x*(double)b.x + (double)a.y*(double)b.y, (double)a.x*(double)a.x + (double)a.y*(double)a.y); }

 
  __device__ double2 dotNormA_(const float4 &a, const float4 & b)
    { return make_double2((double)a.x*(double)b.x + (double)a.y*(double)b.y + (double)a.z*(double)b.z + (double)a.w*(double)b.w,
			  (double)a.x*(double)a.x + (double)a.y*(double)a.y + (double)a.z*(double)a.z + (double)a.w*(double)a.w); }



  template <typename ReduceType, typename Float2, typename FloatN>
  struct DotNormA : public ReduceFunctor<ReduceType, Float2, FloatN> {
    DotNormA(const Float2 &a, const Float2 &b){}
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,  FloatN &w, FloatN &v){sum += dotNormA_(x,y);}
    static int streams() { return 2; }
    static int flops() { return 4; }
  };

  double2 reDotProductNormACuda(cudaColorSpinorField &x,cudaColorSpinorField &y){
    return reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,DotNormA,0,0,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
  }


  /**
     First performs the operation y[i] = a*x[i]
     Return the norm of y
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct axpyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    axpyNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { 
      y += a.x*x; sum += norm2_(y); }
    static int streams() { return 3; } //! total number of input and output streams
    static int flops() { return 4; } //! flops per element
  };

  double axpyNormCuda(const double &a, cudaColorSpinorField &x, cudaColorSpinorField &y) {
    return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,axpyNorm2,0,1,0,0,0,false>
      (make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
  }

  /**
     First performs the operation y[i] = x[i] - y[i]
     Second returns the norm of y
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct xmyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
    xmyNorm2(const Float2 &a, const Float2 &b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { 
      y = x - y; sum += norm2_(y); }
    static int streams() { return 3; } //! total number of input and output streams
    static int flops() { return 3; } //! flops per element
  };

  double xmyNormCuda(cudaColorSpinorField &x, cudaColorSpinorField &y) {
    return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,xmyNorm2,0,1,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
  }


  /**
     Functor to perform the operation y += a * x  (complex-valued)
  */

  __device__ void Caxpy_(const float2 &a, const float4 &x, float4 &y) {
    y.x += a.x*x.x; y.x -= a.y*x.y;
    y.y += a.y*x.x; y.y += a.x*x.y;
    y.z += a.x*x.z; y.z -= a.y*x.w;
    y.w += a.y*x.z; y.w += a.x*x.w;
  }

  __device__ void Caxpy_(const float2 &a, const float2 &x, float2 &y) {
    y.x += a.x*x.x; y.x -= a.y*x.y;
    y.y += a.y*x.x; y.y += a.x*x.y;
  }

  __device__ void Caxpy_(const double2 &a, const double2 &x, double2 &y) {
    y.x += a.x*x.x; y.x -= a.y*x.y;
    y.y += a.y*x.x; y.y += a.x*x.y;
  }

  /**
     First performs the operation y[i] = a*x[i] + y[i] (complex-valued)
     Second returns the norm of y
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct caxpyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    caxpyNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { 
      Caxpy_(a, x, y); sum += norm2_(y); }
    static int streams() { return 3; } //! total number of input and output streams
    static int flops() { return 6; } //! flops per element
  };

  double caxpyNormCuda(const Complex &a, cudaColorSpinorField &x, cudaColorSpinorField &y) {
    return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,caxpyNorm2,0,1,0,0,0,false>
      (make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, x, x, x);
  }

  /**
     double caxpyXmayNormCuda(float a, float *x, float *y, n){}
   
     First performs the operation y[i] += a*x[i]
     Second performs the operator x[i] -= a*z[i]
     Third returns the norm of x
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct caxpyxmaznormx : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    caxpyxmaznormx(const Float2 &a, const Float2 &b) : a(a) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { Caxpy_(a, x, y); Caxpy_(-a, z, x); sum += norm2_(x); }
    static int streams() { return 5; } //! total number of input and output streams
    static int flops() { return 10; } //! flops per element
  };

  double caxpyXmazNormXCuda(const Complex &a, cudaColorSpinorField &x, 
			    cudaColorSpinorField &y, cudaColorSpinorField &z) {
    return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,caxpyxmaznormx,1,1,0,0,0,false>
      (make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
  }

  /**
     double cabxpyAxNormCuda(float a, complex b, float *x, float *y, n){}
   
     First performs the operation y[i] += a*b*x[i]
     Second performs x[i] *= a
     Third returns the norm of x
  */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct cabxpyaxnorm : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    Float2 b;
    cabxpyaxnorm(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { x *= a.x; Caxpy_(b, x, y); sum += norm2_(y); }
    static int streams() { return 4; } //! total number of input and output streams
    static int flops() { return 10; } //! flops per element
  };

  double cabxpyAxNormCuda(const double &a, const Complex &b, 
			  cudaColorSpinorField &x, cudaColorSpinorField &y) {
    return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,cabxpyaxnorm,1,1,0,0,0,false>
      (make_double2(a, 0.0), make_double2(REAL(b), IMAG(b)), x, y, x, x, x);
  }

  /**
     Returns complex-valued dot product of x and y
  */
  __device__ double2 cdot_(const double2 &a, const double2 &b) 
  { return make_double2(a.x*b.x + a.y*b.y, a.x*b.y - a.y*b.x); }
  __device__ double2 cdot_(const float2 &a, const float2 &b) 
      { return make_double2((double)a.x*(double)b.x + (double)a.y*(double)b.y,
			    (double)a.x*(double)b.y - (double)a.y*(double)b.x); }
  __device__ double2 cdot_(const float4 &a, const float4 &b) 
      { return make_double2((double)a.x*(double)b.x + (double)a.y*(double)b.y +
			    (double)a.z*(double)b.z + (double)a.w*(double)b.w,
			    (double)a.x*(double)b.y - (double)a.y*(double)b.x +
			    (double)a.z*(double)b.w - (double)a.w*(double)b.z); }

  template <typename ReduceType, typename Float2, typename FloatN>
  struct Cdot : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Cdot(const Float2 &a, const Float2 &b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { sum += cdot_(x,y); }
    static int streams() { return 2; } //! total number of input and output streams
    static int flops() { return 4; } //! flops per element
  };

  Complex cDotProductCuda(cudaColorSpinorField &x, cudaColorSpinorField &y) {
    double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    return Complex(cdot.x, cdot.y);
  }

  void cDotProductCuda(Complex* result, std::vector<cudaColorSpinorField*>& x, std::vector<cudaColorSpinorField*>& y){
#ifndef SSTEP
    errorQuda("S-step code not built\n");
#else
    double2* cdot = new double2[x.size()];

    switch(x.size()){
      case 1:
        reduce::multiReduceCuda<1,double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
        (cdot, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 6:
        reduce::multiReduceCuda<6,double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
        (cdot, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 10:
        reduce::multiReduceCuda<10,double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
        (cdot, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 14:
        reduce::multiReduceCuda<14,double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
        (cdot, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      default:
        errorQuda("Unsupported vector size\n");
        break;
    }

    for(int i=0; i<x.size(); ++i) result[i] = Complex(cdot[i].x,cdot[i].y);
    delete[] cdot;
#endif
  }

  /**
     double2 xpaycDotzyCuda(float2 *x, float a, float2 *y, float2 *z, int n) {}
   
     First performs the operation y = x + a*y
     Second returns cdot product (z,y)
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct xpaycdotzy : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    xpaycdotzy(const Float2 &a, const Float2 &b) : a(a) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { y = x + a.x*y; sum += cdot_(z,y); }
    static int streams() { return 4; } //! total number of input and output streams
    static int flops() { return 6; } //! flops per element
  };

  Complex xpaycDotzyCuda(cudaColorSpinorField &x, const double &a, cudaColorSpinorField &y, cudaColorSpinorField &z) {
    double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,xpaycdotzy,0,1,0,0,0,false>
      (make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
    return Complex(cdot.x, cdot.y);
  }

  /**
     double caxpyDotzyCuda(float a, float *x, float *y, float *z, n){}
   
     First performs the operation y[i] = a*x[i] + y[i]
     Second returns the dot product (z,y)
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct caxpydotzy : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    caxpydotzy(const Float2 &a, const Float2 &b) : a(a) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { Caxpy_(a, x, y); sum += cdot_(z,y); }
    static int streams() { return 4; } //! total number of input and output streams
    static int flops() { return 8; } //! flops per element
  };

  Complex caxpyDotzyCuda(const Complex &a, cudaColorSpinorField &x, cudaColorSpinorField &y,
			 cudaColorSpinorField &z) {
    double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,caxpydotzy,0,1,0,0,0,false>
      (make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
    return Complex(cdot.x, cdot.y);
  }

  /**
     First returns the dot product (x,y)
     Returns the norm of x
  */
  __device__ double3 cdotNormA_(const double2 &a, const double2 &b) 
  { return make_double3(a.x*b.x + a.y*b.y, a.x*b.y - a.y*b.x, a.x*a.x + a.y*a.y); }
  __device__ double3 cdotNormA_(const float2 &a, const float2 &b) 
  { return make_double3(a.x*b.x + a.y*b.y, a.x*b.y - a.y*b.x, a.x*a.x + a.y*a.y); }
  __device__ double3 cdotNormA_(const float4 &a, const float4 &b) 
  { return make_double3(a.x*b.x + a.y*b.y + a.z*b.z + a.w*b.w, 
			a.x*b.y - a.y*b.x + a.z*b.w - a.w*b.z,
			a.x*a.x + a.y*a.y + a.z*a.z + a.w*a.w); }

  template <typename ReduceType, typename Float2, typename FloatN>
  struct CdotNormA : public ReduceFunctor<ReduceType, Float2, FloatN> {
    CdotNormA(const Float2 &a, const Float2 &b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { sum += cdotNormA_(x,y); }
    static int streams() { return 2; } //! total number of input and output streams
    static int flops() { return 6; } //! flops per element
  };

  double3 cDotProductNormACuda(cudaColorSpinorField &x, cudaColorSpinorField &y) {
    return reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,CdotNormA,0,0,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
  }

  /**
     First returns the dot product (x,y)
     Returns the norm of y
  */
  __device__ double3 cdotNormB_(const double2 &a, const double2 &b) 
  { return make_double3(a.x*b.x + a.y*b.y, a.x*b.y - a.y*b.x, b.x*b.x + b.y*b.y); }
  __device__ double3 cdotNormB_(const float2 &a, const float2 &b) 
  { return make_double3(a.x*b.x + a.y*b.y, a.x*b.y - a.y*b.x, b.x*b.x + b.y*b.y); }
  __device__ double3 cdotNormB_(const float4 &a, const float4 &b) 
  { return make_double3(a.x*b.x + a.y*b.y + a.z*b.z + a.w*b.w, a.x*b.y - a.y*b.x + a.z*b.w - a.w*b.z,
			b.x*b.x + b.y*b.y + b.z*b.z + b.w*b.w); }

  template <typename ReduceType, typename Float2, typename FloatN>
  struct CdotNormB : public ReduceFunctor<ReduceType, Float2, FloatN> {
    CdotNormB(const Float2 &a, const Float2 &b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { sum += cdotNormB_(x,y); }
    static int streams() { return 2; } //! total number of input and output streams
    static int flops() { return 6; } //! flops per element
  };

  double3 cDotProductNormBCuda(cudaColorSpinorField &x, cudaColorSpinorField &y) {
    return reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,CdotNormB,0,0,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
  }

  /**
     This convoluted kernel does the following: 
     z += a*x + b*y, y -= b*w, norm = (y,y), dot = (u, y)
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct caxpbypzYmbwcDotProductUYNormY : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    Float2 b;
    caxpbypzYmbwcDotProductUYNormY(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { Caxpy_(a, x, z); Caxpy_(b, y, z); Caxpy_(-b, w, y); sum += cdotNormB_(v,y); }
    static int streams() { return 7; } //! total number of input and output streams
    static int flops() { return 18; } //! flops per element
  };

  double3 caxpbypzYmbwcDotProductUYNormYCuda(const Complex &a, cudaColorSpinorField &x, 
					     const Complex &b, cudaColorSpinorField &y,
					     cudaColorSpinorField &z, cudaColorSpinorField &w,
					     cudaColorSpinorField &u) {
    if (x.Precision() != z.Precision()) {
      return reduce::mixed::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,caxpbypzYmbwcDotProductUYNormY,0,1,1,0,0,false>
      (make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);

    } else {
      return reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,caxpbypzYmbwcDotProductUYNormY,0,1,1,0,0,false>
      (make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
    }
  }


  /**
     Specialized kernel for the modified CG norm computation for
     computing beta.  Computes y = y + a*x and returns norm(y) and
     dot(y, delta(y)) where delta(y) is the difference between the
     input and out y vector.
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct axpyCGNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    axpyCGNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { 
      FloatN y_new = y + a.x*x;
      sum.x += norm2_(y_new); 
      sum.y += dot_(y_new, y_new-y);
      y = y_new;
    }
    static int streams() { return 3; } //! total number of input and output streams
    static int flops() { return 6; } //! flops per real element
  };

  Complex axpyCGNormCuda(const double &a, cudaColorSpinorField &x, cudaColorSpinorField &y) {
    double2 cg_norm = reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,axpyCGNorm2,0,1,0,0,0,false>
      (make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    return Complex(cg_norm.x, cg_norm.y);
  }

  /**
     This kernel returns (x, x) and (r,r) and also returns the so-called
     heavy quark norm as used by MILC: 1 / N * \sum_i (r, r)_i / (x, x)_i, where
     i is site index and N is the number of sites.
     
     When this kernel is launched, we must enforce that the parameter M
     in the launcher corresponds to the number of FloatN fields used to
     represent the spinor, e.g., M=6 for Wilson and M=3 for staggered.
     This is only the case for half-precision kernels by default.  To
     enable this, the siteUnroll template parameter must be set true
     when reduceCuda is instantiated.
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct HeavyQuarkResidualNorm : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    Float2 b;
    ReduceType aux;
    HeavyQuarkResidualNorm(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
    
    __device__ void pre() { aux.x = 0; aux.y = 0; }
    
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { aux.x += norm2_(x); aux.y += norm2_(y); }
    
    //! sum the solution and residual norms, and compute the heavy-quark norm
    __device__ void post(ReduceType &sum) 
    { 
      sum.x += aux.x; sum.y += aux.y; sum.z += (aux.x > 0.0) ? (aux.y / aux.x) : 1.0; 
    }
    
    static int streams() { return 2; } //! total number of input and output streams
    static int flops() { return 4; } //! undercounts since it excludes the per-site division
  };
  
  double3 HeavyQuarkResidualNormCuda(cudaColorSpinorField &x, cudaColorSpinorField &r) {
    double3 rtn = reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,HeavyQuarkResidualNorm,0,0,0,0,0,true>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, r, r, r, r);
#ifdef MULTI_GPU
    rtn.z /= (x.Volume()*comm_size());
#else
    rtn.z /= x.Volume();
#endif
    return rtn;
  }
  
  /**
     Variant of the HeavyQuarkResidualNorm kernel: this takes three
     arguments, the first two are summed together to form the
     solution, with the third being the residual vector.  This removes
     the need an additional xpy call in the solvers, impriving
     performance.
  */
  template <typename ReduceType, typename Float2, typename FloatN>
  struct xpyHeavyQuarkResidualNorm : public ReduceFunctor<ReduceType, Float2, FloatN> {
    Float2 a;
    Float2 b;
    ReduceType aux;
    xpyHeavyQuarkResidualNorm(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
    
    __device__ void pre() { aux.x = 0; aux.y = 0; }
    
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) 
    { aux.x += norm2_(x + y); aux.y += norm2_(z); }
    
    //! sum the solution and residual norms, and compute the heavy-quark norm
    __device__ void post(ReduceType &sum) 
    { 
      sum.x += aux.x; sum.y += aux.y; sum.z += (aux.x > 0.0) ? (aux.y / aux.x) : 1.0; 
    }
    
    static int streams() { return 3; } //! total number of input and output streams
    static int flops() { return 5; }
  };
  
  double3 xpyHeavyQuarkResidualNormCuda(cudaColorSpinorField &x, cudaColorSpinorField &y,
					cudaColorSpinorField &r) {
    double3 rtn = reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,xpyHeavyQuarkResidualNorm,0,0,0,0,0,true>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, r, r, r);
#ifdef MULTI_GPU
    rtn.z /= (x.Volume()*comm_size());
#else
    rtn.z /= x.Volume();
#endif
    return rtn;
  }

  /**
     double3 tripleCGUpdate(V x, V y, V z){}

     First performs the operation norm2(x)
     Second performs the operatio norm2(y)
     Third performs the operation dotPropduct(y,z)
  */

  template <typename ReduceType, typename Float2, typename FloatN>
  struct tripleCGReduction : public ReduceFunctor<ReduceType, Float2, FloatN> {
    tripleCGReduction(const Float2 &a, const Float2 &b) { ; }
    __device__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) 
    { sum.x += norm2_(x); sum.y += norm2_(y); sum.z += dot_(y,z); }
    static int streams() { return 3; } //! total number of input and output streams
    static int flops() { return 6; } //! flops per element
  };

  double3 tripleCGReductionCuda(cudaColorSpinorField &x, cudaColorSpinorField &y, cudaColorSpinorField &z) {
    return reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,tripleCGReduction,0,0,0,0,0,false>
      (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
  }

} // namespace quda
